#include "hip/hip_runtime.h"
#include "../include/gpuquery.cuh"

#include "../include/gpucommon.cuh"

GPUquery::GPUquery(/* args */) {
}

GPUquery::~GPUquery() {
    // this->transactions_batch_ptr.reset();
}

int GPUquery::test(int input) {
    std::cout << "Task " << input << " is executing" << std::endl;
    return input;
}

void GPUquery::malloc_global_txn(std::shared_ptr<Param> param,
                                 std::shared_ptr<std::vector<std::any> > transactions_batch_ptr,
                                 Global_Txn_Info *global_txn_info) {
    std::cout << "start gpuquery.cu GPUquery::malloc_global_txn()" << std::endl;

    // for (uint32_t i = 0; i < param->device_cnt; ++i) {
    //     CHECK(hipSetDevice(param->device_IDs[i]));
    //     for (uint32_t j = 0; j < param->device_cnt; ++j) {
    //         if (param->device_IDs[i]!= param->device_IDs[j]) {
    //             CHECK(hipDeviceEnablePeerAccess(param->device_IDs[j],0));
    //         }
    //     }
    // }
    // for (uint32_t i = 0; i < param->device_cnt; ++i) {
    //     CHECK(hipSetDevice(param->device_IDs[i]));
    //     for (uint32_t j = 0; j < param->device_cnt; ++j) {
    //         if (param->device_IDs[i]!= param->device_IDs[j]) {
    //             CHECK(hipDeviceDisablePeerAccess(param->device_IDs[j]));
    //         }
    //     }
    // }

    CHECK(hipHostAlloc((void **)&global_txn_info_d, sizeof(Global_Txn_Info *) * param->device_cnt, hipHostMallocDefault
    ));
    CHECK(hipHostAlloc((void **)&global_txn_info_h, sizeof(Global_Txn_Info) * param->get_subtxn_kinds(),
        hipHostMallocDefault));

    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipSetDevice(param->device_IDs[i]));
        CHECK(hipMalloc((void **)&global_txn_info_d[i], sizeof(Global_Txn_Info) * param->get_subtxn_kinds()));
    }

    CHECK(hipHostAlloc((void **)&global_txn_d, sizeof(Global_Txn *) * param->device_cnt, hipHostMallocDefault));
    CHECK(hipHostAlloc((void **)&global_txn_h, sizeof(Global_Txn *) * param->device_cnt, hipHostMallocDefault));
    CHECK(hipHostAlloc((void **)&global_txn, sizeof(Global_Txn) * param->get_sub_txn_size(), hipHostMallocDefault));

    for (size_t i = 0; i < param->device_cnt; ++i) {
        // TODO: 子事务集如何存储/子事务集合内部内存开辟
        CHECK(hipSetDevice(param->device_IDs[i]));
        CHECK(hipMalloc((void **)&global_txn_d[i], sizeof(Global_Txn) * param->get_sub_txn_size()));
        CHECK(hipHostAlloc((void **)&global_txn_h[i], sizeof(Global_Txn) * param->get_sub_txn_size(),
            hipHostMallocDefault));
    }

    CHECK(hipHostAlloc((void **)&global_txn_exec_h, sizeof(Global_Txn_Exec *) * param->device_cnt, hipHostMallocDefault
    ));
    CHECK(hipHostAlloc((void **)&global_txn_exec_d, sizeof(Global_Txn_Exec *) * param->device_cnt, hipHostMallocDefault
    ));
    for (size_t i = 0; i < param->device_cnt; ++i) {
        // TODO: 子事务集如何存储/子事务集合内部内存开辟
        CHECK(hipSetDevice(param->device_IDs[i]));
        CHECK(hipMalloc((void **)&global_txn_exec_d[i], sizeof(Global_Txn_Exec)));
        CHECK(hipHostAlloc((void **)&global_txn_exec_h[i], sizeof(Global_Txn_Exec), hipHostMallocDefault));
    }

    CHECK(hipHostAlloc((void **)&global_txn_result_d, sizeof(Global_Txn *) * param->device_cnt, hipHostMallocDefault));
    CHECK(hipHostAlloc((void **)&global_txn_result_h, sizeof(Global_Txn *) * param->device_cnt, hipHostMallocDefault));

    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipSetDevice(param->device_IDs[i]));
        CHECK(hipMalloc((void **)&global_txn_result_d[i], sizeof(Global_Txn) * param->get_sub_txn_size()));
        CHECK(hipHostAlloc((void **)&global_txn_result_h[i], sizeof(Global_Txn) * param->get_sub_txn_size(),
            hipHostMallocDefault));
    }

    CHECK(hipHostAlloc((void **)&exec_param_h, sizeof(Global_Txn_Exec_Param) * param->get_subtxn_kinds(),
        hipHostMallocDefault));
    CHECK(hipHostAlloc((void **)&exec_param_d, sizeof(Global_Txn_Exec_Param *) * param->device_cnt,
        hipHostMallocDefault));

    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipSetDevice(param->device_IDs[i]));
        CHECK(hipMalloc((void **)&exec_param_d[i], sizeof(Global_Txn_Exec_Param) * param->get_subtxn_kinds()));
    }

    CHECK(hipHostAlloc((void **)&aux_struct_d, sizeof(Global_Txn_Aux_Struct *) * param->device_cnt,
        hipHostMallocDefault));
    CHECK(hipHostAlloc((void **)&aux_struct_d_h, sizeof(Global_Txn_Aux_Struct *) * param->device_cnt,
        hipHostMallocDefault));

    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipSetDevice(param->device_IDs[i]));
        CHECK(hipMalloc((void **)&aux_struct_d[i], sizeof(Global_Txn_Aux_Struct) * param->table_cnt));
        CHECK(hipHostAlloc((void **)&aux_struct_d_h[i], sizeof(Global_Txn_Aux_Struct) * param->table_cnt,
            hipHostMallocDefault));
    }

    CHECK(hipHostAlloc((void **)&data_packet_d, sizeof(Global_Data_Packet *) * param->device_cnt, hipHostMallocDefault
    ));
    CHECK(hipHostAlloc((void **)&data_packet_h, sizeof(Global_Data_Packet *) * param->device_cnt, hipHostMallocDefault
    ));


    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipSetDevice(param->device_IDs[i]));
        CHECK(hipMalloc((void **)&data_packet_d[i],
            sizeof(Global_Data_Packet) * param->get_datapacket_size() ));
        CHECK(hipHostAlloc((void **)&data_packet_h[i],
            sizeof(Global_Data_Packet) * param->get_datapacket_size() ,
            hipHostMallocDefault));
        // #ifdef LTPMG_GPUQUERY_PRINTSIZE
        std::cout << "data_packet_d[" << i << "] is " << param->get_datapacket_size() << std::endl;
        std::cout << "data_packet_h[" << i << "] is " << param->get_datapacket_size() << std::endl;
        // #endif
    }

    // 生成参数
    if (param->benchmark == "TEST") {
        gen_param<Test_Query>(param);

        for (size_t i = 0; i < param->device_cnt; ++i) {
            CHECK(hipSetDevice(param->device_IDs[i]));

            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap_all_row, sizeof(UINT32) * param->get_sub_txn_size()
            ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].bitmap_all_row, sizeof(UINT32) * param->get_sub_txn_size()
            ));

            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap, sizeof(UINT32) * param->get_bitmap_size() * param->
                test_1_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].bitmap, sizeof(UINT32) * param->get_bitmap_size() * param->
                test_2_size/param->device_cnt));

            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap_tmp, sizeof(UINT32) * param->get_bitmap_size() *
                param->test_1_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].bitmap_tmp, sizeof(UINT32) * param->get_bitmap_size() *
                param->test_2_size/param->device_cnt));

            CHECK(hipMalloc((void**)&aux_struct_d_h[i][0].bitmap_mark,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param-> test_1_size/param->device_cnt));
            CHECK(hipMalloc((void**)&aux_struct_d_h[i][1].bitmap_mark,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param-> test_2_size/param->device_cnt));

            CHECK(hipMalloc((void**)&aux_struct_d_h[i][0].bitmap_mark_offset,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param-> test_1_size/param->device_cnt));
            CHECK(hipMalloc((void**)&aux_struct_d_h[i][1].bitmap_mark_offset,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param-> test_2_size/param->device_cnt));

            CHECK(hipMalloc((void**)&aux_struct_d_h[i][0].bitmap_mark_compressed,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param-> test_1_size/param->device_cnt));
            CHECK(hipMalloc((void**)&aux_struct_d_h[i][1].bitmap_mark_compressed,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param-> test_2_size/param->device_cnt));


            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap_used_size,
                sizeof(UINT32) * 1 * param->test_1_size/ param-> device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].bitmap_used_size,
                sizeof(UINT32) * 1 * param->test_2_size/ param-> device_cnt));

            // CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].min_TID, sizeof(UINT32) * 1 * param->test_1_size/param->
            //     device_cnt));
            // CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].min_TID, sizeof(UINT32) * 1 * param->test_2_size/param->
            //     device_cnt));

            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].cnt_TID, sizeof(UINT32) * 1 * param->test_1_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].cnt_TID, sizeof(UINT32) * 1 * param->test_2_size/param->
                device_cnt));

            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].tmp_TID, sizeof(UINT32) * 1 * param->test_1_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].tmp_TID, sizeof(UINT32) * 1 * param->test_2_size/param->
                device_cnt));

            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].mark_TID_offset, sizeof(UINT32) * 1 * param->test_1_size/
                param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].mark_TID_offset, sizeof(UINT32) * 1 * param->test_2_size/
                param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].used_rows, sizeof(UINT32) * param->get_sub_txn_size()*2));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].used_rows, sizeof(UINT32) * param->get_sub_txn_size()*2));

            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].mark_TID,
                sizeof(UINT32) * param->get_sub_txn_size()*2));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].merge_tmp,
                sizeof(UINT32) * param->get_sub_txn_size()*2));


#ifdef LTPMG_GPUQUERY_PRINTSIZE
            std::cout << "aux_struct_d_h[" << i << "][0].bitmap is " << param->get_bitmap_size() * param->test_1_size <<
                std::endl;
            std::cout << "aux_struct_d_h[" << i << "][1].bitmap is " << param->get_bitmap_size() * param->test_2_size <<
                std::endl;
            std::cout << "aux_struct_d_h[" << i << "][0].min_TID is " << 1 * param->test_1_size << std::endl;
            std::cout << "aux_struct_d_h[" << i << "][1].min_TID is " << 1 * param->test_2_size << std::endl;
#endif

            CHECK(hipMalloc((void **)&global_txn_exec_h[i][0].select_txn_mark, sizeof(UINT32) * param->
                get_select_batch_size()));
            CHECK(hipMalloc((void **)&global_txn_exec_h[i][0].insert_txn_mark, sizeof(UINT32) * param->
                get_insert_batch_size()));
            CHECK(hipMalloc((void **)&global_txn_exec_h[i][0].update_txn_mark, sizeof(UINT32) * param->
                get_update_batch_size()));
            CHECK(hipMalloc((void **)&global_txn_exec_h[i][0].scan_txn_mark, sizeof(UINT32) * param->
                get_scan_batch_size()));
            CHECK(hipMalloc((void **)&global_txn_exec_h[i][0].delete_txn_mark, sizeof(UINT32) * param->
                get_delete_batch_size()));

#ifdef LTPMG_GPUQUERY_PRINTSIZE
            std::cout << "global_txn_exec_h[" << i << "][0].select_txn is " << param->get_select_batch_size() << std::endl;
            std::cout << "global_txn_exec_h[" << i << "][0].insert_txn is " << param->get_insert_batch_size() << std::endl;
            std::cout << "global_txn_exec_h[" << i << "][0].update_txn is " << param->get_update_batch_size() << std::endl;
            std::cout << "global_txn_exec_h[" << i << "][0].scan_txn is " << param->get_scan_batch_size() << std::endl;
            std::cout << "global_txn_exec_h[" << i << "][0].delete_txn is " << param->get_delete_batch_size() << std::endl;
#endif
        }
    } else if (param->benchmark == "TPCC_PART") {
        gen_param<TPCC_PART>(param);

        for (size_t i = 0; i < param->device_cnt; ++i) {
            CHECK(hipSetDevice(param->device_IDs[i]));

            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap,
                sizeof(UINT32) * param->get_bitmap_size() * param-> warehouse_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap_all_row,
                sizeof(UINT32) * param->get_sub_txn_size() ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap_tmp,
                sizeof(UINT32) * param->get_bitmap_size() * param-> warehouse_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap_mark,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param-> warehouse_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap_mark_offset,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param-> warehouse_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap_mark_compressed,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param-> warehouse_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap_used_size,
                sizeof(UINT32) * 1 * param->warehouse_size /param-> device_cnt));
            // CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].min_TID, sizeof(UINT32) * 1 * param->warehouse_size/param->
                // device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].cnt_TID, sizeof(UINT32) * 1 * param->warehouse_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].tmp_TID, sizeof(UINT32) * 1 * param->warehouse_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].mark_TID_offset,
                sizeof(UINT32) * 1 * param->warehouse_size/param->device_cnt ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].used_rows, sizeof(UINT32) * param->get_sub_txn_size()*2));


            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].bitmap,
                sizeof(UINT32) * param->get_bitmap_size() * param-> district_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].bitmap_all_row,
                sizeof(UINT32) * param->get_sub_txn_size() ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].bitmap_tmp,
                sizeof(UINT32) * param->get_bitmap_size() * param-> district_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].bitmap_mark,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param-> district_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].bitmap_mark_offset,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param-> district_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].bitmap_mark_compressed,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param-> district_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].bitmap_used_size,
                sizeof(UINT32) * 1 * param->district_size/ param-> device_cnt));
            // CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].min_TID, sizeof(UINT32) * 1 * param->district_size/param->
                // device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].cnt_TID, sizeof(UINT32) * 1 * param->district_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].tmp_TID, sizeof(UINT32) * 1 * param->district_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].mark_TID_offset,
                sizeof(UINT32) * 1 * param->district_size/param->device_cnt ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].used_rows, sizeof(UINT32) * param->get_sub_txn_size()*2));


            // CHECK(hipMalloc((void **)&aux_struct_d_h[i][2].min_TID, sizeof(UINT32) * 1 * param->customer_size/param->
                // device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][2].cnt_TID, sizeof(UINT32) * 1 * param->customer_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][2].tmp_TID, sizeof(UINT32) * 1 * param->customer_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][2].mark_TID_offset,
                sizeof(UINT32) * 1 * param->customer_size/param->device_cnt ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][2].used_rows, sizeof(UINT32) * param->get_sub_txn_size()*2));


            // CHECK(hipMalloc((void **)&aux_struct_d_h[i][3].min_TID, sizeof(UINT32) * 1 * param->neworder_size/param->
                // device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][3].cnt_TID, sizeof(UINT32) * 1 * param->neworder_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][3].tmp_TID, sizeof(UINT32) * 1 * param->neworder_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][3].mark_TID_offset,
                sizeof(UINT32) * 1 * param->neworder_size/param->device_cnt ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][3].used_rows, sizeof(UINT32) * param->get_sub_txn_size()*2));


            // CHECK(hipMalloc((void **)&aux_struct_d_h[i][4].min_TID, sizeof(UINT32) * 1 * param->history_size/param->
                // device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][4].cnt_TID, sizeof(UINT32) * 1 * param->history_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][4].tmp_TID, sizeof(UINT32) * 1 * param->history_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][4].mark_TID_offset,
                sizeof(UINT32) * 1 * param->history_size/param->device_cnt ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][4].used_rows, sizeof(UINT32) * param->get_sub_txn_size()*2));

            // CHECK(hipMalloc((void **)&aux_struct_d_h[i][5].min_TID, sizeof(UINT32) * 1 * param->order_size/param->
                // device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][5].cnt_TID, sizeof(UINT32) * 1 * param->order_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][5].tmp_TID, sizeof(UINT32) * 1 * param->order_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][5].mark_TID_offset,
                sizeof(UINT32) * 1 * param->order_size/param->device_cnt ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][5].used_rows, sizeof(UINT32) * param->get_sub_txn_size()*2));

            // CHECK(hipMalloc((void **)&aux_struct_d_h[i][6].min_TID, sizeof(UINT32) * 1 * param->orderline_size/param->
                // device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][6].cnt_TID, sizeof(UINT32) * 1 * param->orderline_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][6].tmp_TID, sizeof(UINT32) * 1 * param->orderline_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][6].mark_TID_offset,
                sizeof(UINT32) * 1 * param->orderline_size/param->device_cnt ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][6].used_rows, sizeof(UINT32) * param->get_sub_txn_size()*2));

            // CHECK(hipMalloc((void **)&aux_struct_d_h[i][7].min_TID, sizeof(UINT32) * 1 * param->stock_size/param->
                // device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][7].cnt_TID, sizeof(UINT32) * 1 * param->stock_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][7].tmp_TID, sizeof(UINT32) * 1 * param->stock_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][7].mark_TID_offset,
                sizeof(UINT32) * 1 * param->stock_size/param->device_cnt ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][7].used_rows, sizeof(UINT32) * param->get_sub_txn_size()*2));

            // CHECK(hipMalloc((void **)&aux_struct_d_h[i][8].min_TID, sizeof(UINT32) * 1 * param->item_size));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][8].cnt_TID, sizeof(UINT32) * 1 * param->item_size));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][8].tmp_TID, sizeof(UINT32) * 1 * param->item_size));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][8].mark_TID_offset,
                sizeof(UINT32) * 1 * param->item_size ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][8].used_rows, sizeof(UINT32) * param->get_sub_txn_size()*2));


            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].mark_TID, sizeof(UINT32) * param->get_sub_txn_size()*2));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].merge_tmp, sizeof(UINT32) * param->get_sub_txn_size()*2));


            CHECK(hipMalloc((void **)&global_txn_exec_h[i][0].select_txn_mark, sizeof(UINT32) * param->
                get_select_batch_size()));
            CHECK(hipMalloc((void **)&global_txn_exec_h[i][0].insert_txn_mark, sizeof(UINT32) * param->
                get_insert_batch_size()));
            CHECK(hipMalloc((void **)&global_txn_exec_h[i][0].update_txn_mark, sizeof(UINT32) * param->
                get_update_batch_size()));
            CHECK(hipMalloc((void **)&global_txn_exec_h[i][0].scan_txn_mark, sizeof(UINT32) * param->
                get_scan_batch_size()));
            CHECK(hipMalloc((void **)&global_txn_exec_h[i][0].delete_txn_mark, sizeof(UINT32) * param->
                get_delete_batch_size()));
        }
    } else if (param->benchmark == "TPCC_ALL") {
        gen_param<TPCC_ALL>(param);
        for (size_t i = 0; i < param->device_cnt; ++i) {
            CHECK(hipSetDevice(param->device_IDs[i]));

            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap,
                sizeof(UINT32) * param->get_bitmap_size() * param-> warehouse_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap_all_row,
                sizeof(UINT32) * param->get_sub_txn_size() ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap_tmp,
                sizeof(UINT32) * param->get_bitmap_size() * param-> warehouse_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap_mark,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param-> warehouse_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap_mark_offset,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param-> warehouse_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap_mark_compressed,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param-> warehouse_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap_used_size,
                sizeof(UINT32) * 1 * param->warehouse_size /param-> device_cnt));
            // CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].min_TID, sizeof(UINT32) * 1 * param->warehouse_size/param->
            //     device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].cnt_TID, sizeof(UINT32) * 1 * param->warehouse_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].tmp_TID, sizeof(UINT32) * 1 * param->warehouse_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].mark_TID_offset,
                sizeof(UINT32) * 1 * param->warehouse_size/param->device_cnt ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].used_rows, sizeof(UINT32) * param->get_sub_txn_size()*2));


            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].bitmap,
                sizeof(UINT32) * param->get_bitmap_size() * param-> district_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].bitmap_all_row,
                sizeof(UINT32) * param->get_sub_txn_size() ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].bitmap_tmp,
                sizeof(UINT32) * param->get_bitmap_size() * param-> district_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].bitmap_mark,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param-> district_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].bitmap_mark_offset,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param-> district_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].bitmap_mark_compressed,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param-> district_size/param->device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].bitmap_used_size,
                sizeof(UINT32) * 1 * param->district_size/ param-> device_cnt));
            // CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].min_TID,
                // sizeof(UINT32) * 1 * param->district_size/param-> device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].cnt_TID,
                sizeof(UINT32) * 1 * param->district_size/param-> device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].tmp_TID,
                sizeof(UINT32) * 1 * param->district_size/param-> device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].mark_TID_offset,
                sizeof(UINT32) * 1 * param->district_size/param->device_cnt ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][1].used_rows,
                sizeof(UINT32) * param->get_sub_txn_size()*2));


            // CHECK(hipMalloc((void **)&aux_struct_d_h[i][2].min_TID, sizeof(UINT32) * 1 * param->customer_size/param->
            //     device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][2].cnt_TID, sizeof(UINT32) * 1 * param->customer_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][2].tmp_TID, sizeof(UINT32) * 1 * param->customer_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][2].mark_TID_offset,
                sizeof(UINT32) * 1 * param->customer_size/param->device_cnt ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][2].used_rows, sizeof(UINT32) * param->get_sub_txn_size()*2));

            // CHECK(hipMalloc((void **)&aux_struct_d_h[i][3].min_TID, sizeof(UINT32) * 1 * param->neworder_size/param->
            //     device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][3].cnt_TID, sizeof(UINT32) * 1 * param->neworder_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][3].tmp_TID, sizeof(UINT32) * 1 * param->neworder_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][3].mark_TID_offset,
                sizeof(UINT32) * 1 * param->neworder_size/param->device_cnt ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][3].used_rows, sizeof(UINT32) * param->get_sub_txn_size()*2));

            // CHECK(hipMalloc((void **)&aux_struct_d_h[i][4].min_TID, sizeof(UINT32) * 1 * param->history_size/param->
            //     device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][4].cnt_TID, sizeof(UINT32) * 1 * param->history_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][4].tmp_TID, sizeof(UINT32) * 1 * param->history_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][4].mark_TID_offset,
                sizeof(UINT32) * 1 * param->history_size/param->device_cnt ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][4].used_rows, sizeof(UINT32) * param->get_sub_txn_size()*2));

            // CHECK(hipMalloc((void **)&aux_struct_d_h[i][5].min_TID, sizeof(UINT32) * 1 * param->order_size/param->
            //     device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][5].cnt_TID, sizeof(UINT32) * 1 * param->order_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][5].tmp_TID, sizeof(UINT32) * 1 * param->order_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][5].mark_TID_offset,
                sizeof(UINT32) * 1 * param->order_size/param->device_cnt ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][5].used_rows, sizeof(UINT32) * param->get_sub_txn_size()*2));

            // CHECK(hipMalloc((void **)&aux_struct_d_h[i][6].min_TID, sizeof(UINT32) * 1 * param->orderline_size/param->
            //     device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][6].cnt_TID, sizeof(UINT32) * 1 * param->orderline_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][6].tmp_TID, sizeof(UINT32) * 1 * param->orderline_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][6].mark_TID_offset,
                sizeof(UINT32) * 1 * param->orderline_size/param->device_cnt ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][6].used_rows, sizeof(UINT32) * param->get_sub_txn_size()*2));

            // CHECK(hipMalloc((void **)&aux_struct_d_h[i][7].min_TID, sizeof(UINT32) * 1 * param->stock_size/param->
            //     device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][7].cnt_TID, sizeof(UINT32) * 1 * param->stock_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][7].tmp_TID, sizeof(UINT32) * 1 * param->stock_size/param->
                device_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][7].mark_TID_offset,
                sizeof(UINT32) * 1 * param->stock_size/param->device_cnt ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][7].used_rows, sizeof(UINT32) * param->get_sub_txn_size()*2));

            // CHECK(hipMalloc((void **)&aux_struct_d_h[i][8].min_TID, sizeof(UINT32) * 1 * param->item_size));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][8].cnt_TID, sizeof(UINT32) * 1 * param->item_size));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][8].tmp_TID, sizeof(UINT32) * 1 * param->item_size));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][8].mark_TID_offset,
                sizeof(UINT32) * 1 * param->item_size ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][8].used_rows, sizeof(UINT32) * param->get_sub_txn_size()*2));


            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].mark_TID, sizeof(UINT32) * param->get_sub_txn_size()*2));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].merge_tmp, sizeof(UINT32) * param->get_sub_txn_size()*2));


            CHECK(hipMalloc((void **)&global_txn_exec_h[i][0].select_txn_mark, sizeof(UINT32) * param->
                get_select_batch_size()));
            CHECK(hipMalloc((void **)&global_txn_exec_h[i][0].insert_txn_mark, sizeof(UINT32) * param->
                get_insert_batch_size()));
            CHECK(hipMalloc((void **)&global_txn_exec_h[i][0].update_txn_mark, sizeof(UINT32) * param->
                get_update_batch_size()));
            CHECK(hipMalloc((void **)&global_txn_exec_h[i][0].scan_txn_mark, sizeof(UINT32) * param->
                get_scan_batch_size()));
            CHECK(hipMalloc((void **)&global_txn_exec_h[i][0].delete_txn_mark, sizeof(UINT32) * param->
                get_delete_batch_size()));

#ifdef LTPMG_GPUQUERY_PRINTSIZE
                    std::cout << "global_txn_exec_h[" << i << "][0].select_txn is " << param->get_select_batch_size() << std::endl;
                    std::cout << "global_txn_exec_h[" << i << "][0].insert_txn is " << param->get_insert_batch_size() << std::endl;
                    std::cout << "global_txn_exec_h[" << i << "][0].update_txn is " << param->get_update_batch_size() << std::endl;
                    std::cout << "global_txn_exec_h[" << i << "][0].scan_txn is " << param->get_scan_batch_size() << std::endl;
                    std::cout << "global_txn_exec_h[" << i << "][0].delete_txn is " << param->get_delete_batch_size() << std::endl;
#endif
        }
    } else if (param->benchmark == "YCSB_A" ||
               param->benchmark == "YCSB_B" ||
               param->benchmark == "YCSB_C" ||
               param->benchmark == "YCSB_D" ||
               param->benchmark == "YCSB_E") {
        if (param->benchmark == "YCSB_A") {
            gen_param<YCSB_A_Query>(param);
        } else if (param->benchmark == "YCSB_B") {
            gen_param<YCSB_B_Query>(param);
        } else if (param->benchmark == "YCSB_C") {
            gen_param<YCSB_C_Query>(param);
        } else if (param->benchmark == "YCSB_D") {
            gen_param<YCSB_D_Query>(param);
        } else if (param->benchmark == "YCSB_E") {
            gen_param<YCSB_E_Query>(param);
        }

        for (size_t i = 0; i < param->device_cnt; ++i) {
            CHECK(hipSetDevice(param->device_IDs[i]));

            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap,
                sizeof(UINT32) * param->get_bitmap_size() * param-> bitmap_row_cnt));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap_all_row,
                sizeof(UINT32) * param->get_sub_txn_size() ));
            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap_tmp,
                sizeof(UINT32) * param->get_bitmap_size() * param-> bitmap_row_cnt));

            CHECK(hipMalloc((void**)&aux_struct_d_h[i][0].bitmap_mark,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param-> bitmap_row_cnt));
            CHECK(hipMalloc((void**)&aux_struct_d_h[i][0].bitmap_mark_offset,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param-> bitmap_row_cnt));
            CHECK(hipMalloc((void**)&aux_struct_d_h[i][0].bitmap_mark_compressed,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param-> bitmap_row_cnt));

            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].bitmap_used_size,
                sizeof(UINT32) * 1 * param->bitmap_row_cnt));

            // CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].min_TID, sizeof(UINT32) * 1 * param->ycsb_size/param->
            //     device_cnt));

            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].cnt_TID, sizeof(UINT32) * 1 * param->ycsb_size/param->
                device_cnt));

            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].tmp_TID, sizeof(UINT32) * 1 * param->ycsb_size/param->
                device_cnt));

            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].mark_TID_offset, sizeof(UINT32) * 1 * param->ycsb_size/param
                ->device_cnt));

            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].mark_TID, sizeof(UINT32) * param->get_sub_txn_size()*2));

            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].merge_tmp, sizeof(UINT32) * param->get_sub_txn_size()*2));

            CHECK(hipMalloc((void **)&aux_struct_d_h[i][0].used_rows, sizeof(UINT32) * param->get_sub_txn_size()*2));


            CHECK(hipMalloc((void **)&global_txn_exec_h[i][0].select_txn_mark, sizeof(UINT32) * param->
                get_select_batch_size()));
            CHECK(hipMalloc((void **)&global_txn_exec_h[i][0].insert_txn_mark, sizeof(UINT32) * param->
                get_insert_batch_size()));
            CHECK(hipMalloc((void **)&global_txn_exec_h[i][0].update_txn_mark, sizeof(UINT32) * param->
                get_update_batch_size()));
            CHECK(hipMalloc((void **)&global_txn_exec_h[i][0].scan_txn_mark, sizeof(UINT32) * param->
                get_scan_batch_size()));
            CHECK(hipMalloc((void **)&global_txn_exec_h[i][0].delete_txn_mark, sizeof(UINT32) * param->
                get_delete_batch_size()));
        }
    }

    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipSetDevice(param->device_IDs[i]));
        CHECK(hipMemcpy(global_txn_info_d[i], global_txn_info_h, sizeof(Global_Txn_Info) * param->get_subtxn_kinds(),
            hipMemcpyHostToDevice));
        CHECK(hipMemcpy(exec_param_d[i], exec_param_h, sizeof(Global_Txn_Exec_Param) * param->get_subtxn_kinds(),
            hipMemcpyHostToDevice));
        CHECK(hipMemcpy(aux_struct_d[i], aux_struct_d_h[i], sizeof(Global_Txn_Aux_Struct) * param->table_cnt,
            hipMemcpyHostToDevice));
        CHECK(hipMemcpy(global_txn_exec_d[i], global_txn_exec_h[i], sizeof(Global_Txn_Exec),
            hipMemcpyHostToDevice));
    }
    std::cout << "end gpuquery.cu GPUquery::malloc_global_txn()" << std::endl;
}

void GPUquery::copy_global_txn(std::shared_ptr<Param> param,
                               std::shared_ptr<std::vector<std::any> > transactions_batch_ptr,
                               Global_Txn_Info *global_txn_info, Global_Table_Meta **meta, Global_Table_Index **index) {
    std::cout << "start gpuquery.cu GPUquery::copy_global_txn()" << std::endl;
    this->transactions_batch_ptr = transactions_batch_ptr;
    this->global_txn_info = global_txn_info;
    clear_global_txn(param, transactions_batch_ptr, global_txn_info);

    // 解析事务
    if (param->benchmark == "TEST") {
        query_parse<Test_Query>(param, transactions_batch_ptr, meta, index);
        query_parse<Test_Query_2>(param, transactions_batch_ptr, meta, index);
    } else if (param->benchmark == "TPCC_PART") {
        query_parse<Neworder_Query>(param, transactions_batch_ptr, meta, index);
        query_parse<Payment_Query>(param, transactions_batch_ptr, meta, index);
    } else if (param->benchmark == "TPCC_ALL") {
        query_parse<Neworder_Query>(param, transactions_batch_ptr, meta, index);
        query_parse<Payment_Query>(param, transactions_batch_ptr, meta, index);
        query_parse<Orderstatus_Query>(param, transactions_batch_ptr, meta, index);
        query_parse<Delivery_Query>(param, transactions_batch_ptr, meta, index);
        query_parse<Stocklevel_Query>(param, transactions_batch_ptr, meta, index);
    } else if (param->benchmark == "YCSB_A") {
        query_parse<YCSB_A_Query>(param, transactions_batch_ptr, meta, index);
    } else if (param->benchmark == "YCSB_B") {
        query_parse<YCSB_B_Query>(param, transactions_batch_ptr, meta, index);
    } else if (param->benchmark == "YCSB_C") {
        query_parse<YCSB_C_Query>(param, transactions_batch_ptr, meta, index);
    } else if (param->benchmark == "YCSB_D") {
        query_parse<YCSB_D_Query>(param, transactions_batch_ptr, meta, index);
    } else if (param->benchmark == "YCSB_E") {
        query_parse<YCSB_E_Query>(param, transactions_batch_ptr, meta, index);
    }

    std::cout << "end gpuquery.cu GPUquery::copy_global_txn()" << std::endl;
}

void GPUquery::clear_global_txn(std::shared_ptr<Param> param,
                                std::shared_ptr<std::vector<std::any> > transactions_batch_ptr,
                                Global_Txn_Info *global_txn_info) {
    std::cout << "start gpuquery.cu GPUquery::clear_global_txn()" << std::endl;

    hipStream_t *streams;
    streams = new hipStream_t[param->device_cnt];

    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipSetDevice(param->device_IDs[i]));
        CHECK(hipStreamCreate(&streams[i]));
    }

    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipSetDevice(param->device_IDs[i]));
        CHECK(hipMemsetAsync(global_txn_d[i],0,
            sizeof(Global_Txn)*param->get_sub_txn_size(),streams[i]));
    }

    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipSetDevice(param->device_IDs[i]));

        CHECK(hipMemsetAsync(global_txn_exec_h[i][0].select_txn_mark, 0, sizeof(UINT32) * param->
            get_select_batch_size(), streams[i]));
        CHECK(hipMemsetAsync(global_txn_exec_h[i][0].insert_txn_mark, 0, sizeof(UINT32) * param->
            get_insert_batch_size(), streams[i]));
        CHECK(hipMemsetAsync(global_txn_exec_h[i][0].update_txn_mark, 0, sizeof(UINT32) * param->
            get_update_batch_size(), streams[i]));
        CHECK(hipMemsetAsync(global_txn_exec_h[i][0].scan_txn_mark, 0, sizeof(UINT32) * param->
            get_scan_batch_size(), streams[i]));
        CHECK(hipMemsetAsync(global_txn_exec_h[i][0].delete_txn_mark, 0, sizeof(UINT32) * param->
            get_delete_batch_size(), streams[i]));
    }

    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipSetDevice(param->device_IDs[i]));
        CHECK(hipMemsetAsync(&global_txn_exec_d[i][0].select_cur, 0, sizeof(UINT32), streams[i]));
        CHECK(hipMemsetAsync(&global_txn_exec_d[i][0].insert_cur, 0, sizeof(UINT32), streams[i]));
        CHECK(hipMemsetAsync(&global_txn_exec_d[i][0].update_cur, 0, sizeof(UINT32), streams[i]));
        CHECK(hipMemsetAsync(&global_txn_exec_d[i][0].scan_cur, 0, sizeof(UINT32), streams[i]));
        CHECK(hipMemsetAsync(&global_txn_exec_d[i][0].delete_cur, 0, sizeof(UINT32), streams[i]));

        CHECK(hipMemsetAsync(&global_txn_exec_d[i][0].select_tmp, 0, sizeof(UINT32), streams[i]));
        CHECK(hipMemsetAsync(&global_txn_exec_d[i][0].insert_tmp, 0, sizeof(UINT32), streams[i]));
        CHECK(hipMemsetAsync(&global_txn_exec_d[i][0].update_tmp, 0, sizeof(UINT32), streams[i]));
        CHECK(hipMemsetAsync(&global_txn_exec_d[i][0].scan_tmp, 0, sizeof(UINT32), streams[i]));
        CHECK(hipMemsetAsync(&global_txn_exec_d[i][0].delete_tmp, 0, sizeof(UINT32), streams[i]));

        CHECK(hipMemsetAsync(data_packet_d[i], 0xfffffff,
            sizeof(Global_Data_Packet) * param->get_datapacket_size() , streams[i]));
    }

    if (param->benchmark == "TEST") {
        for (size_t i = 0; i < param->device_cnt; ++i) {
            CHECK(hipSetDevice(param->device_IDs[i]));

            CHECK(hipMemsetAsync(&aux_struct_d[i][0].data_packet_cur, 0,
                sizeof(UINT32), streams[i]));

            CHECK(hipMemsetAsync(&aux_struct_d[i][0].used_rows_cnt, 0,
                sizeof(UINT32), streams[i]));
            CHECK(hipMemsetAsync(&aux_struct_d[i][1].used_rows_cnt, 0,
                sizeof(UINT32), streams[i]));

            CHECK(hipMemsetAsync(&aux_struct_d[i][0].bitmap_size,param->get_bitmap_size() * param-> test_1_size/param
                ->device_cnt, sizeof(UINT32), streams[i]));
            CHECK(hipMemsetAsync(&aux_struct_d[i][1].bitmap_size,param->get_bitmap_size() * param-> test_2_size/param
                ->device_cnt, sizeof(UINT32), streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_used_size,0,
                sizeof(UINT32)* param-> test_1_size/param ->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].bitmap_used_size,0,
                sizeof(UINT32)* param-> test_2_size/param ->device_cnt, streams[i]));

            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap, 0,
                sizeof(UINT32) * param->get_bitmap_size() * param-> test_1_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].bitmap, 0,
                sizeof(UINT32) * param->get_bitmap_size() * param-> test_2_size/param->device_cnt, streams[i]));

            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_all_row, 0,
                sizeof(UINT32) * param->get_sub_txn_size(), streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].bitmap_all_row, 0,
                sizeof(UINT32) * param->get_sub_txn_size(), streams[i]));

            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_tmp, 0,
                sizeof(UINT32) * param->get_bitmap_size() * param-> test_1_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].bitmap_tmp, 0,
                sizeof(UINT32) * param->get_bitmap_size() * param-> test_2_size/param->device_cnt, streams[i]));

            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_mark, 0,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param -> test_1_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].bitmap_mark, 0,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param -> test_2_size/param->device_cnt, streams[i]));

            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_mark_offset, 0,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param -> test_1_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].bitmap_mark_offset, 0,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param -> test_2_size/param->device_cnt, streams[i]));

            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_mark_compressed, 0,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param -> test_1_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].bitmap_mark_compressed, 0,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param -> test_2_size/param->device_cnt, streams[i]));

            // CHECK(hipMemsetAsync(aux_struct_d_h[i][0].min_TID, 0xff,
            //     sizeof(UINT32) * 1 * param->test_1_size/param->device_cnt, streams[i]));
            // CHECK(hipMemsetAsync(aux_struct_d_h[i][1].min_TID, 0xff,
            //     sizeof(UINT32) * 1 * param->test_2_size/param->device_cnt, streams[i]));

            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].cnt_TID, 0,
                sizeof(UINT32) * 1 * param->test_1_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].cnt_TID, 0,
                sizeof(UINT32) * 1 * param->test_2_size/param->device_cnt, streams[i]));

            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].tmp_TID, 0,
                sizeof(UINT32) * 1 * param->test_1_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].tmp_TID, 0,
                sizeof(UINT32) * 1 * param->test_2_size/param->device_cnt, streams[i]));

            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].mark_TID_offset, 0,
                sizeof(UINT32) * 1 * param->test_1_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].mark_TID_offset, 0,
                sizeof(UINT32) * 1 * param->test_2_size/param->device_cnt, streams[i]));

            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].mark_TID, 0xff,
                sizeof(UINT32) * param->get_sub_txn_size()*2, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].merge_tmp, 0xff,
                sizeof(UINT32) * param->get_sub_txn_size()*2, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].used_rows, 0,
                sizeof(UINT32) * param->get_sub_txn_size()*2, streams[i]));

            CHECK(hipMemsetAsync(&aux_struct_d[i][0].mark_TID_start_offset, 0,
                sizeof(UINT32) , streams[i]));
            CHECK(hipMemsetAsync(&aux_struct_d[i][1].mark_TID_start_offset, 0,
                sizeof(UINT32) , streams[i]));
        }
    } else if (param->benchmark == "TPCC_PART") {
        for (size_t i = 0; i < param->device_cnt; ++i) {
            CHECK(hipSetDevice(param->device_IDs[i]));


            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_used_size,0,
                sizeof(UINT32)* param-> warehouse_size/ param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap, 0,
                sizeof(UINT32) * param->get_bitmap_size() * param-> warehouse_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_all_row, 0,
                sizeof(UINT32) * param->get_sub_txn_size(), streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_tmp, 0,
                sizeof(UINT32) * param->get_bitmap_size() * param-> warehouse_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_mark, 0,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param -> warehouse_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_mark_offset, 0,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param -> warehouse_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_mark_compressed, 0,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param -> warehouse_size/param->device_cnt, streams[i]));

            // CHECK(hipMemsetAsync(aux_struct_d_h[i][0].min_TID, 0xff,
            //     sizeof(UINT32) * 1 * param->warehouse_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].cnt_TID, 0,
                sizeof(UINT32) * 1 * param->warehouse_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].tmp_TID, 0,
                sizeof(UINT32) * 1 * param->warehouse_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].mark_TID_offset, 0,
                sizeof(UINT32) * 1 * param->warehouse_size/param->device_cnt, streams[i]));


            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].bitmap_used_size,0,
                sizeof(UINT32)* param-> district_size/ param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].bitmap, 0,
                sizeof(UINT32) * param->get_bitmap_size() * param-> district_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].bitmap_all_row, 0,
                sizeof(UINT32) * param->get_sub_txn_size(), streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].bitmap_tmp, 0,
                sizeof(UINT32) * param->get_bitmap_size() * param-> district_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].bitmap_mark, 0,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param -> district_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].bitmap_mark_offset, 0,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param -> district_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].bitmap_mark_compressed, 0,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param -> district_size/param->device_cnt, streams[i]));
            // CHECK(hipMemsetAsync(aux_struct_d_h[i][1].min_TID, 0xff,
            //     sizeof(UINT32) * 1 * param->district_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].cnt_TID, 0,
                sizeof(UINT32) * 1 * param->district_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].tmp_TID, 0,
                sizeof(UINT32) * 1 * param->district_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].mark_TID_offset, 0,
                sizeof(UINT32) * 1 * param->district_size/param->device_cnt, streams[i]));


            // CHECK(hipMemsetAsync(aux_struct_d_h[i][2].min_TID, 0xff,
            //     sizeof(UINT32) * 1 * param->customer_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][2].cnt_TID, 0,
                sizeof(UINT32) * 1 * param->customer_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][2].tmp_TID, 0,
                sizeof(UINT32) * 1 * param->customer_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][2].mark_TID_offset, 0,
                sizeof(UINT32) * 1 * param->customer_size/param->device_cnt, streams[i]));


            // CHECK(hipMemsetAsync(aux_struct_d_h[i][3].min_TID, 0xff,
            //     sizeof(UINT32) * 1 * param->neworder_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][3].cnt_TID, 0,
                sizeof(UINT32) * 1 * param->neworder_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][3].tmp_TID, 0,
                sizeof(UINT32) * 1 * param->neworder_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][3].mark_TID_offset, 0,
                sizeof(UINT32) * 1 * param->neworder_size/param->device_cnt, streams[i]));


            // CHECK(hipMemsetAsync(aux_struct_d_h[i][4].min_TID, 0xff,
            //     sizeof(UINT32) * 1 * param->history_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][4].cnt_TID, 0,
                sizeof(UINT32) * 1 * param->history_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][4].tmp_TID, 0,
                sizeof(UINT32) * 1 * param->history_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][4].mark_TID_offset, 0,
                sizeof(UINT32) * 1 * param->history_size/param->device_cnt, streams[i]));


            // CHECK(hipMemsetAsync(aux_struct_d_h[i][5].min_TID, 0xff,
            //     sizeof(UINT32) * 1 * param->order_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][5].cnt_TID, 0,
                sizeof(UINT32) * 1 * param->order_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][5].tmp_TID, 0,
                sizeof(UINT32) * 1 * param->order_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][5].mark_TID_offset, 0,
                sizeof(UINT32) * 1 * param->order_size/param->device_cnt, streams[i]));


            // CHECK(hipMemsetAsync(aux_struct_d_h[i][6].min_TID, 0xff,
            //     sizeof(UINT32) * 1 * param->orderline_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][6].cnt_TID, 0,
                sizeof(UINT32) * 1 * param->orderline_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][6].tmp_TID, 0,
                sizeof(UINT32) * 1 * param->orderline_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][6].mark_TID_offset, 0,
                sizeof(UINT32) * 1 * param->orderline_size/param->device_cnt, streams[i]));


            // CHECK(hipMemsetAsync(aux_struct_d_h[i][7].min_TID, 0xff,
            //     sizeof(UINT32) * 1 * param->stock_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][7].cnt_TID, 0,
                sizeof(UINT32) * 1 * param->stock_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][7].tmp_TID, 0,
                sizeof(UINT32) * 1 * param->stock_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][7].mark_TID_offset, 0,
                sizeof(UINT32) * 1 * param->stock_size/param->device_cnt, streams[i]));


            // CHECK(hipMemsetAsync(aux_struct_d_h[i][8].min_TID, 0xff,
            //     sizeof(UINT32) * 1 * param->item_size, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][8].cnt_TID, 0,
                sizeof(UINT32) * 1 * param->item_size, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][8].tmp_TID, 0,
                sizeof(UINT32) * 1 * param->item_size, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][8].mark_TID_offset, 0,
                sizeof(UINT32) * 1 * param->item_size, streams[i]));

            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].mark_TID, 0xff,
                sizeof(UINT32) * param->get_sub_txn_size()*2, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].merge_tmp, 0xff,
                sizeof(UINT32) * param->get_sub_txn_size()*2, streams[i]));

            for (uint32_t j = 0; j < param->table_cnt; ++j) {
                CHECK(hipMemsetAsync(aux_struct_d_h[i][j].used_rows, 0,
                    sizeof(UINT32) * param->get_sub_txn_size()*2, streams[i]));

                CHECK(hipMemsetAsync(&aux_struct_d[i][j].mark_TID_start_offset, 0,
                    sizeof(UINT32) , streams[i]));

                CHECK(hipMemsetAsync(&aux_struct_d[i][j].used_rows_cnt, 0,
                    sizeof(UINT32) , streams[i]));
            }

            CHECK(hipMemsetAsync(&aux_struct_d[i][0].data_packet_cur, 0,
                sizeof(UINT32) , streams[i]));


            CHECK(hipMemsetAsync(&aux_struct_d[i][0].bitmap_size,param->get_bitmap_size() * param-> warehouse_size/
                param->device_cnt, sizeof(UINT32), streams[i]));
            CHECK(hipMemsetAsync(&aux_struct_d[i][1].bitmap_size,param->get_bitmap_size() * param-> district_size/
                param->device_cnt, sizeof(UINT32), streams[i]));
            CHECK(hipMemsetAsync(&aux_struct_d[i][2].bitmap_size,param->get_bitmap_size() * param-> customer_size/
                param->device_cnt, sizeof(UINT32), streams[i]));
            CHECK(hipMemsetAsync(&aux_struct_d[i][3].bitmap_size,param->get_bitmap_size() * param-> neworder_size/
                param->device_cnt, sizeof(UINT32), streams[i]));
            CHECK(hipMemsetAsync(&aux_struct_d[i][4].bitmap_size,param->get_bitmap_size() * param-> history_size/
                param->device_cnt, sizeof(UINT32), streams[i]));
            CHECK(hipMemsetAsync(&aux_struct_d[i][5].bitmap_size,param->get_bitmap_size() * param-> order_size/
                param->device_cnt, sizeof(UINT32), streams[i]));
            CHECK(hipMemsetAsync(&aux_struct_d[i][6].bitmap_size,param->get_bitmap_size() * param-> orderline_size/
                param->device_cnt, sizeof(UINT32), streams[i]));
            CHECK(hipMemsetAsync(&aux_struct_d[i][7].bitmap_size,param->get_bitmap_size() * param-> stock_size/
                param->device_cnt, sizeof(UINT32), streams[i]));
            CHECK(hipMemsetAsync(&aux_struct_d[i][8].bitmap_size,param->get_bitmap_size() * param-> item_size,
                sizeof(UINT32), streams[i]));
        }
    } else if (param->benchmark == "TPCC_ALL") {
        for (size_t i = 0; i < param->device_cnt; ++i) {
            CHECK(hipSetDevice(param->device_IDs[i]));


            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_used_size,0,
                sizeof(UINT32)* param-> warehouse_size/ param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap, 0,
                sizeof(UINT32) * param->get_bitmap_size() * param-> warehouse_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_all_row, 0,
                sizeof(UINT32) * param->get_sub_txn_size(), streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_tmp, 0,
                sizeof(UINT32) * param->get_bitmap_size() * param-> warehouse_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_mark, 0,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param -> warehouse_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_mark_offset, 0,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param -> warehouse_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_mark_compressed, 0,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param -> warehouse_size/param->device_cnt, streams[i]));

            // CHECK(hipMemsetAsync(aux_struct_d_h[i][0].min_TID, 0xff,
            //     sizeof(UINT32) * 1 * param->warehouse_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].cnt_TID, 0,
                sizeof(UINT32) * 1 * param->warehouse_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].tmp_TID, 0,
                sizeof(UINT32) * 1 * param->warehouse_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].mark_TID_offset, 0,
                sizeof(UINT32) * 1 * param->warehouse_size/param->device_cnt, streams[i]));


            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].bitmap_used_size,0,
                sizeof(UINT32)* param-> district_size/ param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].bitmap, 0,
                sizeof(UINT32) * param->get_bitmap_size() * param-> district_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].bitmap_all_row, 0,
                sizeof(UINT32) * param->get_sub_txn_size(), streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].bitmap_tmp, 0,
                sizeof(UINT32) * param->get_bitmap_size() * param-> district_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].bitmap_mark, 0,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param -> district_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].bitmap_mark_offset, 0,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param -> district_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].bitmap_mark_compressed, 0,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param -> district_size/param->device_cnt, streams[i]));
            // CHECK(hipMemsetAsync(aux_struct_d_h[i][1].min_TID, 0xff,
            //     sizeof(UINT32) * 1 * param->district_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].cnt_TID, 0,
                sizeof(UINT32) * 1 * param->district_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].tmp_TID, 0,
                sizeof(UINT32) * 1 * param->district_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][1].mark_TID_offset, 0,
                sizeof(UINT32) * 1 * param->district_size/param->device_cnt, streams[i]));


            // CHECK(hipMemsetAsync(aux_struct_d_h[i][2].min_TID, 0xff,
            //     sizeof(UINT32) * 1 * param->customer_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][2].cnt_TID, 0,
                sizeof(UINT32) * 1 * param->customer_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][2].tmp_TID, 0,
                sizeof(UINT32) * 1 * param->customer_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][2].mark_TID_offset, 0,
                sizeof(UINT32) * 1 * param->customer_size/param->device_cnt, streams[i]));


            // CHECK(hipMemsetAsync(aux_struct_d_h[i][3].min_TID, 0xff,
            //     sizeof(UINT32) * 1 * param->neworder_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][3].cnt_TID, 0,
                sizeof(UINT32) * 1 * param->neworder_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][3].tmp_TID, 0,
                sizeof(UINT32) * 1 * param->neworder_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][3].mark_TID_offset, 0,
                sizeof(UINT32) * 1 * param->neworder_size/param->device_cnt, streams[i]));


            // CHECK(hipMemsetAsync(aux_struct_d_h[i][4].min_TID, 0xff,
            //     sizeof(UINT32) * 1 * param->history_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][4].cnt_TID, 0,
                sizeof(UINT32) * 1 * param->history_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][4].tmp_TID, 0,
                sizeof(UINT32) * 1 * param->history_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][4].mark_TID_offset, 0,
                sizeof(UINT32) * 1 * param->history_size/param->device_cnt, streams[i]));


            // CHECK(hipMemsetAsync(aux_struct_d_h[i][5].min_TID, 0xff,
            //     sizeof(UINT32) * 1 * param->order_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][5].cnt_TID, 0,
                sizeof(UINT32) * 1 * param->order_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][5].tmp_TID, 0,
                sizeof(UINT32) * 1 * param->order_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][5].mark_TID_offset, 0,
                sizeof(UINT32) * 1 * param->order_size/param->device_cnt, streams[i]));


            // CHECK(hipMemsetAsync(aux_struct_d_h[i][6].min_TID, 0xff,
            //     sizeof(UINT32) * 1 * param->orderline_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][6].cnt_TID, 0,
                sizeof(UINT32) * 1 * param->orderline_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][6].tmp_TID, 0,
                sizeof(UINT32) * 1 * param->orderline_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][6].mark_TID_offset, 0,
                sizeof(UINT32) * 1 * param->orderline_size/param->device_cnt, streams[i]));


            // CHECK(hipMemsetAsync(aux_struct_d_h[i][7].min_TID, 0xff,
            //     sizeof(UINT32) * 1 * param->stock_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][7].cnt_TID, 0,
                sizeof(UINT32) * 1 * param->stock_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][7].tmp_TID, 0,
                sizeof(UINT32) * 1 * param->stock_size/param->device_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][7].mark_TID_offset, 0,
                sizeof(UINT32) * 1 * param->stock_size/param->device_cnt, streams[i]));


            // CHECK(hipMemsetAsync(aux_struct_d_h[i][8].min_TID, 0xff,
            //     sizeof(UINT32) * 1 * param->item_size, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][8].cnt_TID, 0,
                sizeof(UINT32) * 1 * param->item_size, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][8].tmp_TID, 0,
                sizeof(UINT32) * 1 * param->item_size, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][8].mark_TID_offset, 0,
                sizeof(UINT32) * 1 * param->item_size, streams[i]));

            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].mark_TID, 0xff,
                sizeof(UINT32) * param->get_sub_txn_size()*2, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].merge_tmp, 0xff,
                sizeof(UINT32) * param->get_sub_txn_size()*2, streams[i]));


            for (uint32_t j = 0; j < param->table_cnt; ++j) {
                CHECK(hipMemsetAsync(aux_struct_d_h[i][j].used_rows, 0,
                    sizeof(UINT32) * param->get_sub_txn_size()*2, streams[i]));

                CHECK(hipMemsetAsync(&aux_struct_d[i][j].mark_TID_start_offset, 0,
                    sizeof(UINT32) , streams[i]));

                CHECK(hipMemsetAsync(&aux_struct_d[i][j].used_rows_cnt, 0,
                    sizeof(UINT32) , streams[i]));
            }

            CHECK(hipMemsetAsync(&aux_struct_d[i][0].data_packet_cur, 0,
                sizeof(UINT32) , streams[i]));

            CHECK(hipMemsetAsync(&aux_struct_d[i][0].bitmap_size,param->get_bitmap_size() * param-> warehouse_size/
                param->device_cnt, sizeof(UINT32), streams[i]));
            CHECK(hipMemsetAsync(&aux_struct_d[i][1].bitmap_size,param->get_bitmap_size() * param-> district_size/
                param->device_cnt, sizeof(UINT32), streams[i]));
            CHECK(hipMemsetAsync(&aux_struct_d[i][2].bitmap_size,param->get_bitmap_size() * param-> customer_size/
                param->device_cnt, sizeof(UINT32), streams[i]));
            CHECK(hipMemsetAsync(&aux_struct_d[i][3].bitmap_size,param->get_bitmap_size() * param-> neworder_size/
                param->device_cnt, sizeof(UINT32), streams[i]));
            CHECK(hipMemsetAsync(&aux_struct_d[i][4].bitmap_size,param->get_bitmap_size() * param-> history_size/
                param->device_cnt, sizeof(UINT32), streams[i]));
            CHECK(hipMemsetAsync(&aux_struct_d[i][5].bitmap_size,param->get_bitmap_size() * param-> order_size/
                param->device_cnt, sizeof(UINT32), streams[i]));
            CHECK(hipMemsetAsync(&aux_struct_d[i][6].bitmap_size,param->get_bitmap_size() * param-> orderline_size/
                param->device_cnt, sizeof(UINT32), streams[i]));
            CHECK(hipMemsetAsync(&aux_struct_d[i][7].bitmap_size,param->get_bitmap_size() * param-> stock_size/
                param->device_cnt, sizeof(UINT32), streams[i]));
            CHECK(hipMemsetAsync(&aux_struct_d[i][8].bitmap_size,param->get_bitmap_size() * param-> item_size,
                sizeof(UINT32), streams[i]));
        }
    } else if (param->benchmark == "YCSB_A" ||
               param->benchmark == "YCSB_B" ||
               param->benchmark == "YCSB_C" ||
               param->benchmark == "YCSB_D" ||
               param->benchmark == "YCSB_E") {
        for (size_t i = 0; i < param->device_cnt; ++i) {
            CHECK(hipSetDevice(param->device_IDs[i]));

            CHECK(hipMemsetAsync(&aux_struct_d[i][0].data_packet_cur, 0,
                sizeof(UINT32), streams[i]));

            CHECK(hipMemsetAsync(&aux_struct_d[i][0].used_rows_cnt, 0,
                sizeof(UINT32), streams[i]));

            CHECK(hipMemsetAsync(&aux_struct_d[i][0].bitmap_size,param->get_bitmap_size() *
                param-> bitmap_row_cnt , sizeof(UINT32), streams[i]));

            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_used_size,
                0, sizeof(UINT32)* param-> bitmap_row_cnt, streams[i]));

            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap, 0,
                sizeof(UINT32) * param->get_bitmap_size() * param-> bitmap_row_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_all_row, 0,
                sizeof(UINT32) * param->get_sub_txn_size(), streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_tmp, 0,
                sizeof(UINT32) * param->get_bitmap_size() * param-> bitmap_row_cnt, streams[i]));

            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_mark, 0,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param -> bitmap_row_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_mark_offset, 0,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param -> bitmap_row_cnt, streams[i]));
            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].bitmap_mark_compressed, 0,
                sizeof(UINT32) * param->get_bitmap_size()*32 * param -> bitmap_row_cnt, streams[i]));

            // CHECK(hipMemsetAsync(aux_struct_d_h[i][0].min_TID, 0xff,
            //     sizeof(UINT32) * 1 * param->ycsb_size/param->device_cnt, streams[i]));

            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].cnt_TID, 0,
                sizeof(UINT32) * 1 * param->ycsb_size/param->device_cnt, streams[i]));

            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].tmp_TID, 0,
                sizeof(UINT32) * 1 * param->ycsb_size/param->device_cnt, streams[i]));

            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].mark_TID_offset, 0,
                sizeof(UINT32) * 1 * param->ycsb_size/param->device_cnt, streams[i]));

            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].mark_TID, 0xff,
                sizeof(UINT32) * param->get_sub_txn_size()*2, streams[i]));

            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].merge_tmp, 0xff,
                sizeof(UINT32) * param->get_sub_txn_size()*2, streams[i]));

            CHECK(hipMemsetAsync(aux_struct_d_h[i][0].used_rows, 0,
                sizeof(UINT32) * param->get_sub_txn_size()*2, streams[i]));

            CHECK(hipMemsetAsync(&aux_struct_d[i][0].mark_TID_start_offset, 0,
                sizeof(UINT32) , streams[i]));
        }
    }

    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipStreamSynchronize(streams[i]));
    }
    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipStreamDestroy(streams[i]));
    }
    delete[] streams;

    std::cout << "end gpuquery.cu GPUquery::clear_global_txn()" << std::endl;
}

void GPUquery::free_global_txn(std::shared_ptr<Param> param,
                               std::shared_ptr<std::vector<std::any> > transactions_batch_ptr,
                               Global_Txn_Info *global_txn_info) {
    std::cout << "start gpuquery.cu GPUquery::free_global_txn()" << std::endl;

    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipFree(global_txn_info_d[i]));
    }
    CHECK(hipHostFree(global_txn_info_d));
    CHECK(hipHostFree(global_txn_info_h));

    // for (size_t k = 0; k < param->get_subtxn_kinds(); ++k) {
    //     CHECK(hipHostFree(global_txn[k].subtxn));
    // }
    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipFree(global_txn_d[i]));
        CHECK(hipHostFree(global_txn_h[i]));
    }
    CHECK(hipHostFree(global_txn_d));
    CHECK(hipHostFree(global_txn_h));
    CHECK(hipHostFree(global_txn));

    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipFree(global_txn_exec_h[i][0].select_txn_mark));
        CHECK(hipFree(global_txn_exec_h[i][0].insert_txn_mark));
        CHECK(hipFree(global_txn_exec_h[i][0].update_txn_mark));
        CHECK(hipFree(global_txn_exec_h[i][0].scan_txn_mark));
        CHECK(hipFree(global_txn_exec_h[i][0].delete_txn_mark));

        CHECK(hipFree(global_txn_exec_d[i]));
        CHECK(hipHostFree(global_txn_exec_h[i]));
    }
    CHECK(hipHostFree(global_txn_exec_d));
    CHECK(hipHostFree(global_txn_exec_h));

    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipFree(global_txn_result_d[i]));
        CHECK(hipHostFree(global_txn_result_h[i]));
    }
    CHECK(hipHostFree(global_txn_result_d));
    CHECK(hipHostFree(global_txn_result_h));

    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipFree(exec_param_d[i]));
    }
    CHECK(hipHostFree(exec_param_h));
    CHECK(hipHostFree(exec_param_d));

    for (size_t i = 0; i < param->device_cnt; ++i) {
        if (param->benchmark == "TEST") {
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_all_row));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_tmp));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark_offset));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark_compressed));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_used_size));
            // CHECK(hipFree(aux_struct_d_h[i][0].min_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].cnt_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].tmp_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].mark_TID_offset));
            CHECK(hipFree(aux_struct_d_h[i][0].mark_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].used_rows));

            CHECK(hipFree(aux_struct_d_h[i][1].bitmap));
            CHECK(hipFree(aux_struct_d_h[i][1].bitmap_all_row));
            CHECK(hipFree(aux_struct_d_h[i][1].bitmap_tmp));
            CHECK(hipFree(aux_struct_d_h[i][1].bitmap_mark));
            CHECK(hipFree(aux_struct_d_h[i][1].bitmap_mark_offset));
            CHECK(hipFree(aux_struct_d_h[i][1].bitmap_mark_compressed));
            CHECK(hipFree(aux_struct_d_h[i][1].bitmap_used_size));
            // CHECK(hipFree(aux_struct_d_h[i][1].min_TID));
            CHECK(hipFree(aux_struct_d_h[i][1].cnt_TID));
            CHECK(hipFree(aux_struct_d_h[i][1].tmp_TID));
            CHECK(hipFree(aux_struct_d_h[i][1].mark_TID_offset));
            CHECK(hipFree(aux_struct_d_h[i][1].mark_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].used_rows));
        } else if (param->benchmark == "TPCC_PART") {
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_all_row));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_tmp));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark_offset));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark_compressed));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_used_size));
            // CHECK(hipFree(aux_struct_d_h[i][0].min_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].cnt_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].tmp_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].mark_TID_offset));
            CHECK(hipFree(aux_struct_d_h[i][0].mark_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].used_rows));

            CHECK(hipFree(aux_struct_d_h[i][1].bitmap));
            CHECK(hipFree(aux_struct_d_h[i][1].bitmap_all_row));
            CHECK(hipFree(aux_struct_d_h[i][1].bitmap_tmp));
            CHECK(hipFree(aux_struct_d_h[i][1].bitmap_mark));
            CHECK(hipFree(aux_struct_d_h[i][1].bitmap_mark_offset));
            CHECK(hipFree(aux_struct_d_h[i][1].bitmap_mark_compressed));
            CHECK(hipFree(aux_struct_d_h[i][1].bitmap_used_size));
            // CHECK(hipFree(aux_struct_d_h[i][1].min_TID));
            CHECK(hipFree(aux_struct_d_h[i][1].cnt_TID));
            CHECK(hipFree(aux_struct_d_h[i][1].tmp_TID));
            CHECK(hipFree(aux_struct_d_h[i][1].mark_TID_offset));
            CHECK(hipFree(aux_struct_d_h[i][1].mark_TID));
            CHECK(hipFree(aux_struct_d_h[i][1].used_rows));

            for (uint32_t j = 2; j < param->table_cnt; ++j) {
                // CHECK(hipFree(aux_struct_d_h[i][j].min_TID));
                CHECK(hipFree(aux_struct_d_h[i][j].cnt_TID));
                CHECK(hipFree(aux_struct_d_h[i][j].tmp_TID));
                CHECK(hipFree(aux_struct_d_h[i][j].mark_TID_offset));
                CHECK(hipFree(aux_struct_d_h[i][j].mark_TID));
                CHECK(hipFree(aux_struct_d_h[i][j].used_rows));
            }
        } else if (param->benchmark == "TPCC_ALL") {
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_all_row));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_tmp));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark_offset));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark_compressed));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_used_size));
            // CHECK(hipFree(aux_struct_d_h[i][0].min_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].cnt_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].tmp_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].mark_TID_offset));
            CHECK(hipFree(aux_struct_d_h[i][0].mark_TID));

            CHECK(hipFree(aux_struct_d_h[i][1].bitmap));
            CHECK(hipFree(aux_struct_d_h[i][1].bitmap_all_row));
            CHECK(hipFree(aux_struct_d_h[i][1].bitmap_tmp));
            CHECK(hipFree(aux_struct_d_h[i][1].bitmap_mark));
            CHECK(hipFree(aux_struct_d_h[i][1].bitmap_mark_offset));
            CHECK(hipFree(aux_struct_d_h[i][1].bitmap_mark_compressed));
            CHECK(hipFree(aux_struct_d_h[i][1].bitmap_used_size));
            // CHECK(hipFree(aux_struct_d_h[i][1].min_TID));
            CHECK(hipFree(aux_struct_d_h[i][1].cnt_TID));
            CHECK(hipFree(aux_struct_d_h[i][1].tmp_TID));
            CHECK(hipFree(aux_struct_d_h[i][1].mark_TID_offset));
            CHECK(hipFree(aux_struct_d_h[i][1].mark_TID));

            for (uint32_t j = 2; j < param->table_cnt; ++j) {
                // CHECK(hipFree(aux_struct_d_h[i][j].min_TID));
                CHECK(hipFree(aux_struct_d_h[i][j].cnt_TID));
                CHECK(hipFree(aux_struct_d_h[i][j].tmp_TID));
                CHECK(hipFree(aux_struct_d_h[i][j].mark_TID_offset));
                CHECK(hipFree(aux_struct_d_h[i][j].mark_TID));
                CHECK(hipFree(aux_struct_d_h[i][j].used_rows));
            }
        } else if (param->benchmark == "YCSB_A") {
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_all_row));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_tmp));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark_offset));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark_compressed));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_used_size));
            // CHECK(hipFree(aux_struct_d_h[i][0].min_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].cnt_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].tmp_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].mark_TID_offset));
            CHECK(hipFree(aux_struct_d_h[i][0].mark_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].used_rows));
        } else if (param->benchmark == "YCSB_B") {
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_all_row));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_tmp));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark_offset));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark_compressed));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_used_size));
            // CHECK(hipFree(aux_struct_d_h[i][0].min_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].cnt_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].tmp_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].mark_TID_offset));
            CHECK(hipFree(aux_struct_d_h[i][0].mark_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].used_rows));
        } else if (param->benchmark == "YCSB_C") {
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_all_row));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_tmp));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark_offset));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark_compressed));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_used_size));
            // CHECK(hipFree(aux_struct_d_h[i][0].min_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].cnt_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].tmp_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].mark_TID_offset));
            CHECK(hipFree(aux_struct_d_h[i][0].mark_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].used_rows));
        } else if (param->benchmark == "YCSB_D") {
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_all_row));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_tmp));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark_offset));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark_compressed));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_used_size));
            // CHECK(hipFree(aux_struct_d_h[i][0].min_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].cnt_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].tmp_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].mark_TID_offset));
            CHECK(hipFree(aux_struct_d_h[i][0].mark_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].used_rows));
        } else if (param->benchmark == "YCSB_E") {
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_all_row));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_tmp));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark_offset));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_mark_compressed));
            CHECK(hipFree(aux_struct_d_h[i][0].bitmap_used_size));
            // CHECK(hipFree(aux_struct_d_h[i][0].min_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].cnt_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].tmp_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].mark_TID_offset));
            CHECK(hipFree(aux_struct_d_h[i][0].mark_TID));
            CHECK(hipFree(aux_struct_d_h[i][0].used_rows));
        }
        CHECK(hipFree(aux_struct_d[i]));
        CHECK(hipHostFree(aux_struct_d_h[i]));
    }
    CHECK(hipHostFree(aux_struct_d));
    CHECK(hipHostFree(aux_struct_d_h));

    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipFree(data_packet_d[i]));
        CHECK(hipHostFree(data_packet_h[i]));
    }
    CHECK(hipHostFree(data_packet_d));
    CHECK(hipHostFree(data_packet_h));

    std::cout << "start gpuquery.cu GPUQuery::free_global_txn()" << std::endl;
}

void GPUquery::transfer_data_packet(std::shared_ptr<Param> param, hipStream_t *streams) {
    // std::cout << "start gpuquery.cu GPUQuery::transfer_data_packet()" << std::endl;

    if (param->device_cnt == 2) {
        CHECK(hipMemcpyPeerAsync(data_packet_d[0] + param->get_datapacket_size()/2, param->device_IDs[0],
            data_packet_d[1] , param->device_IDs[1],
            sizeof(Global_Data_Packet) * param->get_datapacket_size()/2, streams[1]));
        // 1->0
        CHECK(hipMemcpyPeerAsync(data_packet_d[1] + param->get_datapacket_size()/2, param->device_IDs[1],
            data_packet_d[0] , param->device_IDs[0],
            sizeof(Global_Data_Packet) * param->get_datapacket_size()/2, streams[0]));
        // 0->1
    }
    if (param->device_cnt == 4) {
#ifndef LTPMG_GPUQUERY_TRANSFER_GROUP
        CHECK(hipMemcpyPeerAsync(data_packet_d[1] + param->get_sub_txn_size()/4, param->device_IDs[1],
            data_packet_d[0], param->device_IDs[0],
            sizeof(Global_Data_Packet) * param->get_sub_txn_size()/4, streams[0]));
        // 0->1
        CHECK(hipMemcpyPeerAsync(data_packet_d[0] + param->get_sub_txn_size()/4, param->device_IDs[0],
            data_packet_d[1], param->device_IDs[1],
            sizeof(Global_Data_Packet) * param->get_sub_txn_size()/4, streams[1]));
        // 1->0
        CHECK(hipMemcpyPeerAsync(data_packet_d[3] + param->get_sub_txn_size()/4, param->device_IDs[3],
            data_packet_d[2], param->device_IDs[2],
            sizeof(Global_Data_Packet) * param->get_sub_txn_size()/4, streams[2]));
        // 2->3
        CHECK(hipMemcpyPeerAsync(data_packet_d[2] + param->get_sub_txn_size()/4, param->device_IDs[2],
            data_packet_d[3], param->device_IDs[3],
            sizeof(Global_Data_Packet) * param->get_sub_txn_size()/4, streams[3]));
        // 3->2

        for (uint32_t i = 0; i < param->device_cnt; ++i) {
            CHECK(hipStreamSynchronize(streams[i]));
        }
#endif

        CHECK(hipMemcpyPeerAsync(data_packet_d[2] + param->get_datapacket_size()/2, param->device_IDs[2],
            data_packet_d[0], param->device_IDs[0],
            sizeof(Global_Data_Packet) * param->get_datapacket_size()/2, streams[0]));
        CHECK(hipMemcpyPeerAsync(data_packet_d[3] + param->get_datapacket_size()/2, param->device_IDs[3],
            data_packet_d[1], param->device_IDs[1],
            sizeof(Global_Data_Packet) * param->get_datapacket_size()/2, streams[1]));
        // 01->23
        CHECK(hipMemcpyPeerAsync(data_packet_d[0] + param->get_datapacket_size()/2, param->device_IDs[0],
            data_packet_d[2], param->device_IDs[2],
            sizeof(Global_Data_Packet) * param->get_datapacket_size()/2, streams[2]));
        CHECK(hipMemcpyPeerAsync(data_packet_d[1] + param->get_datapacket_size()/2, param->device_IDs[1],
            data_packet_d[3], param->device_IDs[3],
            sizeof(Global_Data_Packet) * param->get_datapacket_size()/2, streams[3]));
        // 23->01
    }
    if (param->device_cnt == 8) {
#ifndef LTPMG_GPUQUERY_TRANSFER_GROUP

        CHECK(hipMemcpyPeerAsync(data_packet_d[1] + param->get_sub_txn_size()/8, param->device_IDs[1],
            data_packet_d[0], param->device_IDs[0],
            sizeof(Global_Data_Packet) * param->get_sub_txn_size()/8, streams[0] ));
        // 0->1
        CHECK(hipMemcpyPeerAsync(data_packet_d[0] + param->get_sub_txn_size()/8, param->device_IDs[0],
            data_packet_d[1], param->device_IDs[1],
            sizeof(Global_Data_Packet) * param->get_sub_txn_size()/8, streams[1] ));
        // 1->0
        CHECK(hipMemcpyPeerAsync(data_packet_d[3] + param->get_sub_txn_size()/8, param->device_IDs[3],
            data_packet_d[2], param->device_IDs[2],
            sizeof(Global_Data_Packet) * param->get_sub_txn_size()/8, streams[2] ));
        // 2->3
        CHECK(hipMemcpyPeerAsync(data_packet_d[2] + param->get_sub_txn_size()/8, param->device_IDs[2],
            data_packet_d[3], param->device_IDs[3],
            sizeof(Global_Data_Packet) * param->get_sub_txn_size()/8, streams[3] ));
        // 3->2
        CHECK(hipMemcpyPeerAsync(data_packet_d[5] + param->get_sub_txn_size()/8, param->device_IDs[5],
            data_packet_d[4], param->device_IDs[4],
            sizeof(Global_Data_Packet) * param->get_sub_txn_size()/8, streams[4] ));
        // 4->5
        CHECK(hipMemcpyPeerAsync(data_packet_d[4] + param->get_sub_txn_size()/8, param->device_IDs[4],
            data_packet_d[5], param->device_IDs[5],
            sizeof(Global_Data_Packet) * param->get_sub_txn_size()/8, streams[5] ));
        // 5->4
        CHECK(hipMemcpyPeerAsync(data_packet_d[7] + param->get_sub_txn_size()/8, param->device_IDs[7],
            data_packet_d[6], param->device_IDs[6],
            sizeof(Global_Data_Packet) * param->get_sub_txn_size()/8, streams[6] ));
        // 6->7
        CHECK(hipMemcpyPeerAsync(data_packet_d[6] + param->get_sub_txn_size()/8, param->device_IDs[6],
            data_packet_d[7], param->device_IDs[7],
            sizeof(Global_Data_Packet) * param->get_sub_txn_size()/8, streams[7] ));
        // 7->6

        for (uint32_t i = 0; i < param->device_cnt; ++i) {
            CHECK(hipStreamSynchronize(streams[i]));
        }

        CHECK(hipMemcpyPeerAsync(data_packet_d[2] + param->get_sub_txn_size()/4, param->device_IDs[2],
            data_packet_d[0], param->device_IDs[0],
            sizeof(Global_Data_Packet) * param->get_sub_txn_size()/4, streams[0]));
        CHECK(hipMemcpyPeerAsync(data_packet_d[3] + param->get_sub_txn_size()/4, param->device_IDs[3],
            data_packet_d[1], param->device_IDs[1],
            sizeof(Global_Data_Packet) * param->get_sub_txn_size()/4, streams[1]));
        // 01->23
        CHECK(hipMemcpyPeerAsync(data_packet_d[0] + param->get_sub_txn_size()/4, param->device_IDs[0],
            data_packet_d[2], param->device_IDs[2],
            sizeof(Global_Data_Packet) * param->get_sub_txn_size()/4, streams[2]));
        CHECK(hipMemcpyPeerAsync(data_packet_d[1] + param->get_sub_txn_size()/4, param->device_IDs[1],
            data_packet_d[3], param->device_IDs[3],
            sizeof(Global_Data_Packet) * param->get_sub_txn_size()/4, streams[3]));
        // 23->01
        CHECK(hipMemcpyPeerAsync(data_packet_d[6] + param->get_sub_txn_size()/4, param->device_IDs[6],
            data_packet_d[4], param->device_IDs[4],
            sizeof(Global_Data_Packet) * param->get_sub_txn_size()/4, streams[4]));
        CHECK(hipMemcpyPeerAsync(data_packet_d[7] + param->get_sub_txn_size()/4, param->device_IDs[7],
            data_packet_d[5], param->device_IDs[5],
            sizeof(Global_Data_Packet) * param->get_sub_txn_size()/4, streams[5]));
        // 45->67
        CHECK(hipMemcpyPeerAsync(data_packet_d[4] + param->get_sub_txn_size()/4, param->device_IDs[4],
            data_packet_d[6], param->device_IDs[6],
            sizeof(Global_Data_Packet) * param->get_sub_txn_size()/4, streams[6]));
        CHECK(hipMemcpyPeerAsync(data_packet_d[5] + param->get_sub_txn_size()/4, param->device_IDs[5],
            data_packet_d[7], param->device_IDs[7],
            sizeof(Global_Data_Packet) * param->get_sub_txn_size()/4, streams[7]));
        // 67->45

        for (uint32_t i = 0; i < param->device_cnt; ++i) {
            CHECK(hipStreamSynchronize(streams[i]));
        }
#endif

        CHECK(hipMemcpyPeerAsync(data_packet_d[4] + param->get_datapacket_size()/2, param->device_IDs[4],
            data_packet_d[0], param->device_IDs[0],
            sizeof(Global_Data_Packet) * param->get_datapacket_size()/2, streams[0]));
        CHECK(hipMemcpyPeerAsync(data_packet_d[5] + param->get_datapacket_size()/2, param->device_IDs[5],
            data_packet_d[1], param->device_IDs[1],
            sizeof(Global_Data_Packet) * param->get_datapacket_size()/2, streams[1]));
        CHECK(hipMemcpyPeerAsync(data_packet_d[6] + param->get_datapacket_size()/2, param->device_IDs[6],
            data_packet_d[2], param->device_IDs[2],
            sizeof(Global_Data_Packet) * param->get_datapacket_size()/2, streams[2]));
        CHECK(hipMemcpyPeerAsync(data_packet_d[7] + param->get_datapacket_size()/2, param->device_IDs[7],
            data_packet_d[3], param->device_IDs[3],
            sizeof(Global_Data_Packet) * param->get_datapacket_size()/2, streams[3]));
        // 0123->4567
        CHECK(hipMemcpyPeerAsync(data_packet_d[0] + param->get_datapacket_size()/2, param->device_IDs[0],
            data_packet_d[4], param->device_IDs[4],
            sizeof(Global_Data_Packet) * param->get_datapacket_size()/2, streams[4]));
        CHECK(hipMemcpyPeerAsync(data_packet_d[1] + param->get_datapacket_size()/2, param->device_IDs[1],
            data_packet_d[5], param->device_IDs[5],
            sizeof(Global_Data_Packet) * param->get_datapacket_size()/2, streams[5]));
        CHECK(hipMemcpyPeerAsync(data_packet_d[2] + param->get_datapacket_size()/2, param->device_IDs[2],
            data_packet_d[6], param->device_IDs[6],
            sizeof(Global_Data_Packet) * param->get_datapacket_size()/2, streams[6]));
        CHECK(hipMemcpyPeerAsync(data_packet_d[3] + param->get_datapacket_size()/2, param->device_IDs[3],
            data_packet_d[7], param->device_IDs[7],
            sizeof(Global_Data_Packet) * param->get_datapacket_size()/2, streams[7]));
        // 4567->0123
    }

    // std::cout << "end gpuquery.cu GPUQuery::transfer_data_packet()" << std::endl;
}

template<>
void GPUquery::gen_param<Test_Query>(std::shared_ptr<Param> param) {
    std::cout << "start gpuquery.cu GPUquery::test_query_gen_param()" << std::endl;

    for (uint32_t i = 0; i < param->get_subtxn_kinds(); ++i) {
        exec_param_h[i].bitmap_size = param->batch_size % 32 > 0
                                          ? param->batch_size / 32 + 1
                                          : param->batch_size / 32;
        exec_param_h[i].global_sub_txn_size = param->get_sub_txn_size();
    }

    // test_query
    global_txn_info_h[0].select_cnt = 1 * param->test_query_batch_size;
    global_txn_info_h[0].cur_subtxn_cnt = 1;

    global_txn_info_h[1].insert_cnt = 1 * param->test_query_batch_size;
    global_txn_info_h[1].cur_subtxn_cnt = 1;

    global_txn_info_h[2].update_cnt = 1 * param->test_query_batch_size;
    global_txn_info_h[2].cur_subtxn_cnt = 1;

    global_txn_info_h[3].scan_cnt = 1 * param->test_query_batch_size;
    global_txn_info_h[3].delete_cnt = 1 * param->test_query_batch_size;
    global_txn_info_h[3].cur_subtxn_cnt = 2;

    // test_query_2
    global_txn_info_h[4].select_cnt = 1 * param->test_query_2_batch_size;
    global_txn_info_h[4].cur_subtxn_cnt = 1;

    global_txn_info_h[5].select_cnt = 1 * param->test_query_2_batch_size;
    global_txn_info_h[5].cur_subtxn_cnt = 1;

    global_txn_info_h[6].select_cnt = 1 * param->test_query_2_batch_size;
    global_txn_info_h[6].cur_subtxn_cnt = 1;

    global_txn_info_h[7].select_cnt = 1 * param->test_query_2_batch_size;
    global_txn_info_h[7].cur_subtxn_cnt = 1;

    global_txn_info_h[8].select_cnt = 1 * param->test_query_2_batch_size;
    global_txn_info_h[8].cur_subtxn_cnt = 1;

    exec_param_h[0].target_platform = 1;
    exec_param_h[0].target_GPU = 0xffffffff;
    exec_param_h[0].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[0].global_sub_txn_size = param->test_sub_txn_size;
    exec_param_h[0].batch_size = param->batch_size;

    exec_param_h[1].target_platform = 1;
    exec_param_h[1].target_GPU = 0xffffffff;
    exec_param_h[1].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[1].global_sub_txn_size = param->test_sub_txn_size;
    exec_param_h[1].batch_size = param->batch_size;

    exec_param_h[2].target_platform = 1;
    exec_param_h[2].target_GPU = 0xffffffff;
    exec_param_h[2].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[2].global_sub_txn_size = param->test_sub_txn_size;
    exec_param_h[2].batch_size = param->batch_size;

    exec_param_h[3].target_platform = 1;
    exec_param_h[3].target_GPU = 0xffffffff;
    exec_param_h[3].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[3].global_sub_txn_size = param->test_sub_txn_size;
    exec_param_h[3].batch_size = param->batch_size;

    exec_param_h[4].target_platform = 1;
    exec_param_h[4].target_GPU = 0xffffffff;
    exec_param_h[4].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[4].global_sub_txn_size = param->test_sub_txn_size;
    exec_param_h[4].batch_size = param->batch_size;

    exec_param_h[5].target_platform = 1;
    exec_param_h[5].target_GPU = 0xffffffff;
    exec_param_h[5].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[5].global_sub_txn_size = param->test_sub_txn_size;
    exec_param_h[5].batch_size = param->batch_size;

    exec_param_h[6].target_platform = 1;
    exec_param_h[6].target_GPU = 0xffffffff;
    exec_param_h[6].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[6].global_sub_txn_size = param->test_sub_txn_size;
    exec_param_h[6].batch_size = param->batch_size;

    exec_param_h[7].target_platform = 1;
    exec_param_h[7].target_GPU = 0xffffffff;
    exec_param_h[7].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[7].global_sub_txn_size = param->test_sub_txn_size;
    exec_param_h[7].batch_size = param->batch_size;

    exec_param_h[8].target_platform = 1;
    exec_param_h[8].target_GPU = 0xffffffff;
    exec_param_h[8].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[8].global_sub_txn_size = param->test_sub_txn_size;
    exec_param_h[8].batch_size = param->batch_size;

    std::cout << "end gpuquery.cu GPUquery::test_query_gen_param()" << std::endl;
}

template<>
void GPUquery::gen_param<TPCC_PART>(std::shared_ptr<Param> param) {
    std::cout << "start gpuquery.cu GPUquery::tpcc_part_query_gen_param()" << std::endl;
    for (uint32_t i = 0; i < param->get_subtxn_kinds(); ++i) {
        exec_param_h[i].bitmap_size = param->batch_size % 32 > 0
                                          ? param->batch_size / 32 + 1
                                          : param->batch_size / 32;
    }

    global_txn_info_h[0].select_cnt = 1 * param->neworder_query_batch_size;
    global_txn_info_h[0].cur_subtxn_cnt = 1;
    exec_param_h[0].target_platform = 1;
    exec_param_h[0].target_GPU = 0xffffffff;
    exec_param_h[0].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[0].global_sub_txn_size = param->neworder_query_batch_size;
    exec_param_h[0].batch_size = param->batch_size;

    global_txn_info_h[1].select_cnt = 1 * param->neworder_query_batch_size;
    global_txn_info_h[1].cur_subtxn_cnt = 1;
    exec_param_h[1].target_platform = 1;
    exec_param_h[1].target_GPU = 0xffffffff;
    exec_param_h[1].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[1].global_sub_txn_size = param->neworder_query_batch_size;
    exec_param_h[1].batch_size = param->batch_size;

    global_txn_info_h[2].select_cnt = 1 * param->neworder_query_batch_size;
    global_txn_info_h[2].cur_subtxn_cnt = 1;
    exec_param_h[2].target_platform = 1;
    exec_param_h[2].target_GPU = 0xffffffff;
    exec_param_h[2].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[2].global_sub_txn_size = param->neworder_query_batch_size;
    exec_param_h[2].batch_size = param->batch_size;

    global_txn_info_h[3].insert_cnt = 1 * param->neworder_query_batch_size;
    global_txn_info_h[3].cur_subtxn_cnt = 1;
    exec_param_h[3].target_platform = 1;
    exec_param_h[3].target_GPU = 0xffffffff;
    exec_param_h[3].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[3].global_sub_txn_size = param->neworder_query_batch_size;
    exec_param_h[3].batch_size = param->batch_size;

    global_txn_info_h[4].insert_cnt = 1 * param->neworder_query_batch_size;
    global_txn_info_h[4].cur_subtxn_cnt = 1;
    exec_param_h[4].target_platform = 1;
    exec_param_h[4].target_GPU = 0xffffffff;
    exec_param_h[4].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[4].global_sub_txn_size = param->neworder_query_batch_size;
    exec_param_h[4].batch_size = param->batch_size;

    global_txn_info_h[5].select_cnt = 1 * param->neworder_query_batch_size;
    global_txn_info_h[5].cur_subtxn_cnt = 2;
    exec_param_h[5].target_platform = 1;
    exec_param_h[5].target_GPU = 0xffffffff;
    exec_param_h[5].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[5].global_sub_txn_size = param->neworder_query_batch_size;
    exec_param_h[5].batch_size = param->batch_size;

    global_txn_info_h[6].insert_cnt = 1 * param->neworder_query_batch_size;
    global_txn_info_h[6].cur_subtxn_cnt = 2;
    exec_param_h[6].target_platform = 1;
    exec_param_h[6].target_GPU = 0xffffffff;
    exec_param_h[6].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[6].global_sub_txn_size = param->neworder_query_batch_size;
    exec_param_h[6].batch_size = param->batch_size;

    global_txn_info_h[7].update_cnt = 1 * param->neworder_query_batch_size;
    global_txn_info_h[7].cur_subtxn_cnt = 1;
    exec_param_h[7].target_platform = 1;
    exec_param_h[7].target_GPU = 0xffffffff;
    exec_param_h[7].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[7].global_sub_txn_size = param->neworder_query_batch_size;
    exec_param_h[7].batch_size = param->batch_size;

    global_txn_info_h[8].select_cnt = 1 * param->payment_query_batch_size;
    global_txn_info_h[8].cur_subtxn_cnt = 1;
    exec_param_h[8].target_platform = 1;
    exec_param_h[8].target_GPU = 0xffffffff;
    exec_param_h[8].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[8].global_sub_txn_size = param->payment_query_batch_size;
    exec_param_h[8].batch_size = param->batch_size;

    global_txn_info_h[9].select_cnt = 1 * param->payment_query_batch_size;
    global_txn_info_h[9].cur_subtxn_cnt = 1;
    exec_param_h[9].target_platform = 1;
    exec_param_h[9].target_GPU = 0xffffffff;
    exec_param_h[9].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[9].global_sub_txn_size = param->payment_query_batch_size;
    exec_param_h[9].batch_size = param->batch_size;

    global_txn_info_h[10].select_cnt = 1 * param->payment_query_batch_size;
    global_txn_info_h[10].cur_subtxn_cnt = 1;
    exec_param_h[10].target_platform = 1;
    exec_param_h[10].target_GPU = 0xffffffff;
    exec_param_h[10].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[10].global_sub_txn_size = param->payment_query_batch_size;
    exec_param_h[10].batch_size = param->batch_size;

    global_txn_info_h[11].update_cnt = 1 * param->payment_query_batch_size;
    global_txn_info_h[11].cur_subtxn_cnt = 1;
    exec_param_h[11].target_platform = 1;
    exec_param_h[11].target_GPU = 0xffffffff;
    exec_param_h[11].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[11].global_sub_txn_size = param->payment_query_batch_size;
    exec_param_h[11].batch_size = param->batch_size;

    global_txn_info_h[12].update_cnt = 1 * param->payment_query_batch_size;
    global_txn_info_h[12].cur_subtxn_cnt = 1;
    exec_param_h[12].target_platform = 1;
    exec_param_h[12].target_GPU = 0xffffffff;
    exec_param_h[12].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[12].global_sub_txn_size = param->payment_query_batch_size;
    exec_param_h[12].batch_size = param->batch_size;

    global_txn_info_h[13].update_cnt = 1 * param->payment_query_batch_size;
    global_txn_info_h[13].cur_subtxn_cnt = 1;
    exec_param_h[13].target_platform = 1;
    exec_param_h[13].target_GPU = 0xffffffff;
    exec_param_h[13].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[13].global_sub_txn_size = param->payment_query_batch_size;
    exec_param_h[13].batch_size = param->batch_size;

    global_txn_info_h[14].update_cnt = 1 * param->payment_query_batch_size;
    global_txn_info_h[14].cur_subtxn_cnt = 1;
    exec_param_h[14].target_platform = 1;
    exec_param_h[14].target_GPU = 0xffffffff;
    exec_param_h[14].global_txn_info_size = param->get_subtxn_kinds();
    exec_param_h[14].global_sub_txn_size = param->payment_query_batch_size;
    exec_param_h[14].batch_size = param->batch_size;
    std::cout << "end gpuquery.cu GPUquery::tpcc_part_query_gen_param()" << std::endl;
}

template<>
void GPUquery::gen_param<TPCC_ALL>(std::shared_ptr<Param> param) {
    std::cout << "start gpuquery.cu GPUquery::tpcc_all_query_gen_param()" << std::endl;

    for (uint32_t i = 0; i < param->get_subtxn_kinds(); ++i) {
        exec_param_h[i].bitmap_size = param->batch_size % 32 > 0
                                          ? param->batch_size / 32 + 1
                                          : param->batch_size / 32;
        exec_param_h[i].global_txn_info_size = param->get_subtxn_kinds();
        exec_param_h[i].batch_size = param->batch_size;
    }

    global_txn_info_h[0].select_cnt = 1 * param->neworder_query_batch_size;
    global_txn_info_h[0].cur_subtxn_cnt = 1;
    exec_param_h[0].target_platform = 1;
    exec_param_h[0].target_GPU = 0xffffffff;
    exec_param_h[0].global_sub_txn_size = param->neworder_query_batch_size;


    global_txn_info_h[1].select_cnt = 1 * param->neworder_query_batch_size;
    global_txn_info_h[1].cur_subtxn_cnt = 1;
    exec_param_h[1].target_platform = 1;
    exec_param_h[1].target_GPU = 0xffffffff;
    exec_param_h[1].global_sub_txn_size = param->neworder_query_batch_size;


    global_txn_info_h[2].select_cnt = 1 * param->neworder_query_batch_size;
    global_txn_info_h[2].cur_subtxn_cnt = 1;
    exec_param_h[2].target_platform = 1;
    exec_param_h[2].target_GPU = 0xffffffff;
    exec_param_h[2].global_sub_txn_size = param->neworder_query_batch_size;


    global_txn_info_h[3].insert_cnt = 1 * param->neworder_query_batch_size;
    global_txn_info_h[3].cur_subtxn_cnt = 1;
    exec_param_h[3].target_platform = 1;
    exec_param_h[3].target_GPU = 0xffffffff;
    exec_param_h[3].global_sub_txn_size = param->neworder_query_batch_size;

    global_txn_info_h[4].insert_cnt = 1 * param->neworder_query_batch_size;
    global_txn_info_h[4].cur_subtxn_cnt = 1;
    exec_param_h[4].target_platform = 1;
    exec_param_h[4].target_GPU = 0xffffffff;
    exec_param_h[4].global_sub_txn_size = param->neworder_query_batch_size;

    global_txn_info_h[5].select_cnt = 1 * param->neworder_query_batch_size;
    global_txn_info_h[5].cur_subtxn_cnt = 1;
    exec_param_h[5].target_platform = 1;
    exec_param_h[5].target_GPU = 0xffffffff;
    exec_param_h[5].global_sub_txn_size = param->neworder_query_batch_size;

    global_txn_info_h[6].insert_cnt = 1 * param->neworder_query_batch_size;
    global_txn_info_h[6].cur_subtxn_cnt = 2;
    exec_param_h[6].target_platform = 1;
    exec_param_h[6].target_GPU = 0xffffffff;
    exec_param_h[6].global_sub_txn_size = param->neworder_query_batch_size;

    global_txn_info_h[7].update_cnt = 1 * param->neworder_query_batch_size;
    global_txn_info_h[7].cur_subtxn_cnt = 1;
    exec_param_h[7].target_platform = 1;
    exec_param_h[7].target_GPU = 0xffffffff;
    exec_param_h[7].global_sub_txn_size = param->neworder_query_batch_size;

    //payment
    global_txn_info_h[8].select_cnt = 1 * param->payment_query_batch_size;
    global_txn_info_h[8].cur_subtxn_cnt = 1;
    exec_param_h[8].target_platform = 1;
    exec_param_h[8].target_GPU = 0xffffffff;
    exec_param_h[8].global_sub_txn_size = param->payment_query_batch_size;

    global_txn_info_h[9].select_cnt = 1 * param->payment_query_batch_size;
    global_txn_info_h[9].cur_subtxn_cnt = 1;
    exec_param_h[9].target_platform = 1;
    exec_param_h[9].target_GPU = 0xffffffff;
    exec_param_h[9].global_sub_txn_size = param->payment_query_batch_size;

    global_txn_info_h[10].select_cnt = 1 * param->payment_query_batch_size;
    global_txn_info_h[10].cur_subtxn_cnt = 1;
    exec_param_h[10].target_platform = 1;
    exec_param_h[10].target_GPU = 0xffffffff;
    exec_param_h[10].global_sub_txn_size = param->payment_query_batch_size;

    global_txn_info_h[11].update_cnt = 1 * param->payment_query_batch_size;
    global_txn_info_h[11].cur_subtxn_cnt = 1;
    exec_param_h[11].target_platform = 1;
    exec_param_h[11].target_GPU = 0xffffffff;
    exec_param_h[11].global_sub_txn_size = param->payment_query_batch_size;

    global_txn_info_h[12].update_cnt = 1 * param->payment_query_batch_size;
    global_txn_info_h[12].cur_subtxn_cnt = 1;
    exec_param_h[12].target_platform = 1;
    exec_param_h[12].target_GPU = 0xffffffff;
    exec_param_h[12].global_sub_txn_size = param->payment_query_batch_size;

    global_txn_info_h[13].update_cnt = 1 * param->payment_query_batch_size;
    global_txn_info_h[13].cur_subtxn_cnt = 1;
    exec_param_h[13].target_platform = 1;
    exec_param_h[13].target_GPU = 0xffffffff;
    exec_param_h[13].global_sub_txn_size = param->payment_query_batch_size;

    global_txn_info_h[14].update_cnt = 1 * param->payment_query_batch_size;
    global_txn_info_h[14].cur_subtxn_cnt = 1;
    exec_param_h[14].target_platform = 1;
    exec_param_h[14].target_GPU = 0xffffffff;
    exec_param_h[14].global_sub_txn_size = param->payment_query_batch_size;

    // Orderstatus
    global_txn_info_h[15].select_cnt = 1 * param->orderstatus_query_batch_size;
    global_txn_info_h[15].cur_subtxn_cnt = 1;
    exec_param_h[15].target_platform = 1;
    exec_param_h[15].target_GPU = 0xffffffff;
    exec_param_h[15].global_sub_txn_size = param->orderstatus_query_batch_size;

    global_txn_info_h[16].select_cnt = 1 * param->orderstatus_query_batch_size;
    global_txn_info_h[16].cur_subtxn_cnt = 1;
    exec_param_h[16].target_platform = 1;
    exec_param_h[16].target_GPU = 0xffffffff;
    exec_param_h[16].global_sub_txn_size = param->orderstatus_query_batch_size;

    global_txn_info_h[17].select_cnt = 1 * param->orderstatus_query_batch_size;
    global_txn_info_h[17].cur_subtxn_cnt = 1;
    exec_param_h[17].target_platform = 1;
    exec_param_h[17].target_GPU = 0xffffffff;
    exec_param_h[17].global_sub_txn_size = param->orderstatus_query_batch_size;

    // Delivery
    global_txn_info_h[18].delete_cnt = 1 * param->delivery_query_batch_size;
    global_txn_info_h[18].cur_subtxn_cnt = 1;
    exec_param_h[18].target_platform = 1;
    exec_param_h[18].target_GPU = 0xffffffff;
    exec_param_h[18].global_sub_txn_size = param->delivery_query_batch_size;

    global_txn_info_h[19].update_cnt = 1 * param->delivery_query_batch_size;
    global_txn_info_h[19].cur_subtxn_cnt = 1;
    exec_param_h[19].target_platform = 1;
    exec_param_h[19].target_GPU = 0xffffffff;
    exec_param_h[19].global_sub_txn_size = param->delivery_query_batch_size;

    global_txn_info_h[20].update_cnt = 1 * param->delivery_query_batch_size;
    global_txn_info_h[20].cur_subtxn_cnt = 1;
    exec_param_h[20].target_platform = 1;
    exec_param_h[20].target_GPU = 0xffffffff;
    exec_param_h[20].global_sub_txn_size = param->delivery_query_batch_size;

    global_txn_info_h[21].select_cnt = 1 * param->delivery_query_batch_size;
    global_txn_info_h[21].cur_subtxn_cnt = 1;
    exec_param_h[21].target_platform = 1;
    exec_param_h[21].target_GPU = 0xffffffff;
    exec_param_h[21].global_sub_txn_size = param->delivery_query_batch_size;

    global_txn_info_h[22].update_cnt = 1 * param->delivery_query_batch_size;
    global_txn_info_h[22].cur_subtxn_cnt = 1;
    exec_param_h[22].target_platform = 1;
    exec_param_h[22].target_GPU = 0xffffffff;
    exec_param_h[22].global_sub_txn_size = param->delivery_query_batch_size;

    // Stocklevel
    global_txn_info_h[23].select_cnt = 1 * param->stocklevel_query_batch_size;
    global_txn_info_h[23].cur_subtxn_cnt = 1;
    exec_param_h[23].target_platform = 1;
    exec_param_h[23].target_GPU = 0xffffffff;
    exec_param_h[23].global_sub_txn_size = param->stocklevel_query_batch_size;

    global_txn_info_h[24].select_cnt = 1 * param->stocklevel_query_batch_size;
    global_txn_info_h[24].cur_subtxn_cnt = 1;
    exec_param_h[24].target_platform = 1;
    exec_param_h[24].target_GPU = 0xffffffff;
    exec_param_h[24].global_sub_txn_size = param->stocklevel_query_batch_size;

    global_txn_info_h[25].select_cnt = 1 * param->stocklevel_query_batch_size;
    global_txn_info_h[25].cur_subtxn_cnt = 1;
    exec_param_h[25].target_platform = 1;
    exec_param_h[25].target_GPU = 0xffffffff;
    exec_param_h[25].global_sub_txn_size = param->stocklevel_query_batch_size;
    std::cout << "end gpuquery.cu GPUquery::tpcc_all_query_gen_param()" << std::endl;
}

template<>
void GPUquery::gen_param<YCSB_A_Query>(std::shared_ptr<Param> param) {
    std::cout << "start gpuquery.cu GPUquery::ycsb_a_query_gen_param()" << std::endl;
    for (uint32_t i = 0; i < param->get_subtxn_kinds(); ++i) {
        exec_param_h[i].bitmap_size = param->batch_size % 32 > 0
                                          ? param->batch_size / 32 + 1
                                          : param->batch_size / 32;
        exec_param_h[i].global_txn_info_size = param->get_subtxn_kinds();
        exec_param_h[i].batch_size = param->batch_size;
    }
    for (uint32_t i = 0; i < 5; ++i) {
        global_txn_info_h[i].select_cnt = 1 * param->ycsb_a_query_batch_size;
        global_txn_info_h[i].cur_subtxn_cnt = 1;
        exec_param_h[i].target_platform = 1;
        exec_param_h[i].target_GPU = 0xffffffff;
        exec_param_h[i].global_sub_txn_size = param->ycsb_a_query_batch_size;
    }
    for (uint32_t i = 5; i < 10; ++i) {
        global_txn_info_h[i].update_cnt = 1 * param->ycsb_a_query_batch_size;
        global_txn_info_h[i].cur_subtxn_cnt = 1;
        exec_param_h[i].target_platform = 1;
        exec_param_h[i].target_GPU = 0xffffffff;
        exec_param_h[i].global_sub_txn_size = param->ycsb_a_query_batch_size;
    }
    std::cout << "end gpuquery.cu GPUquery::ycsb_a_query_gen_param()" << std::endl;
}

template<>
void GPUquery::gen_param<YCSB_B_Query>(std::shared_ptr<Param> param) {
    std::cout << "start gpuquery.cu GPUquery::ycsb_b_query_gen_param()" << std::endl;
    for (uint32_t i = 0; i < param->get_subtxn_kinds(); ++i) {
        exec_param_h[i].bitmap_size = param->batch_size % 32 > 0
                                          ? param->batch_size / 32 + 1
                                          : param->batch_size / 32;
        exec_param_h[i].global_txn_info_size = param->get_subtxn_kinds();
        exec_param_h[i].batch_size = param->batch_size;
    }
    for (uint32_t i = 0; i < 9; ++i) {
        global_txn_info_h[i].select_cnt = 1 * param->ycsb_b_query_batch_size;
        global_txn_info_h[i].cur_subtxn_cnt = 1;
        exec_param_h[i].target_platform = 1;
        exec_param_h[i].target_GPU = 0xffffffff;
        exec_param_h[i].global_sub_txn_size = param->ycsb_b_query_batch_size;
    }

    global_txn_info_h[9].update_cnt = 1 * param->ycsb_b_query_batch_size;
    global_txn_info_h[9].cur_subtxn_cnt = 1;
    exec_param_h[9].target_platform = 1;
    exec_param_h[9].target_GPU = 0xffffffff;
    exec_param_h[9].global_sub_txn_size = param->ycsb_b_query_batch_size;
    std::cout << "end gpuquery.cu GPUquery::ycsb_b_query_gen_param()" << std::endl;
}

template<>
void GPUquery::gen_param<YCSB_C_Query>(std::shared_ptr<Param> param) {
    std::cout << "start gpuquery.cu GPUquery::ycsb_c_query_gen_param()" << std::endl;
    for (uint32_t i = 0; i < param->get_subtxn_kinds(); ++i) {
        exec_param_h[i].bitmap_size = param->batch_size % 32 > 0
                                          ? param->batch_size / 32 + 1
                                          : param->batch_size / 32;
        exec_param_h[i].global_txn_info_size = param->get_subtxn_kinds();
        exec_param_h[i].batch_size = param->batch_size;
    }
    for (uint32_t i = 0; i < 10; ++i) {
        global_txn_info_h[i].select_cnt = 1 * param->ycsb_c_query_batch_size;
        global_txn_info_h[i].cur_subtxn_cnt = 1;
        exec_param_h[i].target_platform = 1;
        exec_param_h[i].target_GPU = 0xffffffff;
        exec_param_h[i].global_sub_txn_size = param->ycsb_c_query_batch_size;
    }
    std::cout << "end gpuquery.cu GPUquery::ycsb_c_query_gen_param()" << std::endl;
}

template<>
void GPUquery::gen_param<YCSB_D_Query>(std::shared_ptr<Param> param) {
    std::cout << "start gpuquery.cu GPUquery::ycsb_d_query_gen_param()" << std::endl;
    for (uint32_t i = 0; i < param->get_subtxn_kinds(); ++i) {
        exec_param_h[i].bitmap_size = param->batch_size % 32 > 0
                                          ? param->batch_size / 32 + 1
                                          : param->batch_size / 32;
        exec_param_h[i].global_txn_info_size = param->get_subtxn_kinds();
        exec_param_h[i].batch_size = param->batch_size;
    }
    for (uint32_t i = 0; i < 9; ++i) {
        global_txn_info_h[i].select_cnt = 1 * param->ycsb_d_query_batch_size;
        global_txn_info_h[i].cur_subtxn_cnt = 1;
        exec_param_h[i].target_platform = 1;
        exec_param_h[i].target_GPU = 0xffffffff;
        exec_param_h[i].global_sub_txn_size = param->ycsb_d_query_batch_size;
    }

    global_txn_info_h[9].update_cnt = 1 * param->ycsb_d_query_batch_size;
    global_txn_info_h[9].cur_subtxn_cnt = 1;
    exec_param_h[9].target_platform = 1;
    exec_param_h[9].target_GPU = 0xffffffff;
    exec_param_h[9].global_sub_txn_size = param->ycsb_d_query_batch_size;
    std::cout << "end gpuquery.cu GPUquery::ycsb_d_query_gen_param()" << std::endl;
}

template<>
void GPUquery::gen_param<YCSB_E_Query>(std::shared_ptr<Param> param) {
    std::cout << "start gpuquery.cu GPUquery::ycsb_e_query_gen_param()" << std::endl;
    for (uint32_t i = 0; i < param->get_subtxn_kinds(); ++i) {
        exec_param_h[i].bitmap_size = param->batch_size % 32 > 0
                                          ? param->batch_size / 32 + 1
                                          : param->batch_size / 32;
        exec_param_h[i].global_txn_info_size = param->get_subtxn_kinds();
        exec_param_h[i].batch_size = param->batch_size;
    }
    for (uint32_t i = 0; i < 9; ++i) {
        global_txn_info_h[i].scan_cnt = 1 * param->ycsb_e_query_batch_size;
        global_txn_info_h[i].cur_subtxn_cnt = 1;
        exec_param_h[i].target_platform = 1;
        exec_param_h[i].target_GPU = 0xffffffff;
        exec_param_h[i].global_sub_txn_size = param->ycsb_e_query_batch_size;
    }

    global_txn_info_h[9].insert_cnt = 1 * param->ycsb_e_query_batch_size;
    global_txn_info_h[9].cur_subtxn_cnt = 1;
    exec_param_h[9].target_platform = 1;
    exec_param_h[9].target_GPU = 0xffffffff;
    exec_param_h[9].global_sub_txn_size = param->ycsb_e_query_batch_size;
    std::cout << "end gpuquery.cu GPUquery::ycsb_e_query_gen_param()" << std::endl;
}

template<typename T>
void GPUquery::query_parse(std::shared_ptr<Param> param, std::shared_ptr<std::vector<std::any> > transactions_batch_ptr,
                           Global_Table_Meta **meta, Global_Table_Index **index) {
    std::cout << "start gpuquery.cu GPUquery::query_parse()" << std::endl;

    hipStream_t *streams;
    streams = new hipStream_t[param->device_cnt];
    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipSetDevice(param->device_IDs[i]));
        CHECK(hipStreamCreate(&streams[i]));
    }


    T *t_query_h;
    T **t_query_d;
    CHECK(hipHostAlloc((void **)&t_query_h, sizeof(T) * param->get_txn_batch_size(typeid(T)), hipHostMallocDefault));
    uint32_t cur = 0;
    for (auto transaction = transactions_batch_ptr->begin(); transaction != transactions_batch_ptr->end(); ++
         transaction) {
        if (typeid(T) == transaction->type()) {
            t_query_h[cur] = std::any_cast<T>(*transaction);
            ++cur;
        }
    }

    CHECK(hipHostAlloc((void **)&t_query_d, sizeof(T) * param->device_cnt, hipHostMallocDefault));
    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipSetDevice(param->device_IDs[i]));
        CHECK(hipMalloc((void **)&t_query_d[i], sizeof(T) * param->get_txn_batch_size(typeid(T))));
    }

    // std::cout << get_global_txn_start(param, typeid(T)) << std::endl;

    long long start_parse = gpu_current_time();
    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipSetDevice(param->device_IDs[i]));
        CHECK(hipMemcpyAsync(t_query_d[i], t_query_h, sizeof(T) * param->get_txn_batch_size(typeid(T)),
            hipMemcpyHostToDevice, streams[i]));
        parse<T><<<512, 512, 0, streams[i]>>>(param->get_txn_batch_size(typeid(T)),
                                              get_global_txn_start(param, typeid(T)), global_txn_info_d[i],
                                              t_query_d[i], global_txn_d[i], exec_param_d[i], meta[i], index[i]);
    }

    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipStreamSynchronize(streams[i]));
    }
    long long end_parse = gpu_current_time();
    float cost_parse = gpu_duration(start_parse, end_parse);
    std::cout << "cost_parse:" << cost_parse << "s." << std::endl;
    CHECK(hipMemcpy(global_txn, global_txn_d[0],
        sizeof(Global_Txn) * param->get_sub_txn_size(), hipMemcpyDeviceToHost));


    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipFree(t_query_d[i]));
    }
    CHECK(hipHostFree(t_query_d));
    CHECK(hipHostFree(t_query_h));

    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipStreamDestroy(streams[i]));
    }

    delete[] streams;

    std::cout << "end gpuquery.cu GPUquery::query_parse()" << std::endl;
}

uint32_t GPUquery::get_global_txn_info_ID(std::shared_ptr<Param> param, uint32_t sub_txn_ID,
                                          const std::type_info &txn_type) {
    uint32_t result = sub_txn_ID;
    if (param->benchmark == "TEST") {
        if (txn_type == typeid(Test_Query)) {
            return result;
        } else if (txn_type == typeid(Test_Query_2)) {
            result += 5;
            return result;
        }
    } else if (param->benchmark == "TPCC_PART") {
        if (txn_type == typeid(Neworder_Query)) {
            if (sub_txn_ID < 5) {
                result = sub_txn_ID;
            } else {
                result = 5 + (sub_txn_ID - 5) % 3;
            }
            return result;
        } else if (txn_type == typeid(Payment_Query)) {
            result += 8;
            return result;
        }
    } else if (param->benchmark == "TPCC_ALL") {
        if (txn_type == typeid(Neworder_Query)) {
            if (sub_txn_ID < 5) {
                result = sub_txn_ID;
            } else {
                result = 5 + (sub_txn_ID - 5) % 3;
            }
            return result;
        } else if (txn_type == typeid(Payment_Query)) {
            result += 8;
            return result;
        } else if (txn_type == typeid(Orderstatus_Query)) {
            result += 8;
            result += 7;
            return result;
        } else if (txn_type == typeid(Delivery_Query)) {
            result = sub_txn_ID % 5;
            result += 8;
            result += 7;
            result += 3;
            return result;
        } else if (txn_type == typeid(Stocklevel_Query)) {
            if (sub_txn_ID > 0)
                result = 1 + (sub_txn_ID - 1) % 2;
            result += 8;
            result += 7;
            result += 3;
            result += 5;
            return result;
        }
    } else if (param->benchmark == "YCSB_A") {
        return result;
    } else if (param->benchmark == "YCSB_B") {
        return result;
    } else if (param->benchmark == "YCSB_C") {
        return result;
    } else if (param->benchmark == "YCSB_D") {
        return result;
    } else if (param->benchmark == "YCSB_E") {
        return result;
    }

    return 0;
}

uint32_t GPUquery::get_global_txn_start(std::shared_ptr<Param> param, const std::type_info &txn_type) {
    uint32_t result = 0;
    if (param->benchmark == "TEST") {
        if (txn_type == typeid(Test_Query)) {
            return result;
        } else if (txn_type == typeid(Test_Query_2)) {
            result += param->test_query_subtxn_cnt * param->test_query_batch_size;
            return result;
        }
    } else if (param->benchmark == "TPCC_PART") {
        if (txn_type == typeid(Neworder_Query)) {
            return result;
        } else if (txn_type == typeid(Payment_Query)) {
            result += param->neworder_query_subtxn_cnt * param->neworder_query_batch_size;
            return result;
        }
    } else if (param->benchmark == "TPCC_ALL") {
        if (txn_type == typeid(Neworder_Query)) {
            return result;
        } else if (txn_type == typeid(Payment_Query)) {
            result += param->neworder_query_subtxn_cnt * param->neworder_query_batch_size;
            return result;
        } else if (txn_type == typeid(Orderstatus_Query)) {
            result += param->neworder_query_subtxn_cnt * param->neworder_query_batch_size;
            result += param->payment_query_subtxn_cnt * param->payment_query_batch_size;
            return result;
        } else if (txn_type == typeid(Delivery_Query)) {
            result += param->neworder_query_subtxn_cnt * param->neworder_query_batch_size;
            result += param->payment_query_subtxn_cnt * param->payment_query_batch_size;
            result += param->orderstatus_query_subtxn_cnt * param->orderstatus_query_batch_size;
            return result;
        } else if (txn_type == typeid(Stocklevel_Query)) {
            result += param->neworder_query_subtxn_cnt * param->neworder_query_batch_size;
            result += param->payment_query_subtxn_cnt * param->payment_query_batch_size;
            result += param->orderstatus_query_subtxn_cnt * param->orderstatus_query_batch_size;
            result += param->delivery_query_subtxn_cnt * param->delivery_query_batch_size;
            return result;
        }
    } else if (param->benchmark == "YCSB_A") {
        return result;
    } else if (param->benchmark == "YCSB_B") {
        return result;
    } else if (param->benchmark == "YCSB_C") {
        return result;
    } else if (param->benchmark == "YCSB_D") {
        return result;
    } else if (param->benchmark == "YCSB_E") {
        return result;
    }
    return 0;
}

Global_Txn_Info *GPUquery::get_txn_info(const int deviceID) {
    return global_txn_info_d[deviceID];
}

Global_Txn *GPUquery::get_txn(const int deviceID) {
    return global_txn_d[deviceID];
}

Global_Txn_Exec *GPUquery::get_txn_exec(const int deviceID) {
    return global_txn_exec_d[deviceID];
}

Global_Txn_Result *GPUquery::get_txn_result(const int deviceID) {
    return global_txn_result_d[deviceID];
}

Global_Txn_Exec_Param *GPUquery::get_exec_param(const int deviceID) {
    return exec_param_d[deviceID];
}

Global_Txn_Aux_Struct *GPUquery::get_aux_struct(const int deviceID) {
    return aux_struct_d[deviceID];
}

Global_Data_Packet *GPUquery::get_data_packet(const int deviceID) {
    return data_packet_d[deviceID];
}

template<>
__global__ void parse<Test_Query>(uint32_t size, uint32_t start, Global_Txn_Info *txn_info, Test_Query *query,
                                  Global_Txn *txn, Global_Txn_Exec_Param *param, Global_Table_Meta *meta,
                                  Global_Table_Index *index) {
    uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t thSize = blockDim.x * gridDim.x;
    uint32_t cur = thID;
    while (cur < size) {
        txn[start + 0 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 0 * size + cur].subtxn.ispopular = 0;
        txn[start + 0 * size + cur].subtxn.type = 0; // select
        txn[start + 0 * size + cur].subtxn.table_ID = 1;
        txn[start + 0 * size + cur].subtxn.benchmark = 1;
        txn[start + 0 * size + cur].subtxn.dest_Row_1 = query[cur].Row_0;
        txn[start + 0 * size + cur].subtxn.dest_device = 0xffffffff; // query[cur].Row_0 / meta[1].table_slice_size;
        txn[start + 0 * size + cur].dest_device = 0xffffffff; // txn[start + 0 * size + cur].subtxn.dest_device;
        txn[start + 0 * size + cur].sub_txn_cnt = 1;
        txn[start + 0 * size + cur].global_txn_info_ID = 0;

        txn[start + 1 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 1 * size + cur].subtxn.ispopular = 0;
        txn[start + 1 * size + cur].subtxn.type = 1; // insert
        txn[start + 1 * size + cur].subtxn.table_ID = 1;
        txn[start + 1 * size + cur].subtxn.benchmark = 1;
        txn[start + 1 * size + cur].subtxn.dest_Row_1 = query[cur].Row_1;
        txn[start + 1 * size + cur].subtxn.dest_device = query[cur].Row_1 / meta[1].table_slice_size;
        txn[start + 1 * size + cur].dest_device = txn[start + 1 * size + cur].subtxn.dest_device;
        txn[start + 1 * size + cur].global_txn_info_ID = 1;

        txn[start + 2 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 2 * size + cur].subtxn.ispopular = 1;
        txn[start + 2 * size + cur].subtxn.type = 2; // update
        txn[start + 2 * size + cur].subtxn.table_ID = 1;
        txn[start + 2 * size + cur].subtxn.benchmark = 1;
        txn[start + 2 * size + cur].subtxn.dest_Row_1 = query[cur].Row_2;
        txn[start + 2 * size + cur].subtxn.dest_device = query[cur].Row_2 / meta[1].table_slice_size;
        txn[start + 2 * size + cur].dest_device = txn[start + 2 * size + cur].subtxn.dest_device;
        txn[start + 2 * size + cur].global_txn_info_ID = 2;

        txn[start + 3 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 3 * size + cur].subtxn.ispopular = 0;
        txn[start + 3 * size + cur].subtxn.type = 3; // scan
        txn[start + 3 * size + cur].subtxn.table_ID = 1;
        txn[start + 3 * size + cur].subtxn.benchmark = 1;
        txn[start + 3 * size + cur].subtxn.dest_Row_1 = query[cur].Row_3;
        txn[start + 3 * size + cur].subtxn.dest_device = query[cur].Row_3 / meta[1].table_slice_size;
        txn[start + 3 * size + cur].dest_device = txn[start + 3 * size + cur].subtxn.dest_device;
        txn[start + 3 * size + cur].global_txn_info_ID = 3;

        txn[start + 4 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 4 * size + cur].subtxn.ispopular = 0;
        txn[start + 4 * size + cur].subtxn.type = 4; // delete
        txn[start + 4 * size + cur].subtxn.table_ID = 1;
        txn[start + 4 * size + cur].subtxn.benchmark = 1;
        txn[start + 4 * size + cur].subtxn.dest_Row_1 = query[cur].Row_4;
        txn[start + 4 * size + cur].subtxn.dest_device = query[cur].Row_4 / meta[1].table_slice_size;
        txn[start + 4 * size + cur].subtxn.dest_Row_2 = query[cur].Row_5;
        txn[start + 4 * size + cur].subtxn.dest_device = query[cur].Row_5 / meta[1].table_slice_size;
        txn[start + 4 * size + cur].dest_device = txn[start + 3 * size + cur].subtxn.dest_device;
        txn[start + 4 * size + cur].global_txn_info_ID = 4;

        // printf(
        //     "cur:%d,ID:%d %d %d %d %d,TID:%d,%d %d %d %d %d,Row:%d %d %d %d %d %d,dest_device:%d %d %d %d %d,ispopular:%d %d %d %d %d\n",
        //     cur, start + 0 * size + cur, start + 1 * size + cur,
        //     start + 2 * size + cur, start + 3 * size + cur, start + 3 * size + cur,
        //     txn[start + 0 * size + cur].subtxn.TID,
        //     txn[start + 0 * size + cur].subtxn.type,
        //     txn[start + 1 * size + cur].subtxn.type, txn[start + 2 * size + cur].subtxn.type,
        //     txn[start + 3 * size + cur].subtxn.type, txn[start + 3 * size + cur].subtxn[1].type,
        //     txn[start + 0 * size + cur].subtxn.dest_Row_1, txn[start + 1 * size + cur].subtxn.dest_Row_1,
        //     txn[start + 2 * size + cur].subtxn.dest_Row_1, txn[start + 3 * size + cur].subtxn.dest_Row_1,
        //     txn[start + 3 * size + cur].subtxn[1].dest_Row_1, txn[start + 3 * size + cur].subtxn[1].dest_Row_2,
        //     txn[start + 0 * size + cur].dest_device, txn[start + 1 * size + cur].dest_device,
        //     txn[start + 2 * size + cur].dest_device, txn[start + 3 * size + cur].dest_device,
        //     txn[start + 3 * size + cur].dest_device,
        //     txn[start + 0 * size + cur].subtxn.ispopular, txn[start + 1 * size + cur].subtxn.ispopular,
        //     txn[start + 2 * size + cur].subtxn.ispopular, txn[start + 3 * size + cur].subtxn.ispopular,
        //     txn[start + 3 * size + cur].subtxn[1].ispopular);

        // printf("ID:%d,%d,%d,%d,%d,%d,%d,%d,%d\n",
        //        cur, txn_info[0].cur_subtxn_cnt, txn_info[1].cur_subtxn_cnt,
        //        txn_info[2].cur_subtxn_cnt, txn_info[3].cur_subtxn_cnt,
        //        param[0].target_platform, param[1].target_platform,
        //        param[2].target_platform, param[3].target_platform);

        cur += thSize;
    }
}

template<>
__global__ void parse<Test_Query_2>(uint32_t size, uint32_t start, Global_Txn_Info *txn_info, Test_Query_2 *query,
                                    Global_Txn *txn, Global_Txn_Exec_Param *param, Global_Table_Meta *meta,
                                    Global_Table_Index *index) {
    uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t thSize = blockDim.x * gridDim.x;
    uint32_t cur = thID;
    while (cur < size) {
        txn[start + 0 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 0 * size + cur].subtxn.ispopular = 1;
        txn[start + 0 * size + cur].subtxn.type = 0;
        txn[start + 0 * size + cur].subtxn.table_ID = 0;
        txn[start + 0 * size + cur].subtxn.benchmark = 1;
        txn[start + 0 * size + cur].subtxn.dest_Row_1 = query[cur].Row_0;
        txn[start + 0 * size + cur].subtxn.dest_device = query[cur].Row_0 / meta[0].table_slice_size;
        txn[start + 0 * size + cur].dest_device = txn[start + 0 * size + cur].subtxn.dest_device;
        txn[start + 0 * size + cur].global_txn_info_ID = 5;

        txn[start + 1 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 1 * size + cur].subtxn.ispopular = 0;
        txn[start + 1 * size + cur].subtxn.type = 0;
        txn[start + 1 * size + cur].subtxn.table_ID = 0;
        txn[start + 1 * size + cur].subtxn.benchmark = 1;
        txn[start + 1 * size + cur].subtxn.dest_Row_1 = query[cur].Row_1;
        txn[start + 1 * size + cur].subtxn.dest_device = query[cur].Row_1 / meta[0].table_slice_size;
        txn[start + 1 * size + cur].dest_device = txn[start + 1 * size + cur].subtxn.dest_device;
        txn[start + 1 * size + cur].global_txn_info_ID = 6;

        txn[start + 2 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 2 * size + cur].subtxn.ispopular = 0;
        txn[start + 2 * size + cur].subtxn.type = 0;
        txn[start + 2 * size + cur].subtxn.table_ID = 0;
        txn[start + 2 * size + cur].subtxn.benchmark = 1;
        txn[start + 2 * size + cur].subtxn.dest_Row_1 = query[cur].Row_2;
        txn[start + 2 * size + cur].subtxn.dest_device = query[cur].Row_2 / meta[0].table_slice_size;
        txn[start + 2 * size + cur].dest_device = txn[start + 2 * size + cur].subtxn.dest_device;
        txn[start + 2 * size + cur].global_txn_info_ID = 7;

        txn[start + 3 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 3 * size + cur].subtxn.ispopular = 0;
        txn[start + 3 * size + cur].subtxn.type = 0;
        txn[start + 3 * size + cur].subtxn.table_ID = 0;
        txn[start + 3 * size + cur].subtxn.benchmark = 1;
        txn[start + 3 * size + cur].subtxn.dest_Row_1 = query[cur].Row_3;
        txn[start + 3 * size + cur].subtxn.dest_device = query[cur].Row_3 / meta[0].table_slice_size;
        txn[start + 3 * size + cur].dest_device = txn[start + 3 * size + cur].subtxn.dest_device;
        txn[start + 3 * size + cur].global_txn_info_ID = 8;

        txn[start + 4 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 4 * size + cur].subtxn.ispopular = 0;
        txn[start + 4 * size + cur].subtxn.type = 0;
        txn[start + 4 * size + cur].subtxn.table_ID = 0;
        txn[start + 4 * size + cur].subtxn.benchmark = 1;
        txn[start + 4 * size + cur].subtxn.dest_Row_1 = query[cur].Row_4;
        txn[start + 4 * size + cur].subtxn.dest_device = query[cur].Row_4 / meta[0].table_slice_size;
        txn[start + 4 * size + cur].dest_device = txn[start + 4 * size + cur].subtxn.dest_device;
        txn[start + 4 * size + cur].global_txn_info_ID = 9;

        // printf(
        //     "cur:%d,ID:%d %d %d %d %d,TID:%d,%d %d %d %d %d,Row:%d %d %d %d %d,dest_device:%d %d %d %d %d,ispopular:%d %d %d %d %d\n",
        //     cur, start + 0 * size + cur, start + 1 * size + cur,
        //     start + 2 * size + cur, start + 3 * size + cur, start + 4 * size + cur,
        //     txn[start + 0 * size + cur].subtxn.TID,
        //     txn[start + 0 * size + cur].subtxn.type,
        //     txn[start + 1 * size + cur].subtxn.type, txn[start + 2 * size + cur].subtxn.type,
        //     txn[start + 3 * size + cur].subtxn.type, txn[start + 4 * size + cur].subtxn.type,
        //     txn[start + 0 * size + cur].subtxn.dest_Row_1, txn[start + 1 * size + cur].subtxn.dest_Row_1,
        //     txn[start + 2 * size + cur].subtxn.dest_Row_1, txn[start + 3 * size + cur].subtxn.dest_Row_1,
        //     txn[start + 4 * size + cur].subtxn.dest_Row_1,
        //     txn[start + 0 * size + cur].dest_device, txn[start + 1 * size + cur].dest_device,
        //     txn[start + 2 * size + cur].dest_device, txn[start + 3 * size + cur].dest_device,
        //     txn[start + 4 * size + cur].dest_device,
        //     txn[start + 0 * size + cur].subtxn.ispopular, txn[start + 1 * size + cur].subtxn.ispopular,
        //     txn[start + 2 * size + cur].subtxn.ispopular, txn[start + 3 * size + cur].subtxn.ispopular,
        //     txn[start + 4 * size + cur].subtxn.ispopular);

        // printf("ID:%d,%d,%d,%d,%d,%d,%d,%d,%d\n",
        //        cur, txn_info[0].cur_subtxn_cnt, txn_info[1].cur_subtxn_cnt,
        //        txn_info[2].cur_subtxn_cnt, txn_info[3].cur_subtxn_cnt,
        //        param[0].target_platform, param[1].target_platform,
        //        param[2].target_platform, param[3].target_platform);

        cur += thSize;
    }
}

template<>
__global__ void parse<Neworder_Query>(uint32_t size, uint32_t start, Global_Txn_Info *txn_info, Neworder_Query *query,
                                      Global_Txn *txn, Global_Txn_Exec_Param *param, Global_Table_Meta *meta,
                                      Global_Table_Index *index) {
    uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t thSize = blockDim.x * gridDim.x;
    uint32_t cur = thID;
    while (cur < size) {
        txn[start + 0 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 0 * size + cur].subtxn.ispopular = 1;
        txn[start + 0 * size + cur].subtxn.type = 0;
        txn[start + 0 * size + cur].subtxn.table_ID = 0;
        txn[start + 0 * size + cur].subtxn.benchmark = 2;
        txn[start + 0 * size + cur].subtxn.dest_Row_1 = query[cur].W_ID;
        txn[start + 0 * size + cur].subtxn.dest_device = query[cur].W_ID / meta[0].table_slice_size;
        txn[start + 0 * size + cur].dest_device = txn[start + 0 * size + cur].subtxn.dest_device;
        txn[start + 0 * size + cur].global_txn_info_ID = 0;

        txn[start + 1 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 1 * size + cur].subtxn.ispopular = 1;
        txn[start + 1 * size + cur].subtxn.type = 0;
        txn[start + 1 * size + cur].subtxn.table_ID = 1;
        txn[start + 1 * size + cur].subtxn.benchmark = 2;
        txn[start + 1 * size + cur].subtxn.dest_Row_1 =
                query[cur].W_ID * 10 + query[cur].D_ID;
        txn[start + 1 * size + cur].subtxn.dest_device =
                (query[cur].W_ID * 10 + query[cur].D_ID) / meta[1].table_slice_size;
        txn[start + 1 * size + cur].dest_device = txn[start + 1 * size + cur].subtxn.dest_device;
        txn[start + 1 * size + cur].global_txn_info_ID = 1;

        txn[start + 2 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 2 * size + cur].subtxn.ispopular = 0;
        txn[start + 2 * size + cur].subtxn.type = 0;
        txn[start + 2 * size + cur].subtxn.table_ID = 2;
        txn[start + 2 * size + cur].subtxn.benchmark = 2;
        txn[start + 2 * size + cur].subtxn.dest_Row_1 =
                (query[cur].W_ID * 10 + query[cur].D_ID) * 3000 + query[cur].C_ID;
        txn[start + 2 * size + cur].subtxn.dest_device =
                ((query[cur].W_ID * 10 + query[cur].D_ID) * 3000 + query[cur].C_ID) / meta[2].table_slice_size;
        txn[start + 2 * size + cur].dest_device = txn[start + 2 * size + cur].subtxn.dest_device;
        txn[start + 2 * size + cur].global_txn_info_ID = 2;

        txn[start + 3 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 3 * size + cur].subtxn.ispopular = 0;
        txn[start + 3 * size + cur].subtxn.type = 1;
        txn[start + 3 * size + cur].subtxn.table_ID = 3;
        txn[start + 3 * size + cur].subtxn.benchmark = 2;
        txn[start + 3 * size + cur].subtxn.dest_Row_1 =
                (query[cur].W_ID * 10 + query[cur].D_ID) * 3000 + query[cur].N_O_ID;
        txn[start + 3 * size + cur].subtxn.dest_device =
                ((query[cur].W_ID * 10 + query[cur].D_ID) * 3000 + query[cur].N_O_ID) / meta[3].table_slice_size;
        txn[start + 3 * size + cur].dest_device = txn[start + 3 * size + cur].subtxn.dest_device;
        txn[start + 3 * size + cur].global_txn_info_ID = 3;

        txn[start + 4 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 4 * size + cur].subtxn.ispopular = 0;
        txn[start + 4 * size + cur].subtxn.type = 1;
        txn[start + 4 * size + cur].subtxn.table_ID = 5;
        txn[start + 4 * size + cur].subtxn.benchmark = 2;
        txn[start + 4 * size + cur].subtxn.dest_Row_1 =
                (query[cur].W_ID * 10 + query[cur].D_ID) * 3000 + query[cur].O_ID;
        txn[start + 4 * size + cur].subtxn.dest_device =
                ((query[cur].W_ID * 10 + query[cur].D_ID) * 3000 + query[cur].O_ID) / meta[5].table_slice_size;
        txn[start + 4 * size + cur].dest_device = txn[start + 4 * size + cur].subtxn.dest_device;
        txn[start + 4 * size + cur].global_txn_info_ID = 4;

        for (uint32_t i = 0; i < query[cur].O_OL_CNT; ++i) {
            txn[start + (5 + 3 * i) * size + cur].subtxn.TID = query[cur].TID;
            txn[start + (5 + 3 * i) * size + cur].subtxn.ispopular = 0;
            txn[start + (5 + 3 * i) * size + cur].subtxn.type = 1;
            txn[start + (5 + 3 * i) * size + cur].subtxn.table_ID = 6;
            txn[start + (5 + 3 * i) * size + cur].subtxn.benchmark = 2;
            txn[start + (5 + 3 * i) * size + cur].subtxn.dest_Row_1 =
                    (query[cur].W_ID * 10 + query[cur].D_ID) * 45000 + query[cur].O_OL_ID;
            txn[start + (5 + 3 * i) * size + cur].subtxn.dest_device =
                    ((query[cur].W_ID * 10 + query[cur].D_ID) * 45000 + query[cur].O_OL_ID) / meta[6].table_slice_size;
            txn[start + (5 + 3 * i) * size + cur].dest_device = txn[start + (5 + 3 * i) * size + cur].subtxn.
                    dest_device;
            txn[start + (5 + 3 * i) * size + cur].global_txn_info_ID = 5;

            txn[start + (6 + 3 * i) * size + cur].subtxn.TID = query[cur].TID;
            txn[start + (6 + 3 * i) * size + cur].subtxn.ispopular = 0;
            txn[start + (6 + 3 * i) * size + cur].subtxn.type = 2;
            txn[start + (6 + 3 * i) * size + cur].subtxn.table_ID = 7;
            txn[start + (6 + 3 * i) * size + cur].subtxn.benchmark = 2;
            txn[start + (6 + 3 * i) * size + cur].subtxn.dest_Row_1 =
                    query[cur].INFO[i].OL_SUPPLY_W_ID * 1000000 + query[cur].INFO[i].OL_I_ID;
            txn[start + (6 + 3 * i) * size + cur].subtxn.dest_device =
                    (query[cur].INFO[i].OL_SUPPLY_W_ID * 1000000 + query[cur].INFO[i].OL_I_ID) / meta[7].
                    table_slice_size;
            txn[start + (6 + 3 * i) * size + cur].dest_device = txn[start + (6 + 3 * i) * size + cur].subtxn.
                    dest_device;
            txn[start + (6 + 3 * i) * size + cur].global_txn_info_ID = 6;

            txn[start + (7 + 3 * i) * size + cur].subtxn.TID = query[cur].TID;
            txn[start + (7 + 3 * i) * size + cur].subtxn.ispopular = 0;
            txn[start + (7 + 3 * i) * size + cur].subtxn.type = 2;
            txn[start + (7 + 3 * i) * size + cur].subtxn.table_ID = 7;
            txn[start + (7 + 3 * i) * size + cur].subtxn.benchmark = 2;
            txn[start + (7 + 3 * i) * size + cur].subtxn.dest_Row_1 =
                    query[cur].INFO[i].OL_SUPPLY_W_ID * 1000000 + query[cur].INFO[i].OL_I_ID;
            txn[start + (7 + 3 * i) * size + cur].subtxn.dest_device =
                    (query[cur].INFO[i].OL_SUPPLY_W_ID * 1000000 + query[cur].INFO[i].OL_I_ID) / meta[7].
                    table_slice_size;
            txn[start + (7 + 3 * i) * size + cur].dest_device = txn[start + (7 + 3 * i) * size + cur].subtxn.
                    dest_device;
            txn[start + (7 + 3 * i) * size + cur].global_txn_info_ID = 7;
        }
        // printf(
        //     "cur:%d,ID:%d %d %d %d %d"
        //     ",TID:%d,%d %d %d %d %d"
        //     ",Row:%d %d %d %d %d %d"
        //     ",dest_device:%d %d %d %d %d"
        //     // ",ispopular:%d %d %d %d %d"
        //     "\n",
        //     cur, start + 0 * size + cur, start + 1 * size + cur,
        //     start + 2 * size + cur, start + 3 * size + cur, start + 3 * size + cur,
        //     txn[start + 0 * size + cur].subtxn.TID,
        //     txn[start + 0 * size + cur].subtxn.type,
        //     txn[start + 1 * size + cur].subtxn.type, txn[start + 2 * size + cur].subtxn.type,
        //     txn[start + 3 * size + cur].subtxn.type, txn[start + 4 * size + cur].subtxn.type,
        //     query[cur].W_ID, query[cur].D_ID, query[cur].C_ID, query[cur].O_ID, query[cur].N_O_ID, query[cur].O_OL_CNT,
        //     txn[start + 0 * size + cur].dest_device, txn[start + 1 * size + cur].dest_device,
        //     txn[start + 2 * size + cur].dest_device, txn[start + 3 * size + cur].dest_device,
        //     txn[start + 4 * size + cur].dest_device
        //     // ,
        //     // txn[start + 0 * size + cur].subtxn.ispopular, txn[start + 1 * size + cur].subtxn.ispopular,
        //     // txn[start + 2 * size + cur].subtxn.ispopular, txn[start + 3 * size + cur].subtxn.ispopular,
        //     // txn[start + 3 * size + cur].subtxn[1].ispopular
        // );
        cur += thSize;
    }
}

template<>
__global__ void parse<Payment_Query>(uint32_t size, uint32_t start, Global_Txn_Info *txn_info, Payment_Query *query,
                                     Global_Txn *txn, Global_Txn_Exec_Param *param, Global_Table_Meta *meta,
                                     Global_Table_Index *index) {
    uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t thSize = blockDim.x * gridDim.x;
    uint32_t cur = thID;
    while (cur < size) {
        txn[start + 0 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 0 * size + cur].subtxn.ispopular = 1;
        txn[start + 0 * size + cur].subtxn.type = 0;
        txn[start + 0 * size + cur].subtxn.table_ID = 0;
        txn[start + 0 * size + cur].subtxn.benchmark = 2;
        txn[start + 0 * size + cur].subtxn.dest_Row_1 = query[cur].W_ID;
        txn[start + 0 * size + cur].subtxn.dest_device = query[cur].W_ID / meta[0].table_slice_size;
        txn[start + 0 * size + cur].dest_device = txn[start + 0 * size + cur].subtxn.dest_device;
        txn[start + 0 * size + cur].global_txn_info_ID = 8;

        txn[start + 1 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 1 * size + cur].subtxn.ispopular = 1;
        txn[start + 1 * size + cur].subtxn.type = 0;
        txn[start + 1 * size + cur].subtxn.table_ID = 1;
        txn[start + 1 * size + cur].subtxn.benchmark = 2;
        txn[start + 1 * size + cur].subtxn.dest_Row_1 = query[cur].W_ID * 10 + query[cur].D_ID;
        txn[start + 1 * size + cur].subtxn.dest_device =
                (query[cur].W_ID * 10 + query[cur].D_ID) / meta[1].table_slice_size;
        txn[start + 1 * size + cur].dest_device = txn[start + 1 * size + cur].subtxn.dest_device;
        txn[start + 1 * size + cur].global_txn_info_ID = 9;

        txn[start + 2 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 2 * size + cur].subtxn.ispopular = 0;
        txn[start + 2 * size + cur].subtxn.type = 0;
        txn[start + 2 * size + cur].subtxn.table_ID = 2;
        txn[start + 2 * size + cur].subtxn.benchmark = 2;
        if (query[cur].isName == 1) {
            uint32_t C_ID = 0;
            for (uint32_t i = 0; i < 3000; i++) {
                if (index[2].index[(query[cur].W_ID * 10 + query[cur].D_ID) * 3000 + i] == query[cur].C_LAST) {
                    C_ID = i;
                    break;
                }
            }
            query[cur].C_ID = C_ID;
        }
        txn[start + 2 * size + cur].subtxn.dest_Row_1 =
                (query[cur].C_W_ID * 10 + query[cur].C_D_ID) * 3000 + query[cur].C_ID;
        txn[start + 2 * size + cur].subtxn.dest_device =
                ((query[cur].C_W_ID * 10 + query[cur].C_D_ID) * 3000 + query[cur].C_ID) / meta[2].table_slice_size;
        txn[start + 2 * size + cur].dest_device = txn[start + 2 * size + cur].subtxn.dest_device;
        txn[start + 2 * size + cur].global_txn_info_ID = 10;

        txn[start + 3 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 3 * size + cur].subtxn.ispopular = 0;
        txn[start + 3 * size + cur].subtxn.type = 2;
        txn[start + 3 * size + cur].subtxn.table_ID = 2;
        txn[start + 3 * size + cur].subtxn.benchmark = 2;
        txn[start + 3 * size + cur].subtxn.dest_Row_1 =
                (query[cur].C_W_ID * 10 + query[cur].C_D_ID) * 3000 + query[cur].C_ID;
        txn[start + 3 * size + cur].subtxn.dest_device =
                ((query[cur].C_W_ID * 10 + query[cur].C_D_ID) * 3000 + query[cur].C_ID) / meta[2].table_slice_size;
        txn[start + 3 * size + cur].dest_device = txn[start + 3 * size + cur].subtxn.dest_device;
        txn[start + 3 * size + cur].global_txn_info_ID = 11;

        txn[start + 4 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 4 * size + cur].subtxn.ispopular = 1;
        txn[start + 4 * size + cur].subtxn.type = 2;
        txn[start + 4 * size + cur].subtxn.table_ID = 0;
        txn[start + 4 * size + cur].subtxn.benchmark = 2;
        txn[start + 4 * size + cur].subtxn.dest_Row_1 = query[cur].W_ID;
        txn[start + 4 * size + cur].subtxn.dest_device = query[cur].W_ID / meta[0].table_slice_size;
        txn[start + 4 * size + cur].dest_device = txn[start + 4 * size + cur].subtxn.dest_device;
        txn[start + 4 * size + cur].global_txn_info_ID = 12;

        txn[start + 5 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 5 * size + cur].subtxn.ispopular = 1;
        txn[start + 5 * size + cur].subtxn.type = 2;
        txn[start + 5 * size + cur].subtxn.table_ID = 1;
        txn[start + 5 * size + cur].subtxn.benchmark = 2;
        txn[start + 5 * size + cur].subtxn.dest_Row_1 =
                query[cur].W_ID * 10 + query[cur].D_ID;
        txn[start + 5 * size + cur].subtxn.dest_device =
                (query[cur].W_ID * 10 + query[cur].D_ID) / meta[1].table_slice_size;
        txn[start + 5 * size + cur].dest_device = txn[start + 5 * size + cur].subtxn.dest_device;
        txn[start + 5 * size + cur].global_txn_info_ID = 13;

        txn[start + 6 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 6 * size + cur].subtxn.ispopular = 0;
        txn[start + 6 * size + cur].subtxn.type = 1;
        txn[start + 6 * size + cur].subtxn.table_ID = 4;
        txn[start + 6 * size + cur].subtxn.benchmark = 2;
        txn[start + 6 * size + cur].subtxn.dest_Row_1 =
                (query[cur].W_ID * 10 + query[cur].D_ID) * 3000 + query[cur].H_ID;
        txn[start + 6 * size + cur].subtxn.dest_device =
                ((query[cur].W_ID * 10 + query[cur].D_ID) * 3000 + query[cur].H_ID) / meta[4].table_slice_size;
        txn[start + 6 * size + cur].dest_device = txn[start + 6 * size + cur].subtxn.dest_device;
        txn[start + 6 * size + cur].global_txn_info_ID = 14;

        // printf(
        //     "cur:%d,ID:%d %d %d %d %d %d %d"
        //     ",TID:%d"
        //     ",type:%d %d %d %d %d"
        //     // ",Row:%d %d %d %d %d %d"
        //     // ",dest_device:%d %d %d %d %d"
        //     ",ispopular:%d %d %d %d %d %d %d"
        //     "\n",
        //     cur, start + 0 * size + cur, start + 1 * size + cur,
        //     start + 2 * size + cur, start + 3 * size + cur,
        //     start + 4 * size + cur, start + 5 * size + cur, start + 6 * size + cur,
        //     txn[start + 0 * size + cur].subtxn.TID,
        //     txn[start + 0 * size + cur].subtxn.type,
        //     txn[start + 1 * size + cur].subtxn.type, txn[start + 2 * size + cur].subtxn.type,
        //     txn[start + 3 * size + cur].subtxn.type, txn[start + 4 * size + cur].subtxn.type,
        //     // query[cur].W_ID, query[cur].D_ID, query[cur].isName, query[cur].C_ID, query[cur].C_LAST,
        //     // query[cur].H_AMOUNT,
        //     // txn[start + 0 * size + cur].dest_device, txn[start + 1 * size + cur].dest_device,
        //     // txn[start + 2 * size + cur].dest_device, txn[start + 3 * size + cur].dest_device,
        //     // txn[start + 4 * size + cur].dest_device
        //     // ,
        //     txn[start + 0 * size + cur].subtxn.ispopular, txn[start + 1 * size + cur].subtxn.ispopular,
        //     txn[start + 2 * size + cur].subtxn.ispopular, txn[start + 3 * size + cur].subtxn.ispopular,
        //     txn[start + 4 * size + cur].subtxn.ispopular, txn[start + 5 * size + cur].subtxn.ispopular,
        //     txn[start + 6 * size + cur].subtxn.ispopular
        // );

        cur += thSize;
    }
}

template<>
__global__ void parse<Orderstatus_Query>(uint32_t size, uint32_t start, Global_Txn_Info *txn_info,
                                         Orderstatus_Query *query, Global_Txn *txn, Global_Txn_Exec_Param *param,
                                         Global_Table_Meta *meta, Global_Table_Index *index) {
    uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t thSize = blockDim.x * gridDim.x;
    uint32_t cur = thID;
    while (cur < size) {
        txn[start + 0 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 0 * size + cur].subtxn.ispopular = 0;
        txn[start + 0 * size + cur].subtxn.type = 0;
        txn[start + 0 * size + cur].subtxn.table_ID = 0;
        txn[start + 0 * size + cur].subtxn.benchmark = 2;
        if (query[cur].isName == 1) {
            uint32_t C_ID = 0;
            for (uint32_t i = 0; i < 3000; i++) {
                if (index[2].index[(query[cur].W_ID * 10 + query[cur].D_ID) * 3000 + i] == query[cur].C_LAST) {
                    C_ID = i;
                    break;
                }
            }
            query[cur].C_ID = C_ID;
        }
        txn[start + 0 * size + cur].subtxn.dest_Row_1 =
                (query[cur].W_ID * 10 + query[cur].D_ID) * 3000 + query[cur].C_ID;
        txn[start + 0 * size + cur].subtxn.dest_device =
                ((query[cur].W_ID * 10 + query[cur].D_ID) * 3000 + query[cur].C_ID) / meta[0].table_slice_size;
        txn[start + 0 * size + cur].dest_device = txn[start + 0 * size + cur].subtxn.dest_device;
        txn[start + 0 * size + cur].global_txn_info_ID = 15;

        txn[start + 1 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 1 * size + cur].subtxn.ispopular = 0;
        txn[start + 1 * size + cur].subtxn.type = 0;
        txn[start + 1 * size + cur].subtxn.table_ID = 5;
        txn[start + 1 * size + cur].subtxn.benchmark = 2;
        txn[start + 1 * size + cur].subtxn.dest_Row_1 =
                (query[cur].W_ID * 10 + query[cur].D_ID) * 3000 + query[cur].O_ID;
        txn[start + 1 * size + cur].subtxn.dest_device =
                ((query[cur].W_ID * 10 + query[cur].D_ID) * 3000 + query[cur].O_ID) / meta[5].table_slice_size;
        txn[start + 1 * size + cur].dest_device = txn[start + 1 * size + cur].subtxn.dest_device;
        txn[start + 1 * size + cur].global_txn_info_ID = 16;

        txn[start + 2 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 2 * size + cur].subtxn.ispopular = 0;
        txn[start + 2 * size + cur].subtxn.type = 0;
        txn[start + 2 * size + cur].subtxn.table_ID = 6;
        txn[start + 2 * size + cur].subtxn.benchmark = 2;
        txn[start + 2 * size + cur].subtxn.dest_Row_1 =
                (query[cur].W_ID * 10 + query[cur].D_ID) * 45000 + query[cur].OL_ID;
        txn[start + 2 * size + cur].subtxn.dest_device =
                ((query[cur].W_ID * 10 + query[cur].D_ID) * 45000 + query[cur].OL_ID) / meta[6].table_slice_size;
        txn[start + 2 * size + cur].dest_device = txn[start + 2 * size + cur].subtxn.dest_device;
        txn[start + 2 * size + cur].global_txn_info_ID = 17;

        cur += thSize;
    }
}

template<>
__global__ void parse<Delivery_Query>(uint32_t size, uint32_t start, Global_Txn_Info *txn_info, Delivery_Query *query,
                                      Global_Txn *txn, Global_Txn_Exec_Param *param, Global_Table_Meta *meta,
                                      Global_Table_Index *index) {
    uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t thSize = blockDim.x * gridDim.x;
    uint32_t cur = thID;
    while (cur < size) {
        for (uint32_t i = 0; i < 10; ++i) {
            txn[start + (0 + i * 5) * size + cur].subtxn.TID = query[cur].TID;
            txn[start + (0 + i * 5) * size + cur].subtxn.ispopular = 0;
            txn[start + (0 + i * 5) * size + cur].subtxn.type = 3;
            txn[start + (0 + i * 5) * size + cur].subtxn.table_ID = 3;
            txn[start + (0 + i * 5) * size + cur].subtxn.benchmark = 2;
            txn[start + (0 + i * 5) * size + cur].subtxn.dest_Row_1 =
                    (query[cur].NO_W_ID[i] * 10 + query[cur].NO_D_ID[i]) * 3000 + query[cur].NO_O_ID[i];
            txn[start + (0 + i * 5) * size + cur].subtxn.dest_device =
                    ((query[cur].NO_W_ID[i] * 10 + query[cur].NO_D_ID[i]) * 3000 + query[cur].NO_O_ID[i]) / meta[3].
                    table_slice_size;
            txn[start + (0 + i * 5) * size + cur].dest_device = txn[start + (0 + i * 5) * size + cur].subtxn.
                    dest_device;
            txn[start + (0 + i * 5) * size + cur].global_txn_info_ID = 18;

            txn[start + (1 + i * 5) * size + cur].subtxn.TID = query[cur].TID;
            txn[start + (1 + i * 5) * size + cur].subtxn.ispopular = 0;
            txn[start + (1 + i * 5) * size + cur].subtxn.type = 2;
            txn[start + (1 + i * 5) * size + cur].subtxn.table_ID = 5;
            txn[start + (1 + i * 5) * size + cur].subtxn.benchmark = 2;
            txn[start + (1 + i * 5) * size + cur].subtxn.dest_Row_1 =
                    (query[cur].NO_W_ID[i] * 10 + query[cur].NO_D_ID[i]) * 3000 + query[cur].NO_O_ID[i];
            txn[start + (1 + i * 5) * size + cur].subtxn.dest_device =
                    ((query[cur].NO_W_ID[i] * 10 + query[cur].NO_D_ID[i]) * 3000 + query[cur].NO_O_ID[i]) / meta[5].
                    table_slice_size;
            txn[start + (1 + i * 5) * size + cur].dest_device = txn[start + (1 + i * 5) * size + cur].subtxn.
                    dest_device;
            txn[start + (1 + i * 5) * size + cur].global_txn_info_ID = 19;

            txn[start + (2 + i * 5) * size + cur].subtxn.TID = query[cur].TID;
            txn[start + (2 + i * 5) * size + cur].subtxn.ispopular = 0;
            txn[start + (2 + i * 5) * size + cur].subtxn.type = 2;
            txn[start + (2 + i * 5) * size + cur].subtxn.table_ID = 6;
            txn[start + (2 + i * 5) * size + cur].subtxn.benchmark = 2;
            txn[start + (2 + i * 5) * size + cur].subtxn.dest_Row_1 =
                    (query[cur].NO_W_ID[i] * 10 + query[cur].NO_D_ID[i]) * 3000 + query[cur].NO_O_ID[i];
            txn[start + (2 + i * 5) * size + cur].subtxn.dest_device =
                    ((query[cur].NO_W_ID[i] * 10 + query[cur].NO_D_ID[i]) * 3000 + query[cur].NO_O_ID[i]) / meta[6].
                    table_slice_size;
            txn[start + (2 + i * 5) * size + cur].dest_device = txn[start + (2 + i * 5) * size + cur].subtxn.
                    dest_device;
            txn[start + (2 + i * 5) * size + cur].global_txn_info_ID = 20;

            txn[start + (3 + i * 5) * size + cur].subtxn.TID = query[cur].TID;
            txn[start + (3 + i * 5) * size + cur].subtxn.ispopular = 0;
            txn[start + (3 + i * 5) * size + cur].subtxn.type = 0;
            txn[start + (3 + i * 5) * size + cur].subtxn.table_ID = 6;
            txn[start + (3 + i * 5) * size + cur].subtxn.benchmark = 2;
            txn[start + (3 + i * 5) * size + cur].subtxn.dest_Row_1 =
                    (query[cur].NO_W_ID[i] * 10 + query[cur].NO_D_ID[i]) * 3000 + query[cur].NO_O_ID[i];
            txn[start + (3 + i * 5) * size + cur].subtxn.dest_device =
                    ((query[cur].NO_W_ID[i] * 10 + query[cur].NO_D_ID[i]) * 3000 + query[cur].NO_O_ID[i]) / meta[6].
                    table_slice_size;
            txn[start + (3 + i * 5) * size + cur].dest_device = txn[start + (3 + i * 5) * size + cur].subtxn.
                    dest_device;
            txn[start + (3 + i * 5) * size + cur].global_txn_info_ID = 21;

            txn[start + (4 + i * 5) * size + cur].subtxn.TID = query[cur].TID;
            txn[start + (4 + i * 5) * size + cur].subtxn.ispopular = 0;
            txn[start + (4 + i * 5) * size + cur].subtxn.type = 2;
            txn[start + (4 + i * 5) * size + cur].subtxn.table_ID = 2;
            txn[start + (4 + i * 5) * size + cur].subtxn.benchmark = 2;
            txn[start + (4 + i * 5) * size + cur].subtxn.dest_Row_1 =
                    (query[cur].NO_W_ID[i] * 10 + query[cur].NO_D_ID[i]) * 3000 + query[cur].NO_C_ID[i];
            txn[start + (4 + i * 5) * size + cur].subtxn.dest_device =
                    ((query[cur].NO_W_ID[i] * 10 + query[cur].NO_D_ID[i]) * 3000 + query[cur].NO_C_ID[i]) / meta[2].
                    table_slice_size;
            txn[start + (4 + i * 5) * size + cur].dest_device = txn[start + (4 + i * 5) * size + cur].subtxn.
                    dest_device;
            txn[start + (4 + i * 5) * size + cur].global_txn_info_ID = 22;
        }

        cur += thSize;
    }
}

template<>
__global__ void parse<Stocklevel_Query>(uint32_t size, uint32_t start, Global_Txn_Info *txn_info,
                                        Stocklevel_Query *query, Global_Txn *txn, Global_Txn_Exec_Param *param,
                                        Global_Table_Meta *meta, Global_Table_Index *index) {
    uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t thSize = blockDim.x * gridDim.x;
    uint32_t cur = thID;
    while (cur < size) {
        txn[start + 0 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 0 * size + cur].subtxn.ispopular = 0;
        txn[start + 0 * size + cur].subtxn.type = 0;
        txn[start + 0 * size + cur].subtxn.table_ID = 1;
        txn[start + 0 * size + cur].subtxn.benchmark = 2;
        txn[start + 0 * size + cur].subtxn.dest_Row_1 =
                query[cur].W_ID * 10 + query[cur].D_ID;
        txn[start + 0 * size + cur].subtxn.dest_device =
                (query[cur].W_ID * 10 + query[cur].D_ID) / meta[1].table_slice_size;
        txn[start + 0 * size + cur].dest_device = txn[start + 0 * size + cur].subtxn.dest_device;
        txn[start + 0 * size + cur].global_txn_info_ID = 23;

        for (uint32_t i = 0; i < 10; ++i) {
            txn[start + (1 + i * 2) * size + cur].subtxn.TID = query[cur].TID;
            txn[start + (1 + i * 2) * size + cur].subtxn.ispopular = 0;
            txn[start + (1 + i * 2) * size + cur].subtxn.type = 0;
            txn[start + (1 + i * 2) * size + cur].subtxn.table_ID = 6;
            txn[start + (1 + i * 2) * size + cur].subtxn.benchmark = 2;
            txn[start + (1 + i * 2) * size + cur].subtxn.dest_Row_1 =
                    (query[cur].W_ID * 10 + query[cur].D_ID) * 45000 + query[cur].O_OL_ID[i];
            txn[start + (1 + i * 2) * size + cur].subtxn.dest_device =
                    ((query[cur].W_ID * 10 + query[cur].D_ID) * 45000 + query[cur].O_OL_ID[i]) / meta[6].
                    table_slice_size;
            txn[start + (1 + i * 2) * size + cur].dest_device = txn[start + (1 + i * 2) * size + cur].subtxn.
                    dest_device;
            txn[start + (1 + i * 5) * size + cur].global_txn_info_ID = 24;

            txn[start + (2 + i * 2) * size + cur].subtxn.TID = query[cur].TID;
            txn[start + (2 + i * 2) * size + cur].subtxn.ispopular = 0;
            txn[start + (2 + i * 2) * size + cur].subtxn.type = 0;
            txn[start + (2 + i * 2) * size + cur].subtxn.table_ID = 7;
            txn[start + (2 + i * 2) * size + cur].subtxn.benchmark = 2;
            txn[start + (2 + i * 2) * size + cur].subtxn.dest_Row_1 =
                    query[cur].W_ID * 100000 + query[cur].I_ID[i];
            txn[start + (2 + i * 2) * size + cur].subtxn.dest_device =
                    (query[cur].W_ID * 100000 + query[cur].I_ID[i]) / meta[7].table_slice_size;
            txn[start + (2 + i * 2) * size + cur].dest_device = txn[start + (2 + i * 2) * size + cur].subtxn.
                    dest_device;
            txn[start + (2 + i * 5) * size + cur].global_txn_info_ID = 25;
        }
        cur += thSize;
    }
}

template<>
__global__ void parse<YCSB_A_Query>(uint32_t size, uint32_t start, Global_Txn_Info *txn_info, YCSB_A_Query *query,
                                    Global_Txn *txn, Global_Txn_Exec_Param *param, Global_Table_Meta *meta,
                                    Global_Table_Index *index) {
    // 50% Read, 50% Write
    uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t thSize = blockDim.x * gridDim.x;
    uint32_t cur = thID;
    while (cur < size) {
#pragma unroll
        for (uint32_t i = 0; i < 5; ++i) {
            uint32_t ispopular = 0;
            if (query[cur].ROW_ID[i] < 100) {
                ispopular = 1;
            } else {
                ispopular = 0;
            }
            txn[start + i * size + cur].subtxn.TID = query[cur].TID;
            txn[start + i * size + cur].subtxn.ispopular = ispopular;
            txn[start + i * size + cur].subtxn.type = 0;
            txn[start + i * size + cur].subtxn.table_ID = 0;
            txn[start + i * size + cur].subtxn.benchmark = 4;
            txn[start + i * size + cur].subtxn.dest_Row_1 = query[cur].ROW_ID[i];
            txn[start + i * size + cur].subtxn.dest_device = query[cur].ROW_ID[i] / meta[0].table_slice_size;
            txn[start + i * size + cur].subtxn.ispopular = 0;
            txn[start + i * size + cur].dest_device = txn[start + i * size + cur].subtxn.dest_device;
            txn[start + i * size + cur].global_txn_info_ID = i;
        }
        for (uint32_t i = 5; i < 10; ++i) {
            uint32_t ispopular = 0;
            if (query[cur].ROW_ID[i] < 100) {
                ispopular = 1;
            } else {
                ispopular = 0;
            }
            txn[start + i * size + cur].subtxn.TID = query[cur].TID;
            txn[start + i * size + cur].subtxn.ispopular = ispopular;
            txn[start + i * size + cur].subtxn.type = 2;
            txn[start + i * size + cur].subtxn.table_ID = 0;
            txn[start + i * size + cur].subtxn.benchmark = 4;
            txn[start + i * size + cur].subtxn.dest_Row_1 = query[cur].ROW_ID[i];
            txn[start + i * size + cur].subtxn.dest_device = query[cur].ROW_ID[i] / meta[0].table_slice_size;
            txn[start + i * size + cur].subtxn.ispopular = 0;
            txn[start + i * size + cur].dest_device = txn[start + i * size + cur].subtxn.dest_device;
            txn[start + i * size + cur].global_txn_info_ID = i;
        }
        // printf(
        //     "cur:%d,"
        //     // "ID:%d %d %d %d %d,"
        //     "TID:%d,%d %d %d %d %d %d %d %d %d %d,"
        //     // "Row:%d %d %d %d %d %d %d %d %d %d,"
        //     "dest_device:%d %d %d %d %d %d %d %d %d %d\n"
        //     // "ispopular:%d %d %d %d %d %d %d %d %d %d\n"
        //     ,
        //     cur,
        //     // start + 0 * size + cur, start + 1 * size + cur,
        //     // start + 2 * size + cur, start + 3 * size + cur,
        //     // start + 4 * size + cur,
        //     txn[start + 0 * size + cur].subtxn.TID,
        //     txn[start + 0 * size + cur].subtxn.type, txn[start + 1 * size + cur].subtxn.type,
        //     txn[start + 2 * size + cur].subtxn.type, txn[start + 3 * size + cur].subtxn.type,
        //     txn[start + 4 * size + cur].subtxn.type, txn[start + 5 * size + cur].subtxn.type,
        //     txn[start + 6 * size + cur].subtxn.type, txn[start + 7 * size + cur].subtxn.type,
        //     txn[start + 8 * size + cur].subtxn.type, txn[start + 9 * size + cur].subtxn.type,
        //     // txn[start + 0 * size + cur].subtxn.dest_Row_1, txn[start + 1 * size + cur].subtxn.dest_Row_1,
        //     // txn[start + 2 * size + cur].subtxn.dest_Row_1, txn[start + 3 * size + cur].subtxn.dest_Row_1,
        //     // txn[start + 4 * size + cur].subtxn.dest_Row_1, txn[start + 5 * size + cur].subtxn.dest_Row_1,
        //     // txn[start + 6 * size + cur].subtxn.dest_Row_1, txn[start + 7 * size + cur].subtxn.dest_Row_1,
        //     // txn[start + 8 * size + cur].subtxn.dest_Row_1, txn[start + 9 * size + cur].subtxn.dest_Row_1,
        //     txn[start + 0 * size + cur].dest_device, txn[start + 1 * size + cur].dest_device,
        //     txn[start + 2 * size + cur].dest_device, txn[start + 3 * size + cur].dest_device,
        //     txn[start + 4 * size + cur].dest_device, txn[start + 5 * size + cur].dest_device,
        //     txn[start + 6 * size + cur].dest_device, txn[start + 7 * size + cur].dest_device,
        //     txn[start + 8 * size + cur].dest_device, txn[start + 9 * size + cur].dest_device);
        // // txn[start + 0 * size + cur].subtxn.ispopular, txn[start + 1 * size + cur].subtxn.ispopular,
        // // txn[start + 2 * size + cur].subtxn.ispopular, txn[start + 3 * size + cur].subtxn.ispopular,
        // // txn[start + 4 * size + cur].subtxn.ispopular, txn[start + 5 * size + cur].subtxn.ispopular,
        // // txn[start + 6 * size + cur].subtxn.ispopular, txn[start + 7 * size + cur].subtxn.ispopular,
        // // txn[start + 8 * size + cur].subtxn.ispopular, txn[start + 9 * size + cur].subtxn.ispopular);

        cur += thSize;
    }
}


template<>
__global__ void parse<YCSB_B_Query>(uint32_t size, uint32_t start, Global_Txn_Info *txn_info, YCSB_B_Query *query,
                                    Global_Txn *txn, Global_Txn_Exec_Param *param, Global_Table_Meta *meta,
                                    Global_Table_Index *index) {
    // 90% Read, 10% Write
    uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t thSize = blockDim.x * gridDim.x;
    uint32_t cur = thID;
    while (cur < size) {
        uint32_t ispopular = 0;
        for (uint32_t i = 0; i < 9; ++i) {
            if (query[cur].ROW_ID[i] < 100) {
                ispopular = 1;
            } else {
                ispopular = 0;
            }
            txn[start + i * size + cur].subtxn.TID = query[cur].TID;
            txn[start + i * size + cur].subtxn.ispopular = ispopular;
            txn[start + i * size + cur].subtxn.type = 0;
            txn[start + i * size + cur].subtxn.table_ID = 0;
            txn[start + i * size + cur].subtxn.benchmark = 4;
            txn[start + i * size + cur].subtxn.dest_Row_1 = query[cur].ROW_ID[i];
            txn[start + i * size + cur].subtxn.dest_device = query[cur].ROW_ID[i] / meta[0].table_slice_size;
            txn[start + i * size + cur].subtxn.ispopular = 0;
            txn[start + i * size + cur].dest_device = txn[start + i * size + cur].subtxn.dest_device;
            txn[start + i * size + cur].global_txn_info_ID = i;
        }
        if (query[cur].ROW_ID[9] < 100) {
            ispopular = 1;
        } else {
            ispopular = 0;
        }
        txn[start + 9 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 9 * size + cur].subtxn.ispopular = ispopular;
        txn[start + 9 * size + cur].subtxn.type = 2;
        txn[start + 9 * size + cur].subtxn.table_ID = 0;
        txn[start + 9 * size + cur].subtxn.benchmark = 4;
        txn[start + 9 * size + cur].subtxn.dest_Row_1 = query[cur].ROW_ID[9];
        txn[start + 9 * size + cur].subtxn.dest_device = query[cur].ROW_ID[9] / meta[0].table_slice_size;
        txn[start + 9 * size + cur].dest_device = txn[start + 9 * size + cur].subtxn.dest_device;
        txn[start + 9 * size + cur].global_txn_info_ID = 9;

        // printf(
        //     "cur:%d,"
        //     // "ID:%d %d %d %d %d,"
        //     "TID:%d,%d %d %d %d %d %d %d %d %d %d,"
        //     // "Row:%d %d %d %d %d %d %d %d %d %d,"
        //     "dest_device:%d %d %d %d %d %d %d %d %d %d\n"
        //     // "ispopular:%d %d %d %d %d %d %d %d %d %d\n"
        //     ,
        //     cur,
        //     // start + 0 * size + cur, start + 1 * size + cur,
        //     // start + 2 * size + cur, start + 3 * size + cur,
        //     // start + 4 * size + cur,
        //     txn[start + 0 * size + cur].subtxn.TID,
        //     txn[start + 0 * size + cur].subtxn.type, txn[start + 1 * size + cur].subtxn.type,
        //     txn[start + 2 * size + cur].subtxn.type, txn[start + 3 * size + cur].subtxn.type,
        //     txn[start + 4 * size + cur].subtxn.type, txn[start + 5 * size + cur].subtxn.type,
        //     txn[start + 6 * size + cur].subtxn.type, txn[start + 7 * size + cur].subtxn.type,
        //     txn[start + 8 * size + cur].subtxn.type, txn[start + 9 * size + cur].subtxn.type,
        //     // txn[start + 0 * size + cur].subtxn.dest_Row_1, txn[start + 1 * size + cur].subtxn.dest_Row_1,
        //     // txn[start + 2 * size + cur].subtxn.dest_Row_1, txn[start + 3 * size + cur].subtxn.dest_Row_1,
        //     // txn[start + 4 * size + cur].subtxn.dest_Row_1, txn[start + 5 * size + cur].subtxn.dest_Row_1,
        //     // txn[start + 6 * size + cur].subtxn.dest_Row_1, txn[start + 7 * size + cur].subtxn.dest_Row_1,
        //     // txn[start + 8 * size + cur].subtxn.dest_Row_1, txn[start + 9 * size + cur].subtxn.dest_Row_1,
        //     txn[start + 0 * size + cur].dest_device, txn[start + 1 * size + cur].dest_device,
        //     txn[start + 2 * size + cur].dest_device, txn[start + 3 * size + cur].dest_device,
        //     txn[start + 4 * size + cur].dest_device, txn[start + 5 * size + cur].dest_device,
        //     txn[start + 6 * size + cur].dest_device, txn[start + 7 * size + cur].dest_device,
        //     txn[start + 8 * size + cur].dest_device, txn[start + 9 * size + cur].dest_device);
        // // txn[start + 0 * size + cur].subtxn.ispopular, txn[start + 1 * size + cur].subtxn.ispopular,
        // // txn[start + 2 * size + cur].subtxn.ispopular, txn[start + 3 * size + cur].subtxn.ispopular,
        // // txn[start + 4 * size + cur].subtxn.ispopular, txn[start + 5 * size + cur].subtxn.ispopular,
        // // txn[start + 6 * size + cur].subtxn.ispopular, txn[start + 7 * size + cur].subtxn.ispopular,
        // // txn[start + 8 * size + cur].subtxn.ispopular, txn[start + 9 * size + cur].subtxn.ispopular);

        cur += thSize;
    }
}

template<>
__global__ void parse<YCSB_C_Query>(uint32_t size, uint32_t start, Global_Txn_Info *txn_info, YCSB_C_Query *query,
                                    Global_Txn *txn, Global_Txn_Exec_Param *param, Global_Table_Meta *meta,
                                    Global_Table_Index *index) {
    // 100% Read
    uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t thSize = blockDim.x * gridDim.x;
    uint32_t cur = thID;
    while (cur < size) {
        for (uint32_t i = 0; i < 10; ++i) {
            txn[start + i * size + cur].subtxn.TID = query[cur].TID;
            txn[start + i * size + cur].subtxn.ispopular = 0;
            txn[start + i * size + cur].subtxn.type = 0;
            txn[start + i * size + cur].subtxn.table_ID = 0;
            txn[start + i * size + cur].subtxn.benchmark = 4;
            txn[start + i * size + cur].subtxn.dest_Row_1 = query[cur].ROW_ID[i];
            txn[start + i * size + cur].subtxn.dest_device = query[cur].ROW_ID[i] / meta[0].table_slice_size;
            txn[start + i * size + cur].dest_device = txn[start + i * size + cur].subtxn.dest_device;
            txn[start + i * size + cur].global_txn_info_ID = i;
        }

        // printf(
        //     "cur:%d,"
        //     // "ID:%d %d %d %d %d,"
        //     "TID:%d,%d %d %d %d %d %d %d %d %d %d,"
        //     // "Row:%d %d %d %d %d %d %d %d %d %d,"
        //     "dest_device:%d %d %d %d %d %d %d %d %d %d\n"
        //     // "ispopular:%d %d %d %d %d %d %d %d %d %d\n"
        //     ,
        //     cur,
        //     // start + 0 * size + cur, start + 1 * size + cur,
        //     // start + 2 * size + cur, start + 3 * size + cur,
        //     // start + 4 * size + cur,
        //     txn[start + 0 * size + cur].subtxn.TID,
        //     txn[start + 0 * size + cur].subtxn.type, txn[start + 1 * size + cur].subtxn.type,
        //     txn[start + 2 * size + cur].subtxn.type, txn[start + 3 * size + cur].subtxn.type,
        //     txn[start + 4 * size + cur].subtxn.type, txn[start + 5 * size + cur].subtxn.type,
        //     txn[start + 6 * size + cur].subtxn.type, txn[start + 7 * size + cur].subtxn.type,
        //     txn[start + 8 * size + cur].subtxn.type, txn[start + 9 * size + cur].subtxn.type,
        //     // txn[start + 0 * size + cur].subtxn.dest_Row_1, txn[start + 1 * size + cur].subtxn.dest_Row_1,
        //     // txn[start + 2 * size + cur].subtxn.dest_Row_1, txn[start + 3 * size + cur].subtxn.dest_Row_1,
        //     // txn[start + 4 * size + cur].subtxn.dest_Row_1, txn[start + 5 * size + cur].subtxn.dest_Row_1,
        //     // txn[start + 6 * size + cur].subtxn.dest_Row_1, txn[start + 7 * size + cur].subtxn.dest_Row_1,
        //     // txn[start + 8 * size + cur].subtxn.dest_Row_1, txn[start + 9 * size + cur].subtxn.dest_Row_1,
        //     txn[start + 0 * size + cur].dest_device, txn[start + 1 * size + cur].dest_device,
        //     txn[start + 2 * size + cur].dest_device, txn[start + 3 * size + cur].dest_device,
        //     txn[start + 4 * size + cur].dest_device, txn[start + 5 * size + cur].dest_device,
        //     txn[start + 6 * size + cur].dest_device, txn[start + 7 * size + cur].dest_device,
        //     txn[start + 8 * size + cur].dest_device, txn[start + 9 * size + cur].dest_device);
        // // txn[start + 0 * size + cur].subtxn.ispopular, txn[start + 1 * size + cur].subtxn.ispopular,
        // // txn[start + 2 * size + cur].subtxn.ispopular, txn[start + 3 * size + cur].subtxn.ispopular,
        // // txn[start + 4 * size + cur].subtxn.ispopular, txn[start + 5 * size + cur].subtxn.ispopular,
        // // txn[start + 6 * size + cur].subtxn.ispopular, txn[start + 7 * size + cur].subtxn.ispopular,
        // // txn[start + 8 * size + cur].subtxn.ispopular, txn[start + 9 * size + cur].subtxn.ispopular);

        cur += thSize;
    }
}

template<>
__global__ void parse<YCSB_D_Query>(uint32_t size, uint32_t start, Global_Txn_Info *txn_info, YCSB_D_Query *query,
                                    Global_Txn *txn, Global_Txn_Exec_Param *param, Global_Table_Meta *meta,
                                    Global_Table_Index *index) {
    // 95% Read, 5% Write
    uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t thSize = blockDim.x * gridDim.x;
    uint32_t cur = thID;
    while (cur < size) {
        uint32_t ispopular = 0;

        for (uint32_t i = 0; i < 9; ++i) {
            if (query[cur].ROW_ID[i] < 100) {
                ispopular = 1;
            } else {
                ispopular = 0;
            }
            txn[start + i * size + cur].subtxn.TID = query[cur].TID;
            txn[start + i * size + cur].subtxn.ispopular = ispopular;
            txn[start + i * size + cur].subtxn.type = 0;
            txn[start + i * size + cur].subtxn.table_ID = 0;
            txn[start + i * size + cur].subtxn.benchmark = 4;
            txn[start + i * size + cur].subtxn.dest_Row_1 = query[cur].ROW_ID[i];
            txn[start + i * size + cur].subtxn.dest_device = query[cur].ROW_ID[i] / meta[0].table_slice_size;
            txn[start + i * size + cur].dest_device = txn[start + i * size + cur].subtxn.dest_device;
            txn[start + i * size + cur].global_txn_info_ID = i;
        }
        if (query[cur].ROW_ID[9] < 100) {
            ispopular = 1;
        } else {
            ispopular = 0;
        }
        txn[start + 9 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 9 * size + cur].subtxn.ispopular = ispopular;
        txn[start + 9 * size + cur].subtxn.type = 2;
        txn[start + 9 * size + cur].subtxn.table_ID = 0;
        txn[start + 9 * size + cur].subtxn.benchmark = 4;
        txn[start + 9 * size + cur].subtxn.dest_Row_1 = query[cur].ROW_ID[9];
        txn[start + 9 * size + cur].subtxn.dest_device = query[cur].ROW_ID[9] / meta[0].table_slice_size;
        txn[start + 9 * size + cur].dest_device = txn[start + 9 * size + cur].subtxn.dest_device;
        txn[start + 9 * size + cur].global_txn_info_ID = 9;


        // printf(
        //     "cur:%d,"
        //     // "ID:%d %d %d %d %d,"
        //     "TID:%d,%d %d %d %d %d %d %d %d %d %d,"
        //     // "Row:%d %d %d %d %d %d %d %d %d %d,"
        //     "dest_device:%d %d %d %d %d %d %d %d %d %d\n"
        //     // "ispopular:%d %d %d %d %d %d %d %d %d %d\n"
        //     ,
        //     cur,
        //     // start + 0 * size + cur, start + 1 * size + cur,
        //     // start + 2 * size + cur, start + 3 * size + cur,
        //     // start + 4 * size + cur,
        //     txn[start + 0 * size + cur].subtxn.TID,
        //     txn[start + 0 * size + cur].subtxn.type, txn[start + 1 * size + cur].subtxn.type,
        //     txn[start + 2 * size + cur].subtxn.type, txn[start + 3 * size + cur].subtxn.type,
        //     txn[start + 4 * size + cur].subtxn.type, txn[start + 5 * size + cur].subtxn.type,
        //     txn[start + 6 * size + cur].subtxn.type, txn[start + 7 * size + cur].subtxn.type,
        //     txn[start + 8 * size + cur].subtxn.type, txn[start + 9 * size + cur].subtxn.type,
        //     // txn[start + 0 * size + cur].subtxn.dest_Row_1, txn[start + 1 * size + cur].subtxn.dest_Row_1,
        //     // txn[start + 2 * size + cur].subtxn.dest_Row_1, txn[start + 3 * size + cur].subtxn.dest_Row_1,
        //     // txn[start + 4 * size + cur].subtxn.dest_Row_1, txn[start + 5 * size + cur].subtxn.dest_Row_1,
        //     // txn[start + 6 * size + cur].subtxn.dest_Row_1, txn[start + 7 * size + cur].subtxn.dest_Row_1,
        //     // txn[start + 8 * size + cur].subtxn.dest_Row_1, txn[start + 9 * size + cur].subtxn.dest_Row_1,
        //     txn[start + 0 * size + cur].dest_device, txn[start + 1 * size + cur].dest_device,
        //     txn[start + 2 * size + cur].dest_device, txn[start + 3 * size + cur].dest_device,
        //     txn[start + 4 * size + cur].dest_device, txn[start + 5 * size + cur].dest_device,
        //     txn[start + 6 * size + cur].dest_device, txn[start + 7 * size + cur].dest_device,
        //     txn[start + 8 * size + cur].dest_device, txn[start + 9 * size + cur].dest_device);
        // // txn[start + 0 * size + cur].subtxn.ispopular, txn[start + 1 * size + cur].subtxn.ispopular,
        // // txn[start + 2 * size + cur].subtxn.ispopular, txn[start + 3 * size + cur].subtxn.ispopular,
        // // txn[start + 4 * size + cur].subtxn.ispopular, txn[start + 5 * size + cur].subtxn.ispopular,
        // // txn[start + 6 * size + cur].subtxn.ispopular, txn[start + 7 * size + cur].subtxn.ispopular,
        // // txn[start + 8 * size + cur].subtxn.ispopular, txn[start + 9 * size + cur].subtxn.ispopular);

        cur += thSize;
    }
}

template<>
__global__ void parse<YCSB_E_Query>(uint32_t size, uint32_t start, Global_Txn_Info *txn_info, YCSB_E_Query *query,
                                    Global_Txn *txn, Global_Txn_Exec_Param *param, Global_Table_Meta *meta,
                                    Global_Table_Index *index) {
    // 95% Scan, 5% Insert
    uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t thSize = blockDim.x * gridDim.x;
    uint32_t cur = thID;
    while (cur < size) {
        for (uint32_t i = 0; i < 9; ++i) {
            // scan 20 items
            txn[start + i * size + cur].subtxn.TID = query[cur].TID;
            txn[start + i * size + cur].subtxn.type = 4;
            txn[start + i * size + cur].subtxn.table_ID = 0;
            txn[start + i * size + cur].subtxn.benchmark = 4;
            txn[start + i * size + cur].subtxn.dest_Row_1 = query[cur].ROW_ID[i];
            txn[start + i * size + cur].subtxn.dest_Row_2 = query[cur].ROW_ID[i] + 2;
            txn[start + i * size + cur].subtxn.dest_device = query[cur].ROW_ID[i] / meta[0].table_slice_size;
            txn[start + i * size + cur].subtxn.ispopular = 0;
            txn[start + i * size + cur].dest_device = txn[start + i * size + cur].subtxn.dest_device;
            txn[start + i * size + cur].global_txn_info_ID = i;
        }
        txn[start + 9 * size + cur].subtxn.TID = query[cur].TID;
        txn[start + 9 * size + cur].subtxn.type = 1;
        txn[start + 9 * size + cur].subtxn.table_ID = 0;
        txn[start + 9 * size + cur].subtxn.benchmark = 4;
        txn[start + 9 * size + cur].subtxn.dest_Row_1 = query[cur].ROW_ID[9];
        txn[start + 9 * size + cur].subtxn.dest_device = query[cur].ROW_ID[9] / meta[0].table_slice_size;
        txn[start + 9 * size + cur].subtxn.ispopular = 0;
        txn[start + 9 * size + cur].dest_device = txn[start + 9 * size + cur].subtxn.dest_device;
        txn[start + 9 * size + cur].global_txn_info_ID = 9;


        // printf(
        //     "cur:%d,"
        //     // "ID:%d %d %d %d %d,"
        //     "TID:%d,%d %d %d %d %d %d %d %d %d %d,"
        //     // "Row:%d %d %d %d %d %d %d %d %d %d,"
        //     "dest_device:%d %d %d %d %d %d %d %d %d %d\n"
        //     // "ispopular:%d %d %d %d %d %d %d %d %d %d\n"
        //     ,
        //     cur,
        //     // start + 0 * size + cur, start + 1 * size + cur,
        //     // start + 2 * size + cur, start + 3 * size + cur,
        //     // start + 4 * size + cur,
        //     txn[start + 0 * size + cur].subtxn.TID,
        //     txn[start + 0 * size + cur].subtxn.type, txn[start + 1 * size + cur].subtxn.type,
        //     txn[start + 2 * size + cur].subtxn.type, txn[start + 3 * size + cur].subtxn.type,
        //     txn[start + 4 * size + cur].subtxn.type, txn[start + 5 * size + cur].subtxn.type,
        //     txn[start + 6 * size + cur].subtxn.type, txn[start + 7 * size + cur].subtxn.type,
        //     txn[start + 8 * size + cur].subtxn.type, txn[start + 9 * size + cur].subtxn.type,
        //     // txn[start + 0 * size + cur].subtxn.dest_Row_1, txn[start + 1 * size + cur].subtxn.dest_Row_1,
        //     // txn[start + 2 * size + cur].subtxn.dest_Row_1, txn[start + 3 * size + cur].subtxn.dest_Row_1,
        //     // txn[start + 4 * size + cur].subtxn.dest_Row_1, txn[start + 5 * size + cur].subtxn.dest_Row_1,
        //     // txn[start + 6 * size + cur].subtxn.dest_Row_1, txn[start + 7 * size + cur].subtxn.dest_Row_1,
        //     // txn[start + 8 * size + cur].subtxn.dest_Row_1, txn[start + 9 * size + cur].subtxn.dest_Row_1,
        //     txn[start + 0 * size + cur].dest_device, txn[start + 1 * size + cur].dest_device,
        //     txn[start + 2 * size + cur].dest_device, txn[start + 3 * size + cur].dest_device,
        //     txn[start + 4 * size + cur].dest_device, txn[start + 5 * size + cur].dest_device,
        //     txn[start + 6 * size + cur].dest_device, txn[start + 7 * size + cur].dest_device,
        //     txn[start + 8 * size + cur].dest_device, txn[start + 9 * size + cur].dest_device);
        // // txn[start + 0 * size + cur].subtxn.ispopular, txn[start + 1 * size + cur].subtxn.ispopular,
        // // txn[start + 2 * size + cur].subtxn.ispopular, txn[start + 3 * size + cur].subtxn.ispopular,
        // // txn[start + 4 * size + cur].subtxn.ispopular, txn[start + 5 * size + cur].subtxn.ispopular,
        // // txn[start + 6 * size + cur].subtxn.ispopular, txn[start + 7 * size + cur].subtxn.ispopular,
        // // txn[start + 8 * size + cur].subtxn.ispopular, txn[start + 9 * size + cur].subtxn.ispopular);

        cur += thSize;
    }
}
