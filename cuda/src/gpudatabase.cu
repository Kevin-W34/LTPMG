#include "hip/hip_runtime.h"
#include "../include/gpudatabase.cuh"

GPUdatabase::GPUdatabase(/* args */) {
}

GPUdatabase::~GPUdatabase() {
}

void GPUdatabase::malloc_global_row(std::shared_ptr<Param> param, Global_Table_Info *table_for_gpu_info,
                                    Global_Table *table_for_gpu, Global_Table_Index *index_for_GPU) {
    std::cout << "start gpudatabase.cu GPUdatabase::malloc_global_row()" << std::endl;
    this->table_for_gpu_info = table_for_gpu_info;
    this->table_for_gpu = table_for_gpu;
    // TODO: 此处需要数据划分策略, 并构造数据map以展示每张卡上存了什么数据
    // CHECK(hipHostAlloc((void **)&tables_info_d, sizeof(Global_Table_Info *) * param->device_cnt,
    //     hipHostMallocDefault));
    tables_info_d = new Global_Table_Info *[param->device_cnt];
    // CHECK(hipHostAlloc((void **)&tables_info_h, sizeof(Global_Table_Info) * param->table_cnt,
    //     hipHostMallocDefault));
    tables_info_h = new Global_Table_Info [param->table_cnt];
    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipSetDevice(param->device_IDs[i]));
        CHECK(hipMalloc((void **)&tables_info_d[i], sizeof(Global_Table_Info) * param->table_cnt));
    }

    CHECK(hipHostAlloc((void **)&tables_d, sizeof(Global_Table *) * param->device_cnt, hipHostMallocDefault));
    CHECK(hipHostAlloc((void **)&tables_d_h, sizeof(Global_Table *) * param->device_cnt, hipHostMallocDefault));
    CHECK(hipHostAlloc((void **)&tables_h, sizeof(Global_Table) * param->table_cnt, hipHostMallocDefault));

    for (size_t i = 0; i < param->table_cnt; ++i) {
        // CHECK(hipHostAlloc((void **)&tables_h[i].int_data, sizeof(INT32) * table_for_gpu_info[i].int_size *
        //     table_for_gpu_info[i].table_size, hipHostMallocDefault));
        // CHECK(hipHostAlloc((void **)&tables_h[i].string_data, sizeof(UINT32) * table_for_gpu_info[i].string_size *
        //     table_for_gpu_info[i].table_size * table_for_gpu_info[i].string_length, hipHostMallocDefault));
        // CHECK(hipHostAlloc((void **)&tables_h[i].double_data, sizeof(DOUBLE) * table_for_gpu_info[i].double_size *
        //     table_for_gpu_info[i].table_size, hipHostMallocDefault));
        tables_h[i].int_data = new INT32[table_for_gpu_info[i].int_size * table_for_gpu_info[i].table_size];
        tables_h[i].string_data = new UINT32[table_for_gpu_info[i].string_size *
                                             table_for_gpu_info[i].table_size * table_for_gpu_info[i].string_length];
        tables_h[i].double_data = new DOUBLE[table_for_gpu_info[i].double_size * table_for_gpu_info[i].table_size];
    }

    for (size_t j = 0; j < param->device_cnt; ++j) {
        CHECK(hipSetDevice(param->device_IDs[j]));

        CHECK(hipMalloc((void **)&tables_d[j], sizeof(Global_Table) * param->table_cnt));
        CHECK(hipHostAlloc((void **)&tables_d_h[j], sizeof(Global_Table) * param->table_cnt, hipHostMallocDefault));

        for (size_t i = 0; i < param->table_cnt; ++i) {
            CHECK(hipMalloc((void **)&tables_d_h[j][i].int_data, sizeof(INT32) * table_for_gpu_info[i].int_size *
                table_for_gpu_info[i].table_size/param->device_cnt));
            CHECK(hipMalloc((void **)&tables_d_h[j][i].string_data, sizeof(UINT32) * table_for_gpu_info[i].string_size
                * table_for_gpu_info[i].table_size * table_for_gpu_info[i].string_length/param->device_cnt));
            CHECK(hipMalloc((void **)&tables_d_h[j][i].double_data, sizeof(DOUBLE) * table_for_gpu_info[i].double_size
                * table_for_gpu_info[i].table_size/param->device_cnt));
        }
    }

    CHECK(hipHostAlloc((void **)&strategy_h, sizeof(Global_Table_Strategy) * param->table_cnt, hipHostMallocDefault));
    for (size_t i = 0; i < param->table_cnt; ++i) {
        CHECK(hipHostAlloc((void **)&strategy_h[i].int_target_GPU, sizeof(UINT32) * table_for_gpu_info[i].int_size,
            hipHostMallocDefault));
        CHECK(hipHostAlloc((void **)&strategy_h[i].int_target_GPU_platform, sizeof(UINT32) * table_for_gpu_info[i].
            int_size, hipHostMallocDefault));
        CHECK(hipHostAlloc((void **)&strategy_h[i].string_target_GPU, sizeof(UINT32) * table_for_gpu_info[i].
            string_size, hipHostMallocDefault));
        CHECK(hipHostAlloc((void **)&strategy_h[i].string_target_GPU_platform, sizeof(UINT32) * table_for_gpu_info[i].
            string_size, hipHostMallocDefault));
        CHECK(hipHostAlloc((void **)&strategy_h[i].double_target_GPU, sizeof(UINT32) * table_for_gpu_info[i].
            double_size, hipHostMallocDefault));
        CHECK(hipHostAlloc((void **)&strategy_h[i].double_target_GPU_platform, sizeof(UINT32) * table_for_gpu_info[i].
            double_size, hipHostMallocDefault));
    }

    CHECK(hipHostAlloc((void **)&strategy_d, sizeof(Global_Table_Strategy *) * param->device_cnt, hipHostMallocDefault
    ));
    CHECK(hipHostAlloc((void **)&strategy_d_h, sizeof(Global_Table_Strategy *) * param->device_cnt,
        hipHostMallocDefault));
    for (size_t j = 0; j < param->device_cnt; ++j) {
        CHECK(hipSetDevice(param->device_IDs[j]));
        CHECK(hipMalloc((void **)&strategy_d[j], sizeof(Global_Table_Strategy) * param->table_cnt));
        CHECK(hipHostAlloc((void **)&strategy_d_h[j], sizeof(Global_Table_Strategy) * param->table_cnt,
            hipHostMallocDefault));
        for (size_t i = 0; i < param->table_cnt; ++i) {
            CHECK(hipMalloc((void **)&strategy_d_h[j][i].int_target_GPU, sizeof(UINT32) * table_for_gpu_info[i].
                int_size));
            CHECK(hipMalloc((void **)&strategy_d_h[j][i].int_target_GPU_platform, sizeof(UINT32) * table_for_gpu_info[i
            ].int_size));
            CHECK(hipMalloc((void **)&strategy_d_h[j][i].string_target_GPU, sizeof(UINT32) * table_for_gpu_info[i].
                string_size));
            CHECK(hipMalloc((void **)&strategy_d_h[j][i].string_target_GPU_platform, sizeof(UINT32) *
                table_for_gpu_info[i].string_size));
            CHECK(hipMalloc((void **)&strategy_d_h[j][i].double_target_GPU, sizeof(UINT32) * table_for_gpu_info[i].
                double_size));
            CHECK(hipMalloc((void **)&strategy_d_h[j][i].double_target_GPU_platform, sizeof(UINT32) *
                table_for_gpu_info[i].double_size));
        }
    }

    CHECK(hipHostAlloc((void **)&metainfo_d, sizeof(Global_Table_Meta *) * param->device_cnt, hipHostMallocDefault));
    CHECK(hipHostAlloc((void **)&metainfo_h, sizeof(Global_Table_Meta *) * param->device_cnt, hipHostMallocDefault));
    for (size_t j = 0; j < param->device_cnt; ++j) {
        CHECK(hipSetDevice(param->device_IDs[j]));
        CHECK(hipMalloc((void **)&metainfo_d[j], sizeof(Global_Table_Meta) * param->table_cnt));
        CHECK(hipHostAlloc((void **)&metainfo_h[j], sizeof(Global_Table_Meta) * param->table_cnt, hipHostMallocDefault
        ));
    }

    //TODO: 索引分配空间，释放，生成索引并在预处理阶段使用，可能需要改造sub_txn部分以增加在launcher中使用index的适配
    CHECK(hipHostAlloc((void**)&index_h,sizeof(Global_Table_Index *) * param->device_cnt, hipHostMallocDefault));
    for (size_t j = 0; j < param->device_cnt; ++j) {
        CHECK(hipSetDevice(param->device_IDs[j]));
        CHECK(hipHostAlloc((void**)&index_h[j], sizeof(Global_Table_Index) * param->table_cnt, hipHostMallocDefault));
    }
    CHECK(hipHostAlloc((void**)&index_d_h,sizeof(Global_Table_Index *) * param->device_cnt, hipHostMallocDefault));
    for (size_t j = 0; j < param->device_cnt; ++j) {
        CHECK(hipSetDevice(param->device_IDs[j]));
        CHECK(hipHostAlloc((void**)&index_d_h[j], sizeof(Global_Table_Index) * param->table_cnt, hipHostMallocDefault
        ));
    }
    CHECK(hipHostAlloc((void**)&index_d,sizeof(Global_Table_Index *) * param->device_cnt, hipHostMallocDefault));
    for (size_t j = 0; j < param->device_cnt; ++j) {
        CHECK(hipSetDevice(param->device_IDs[j]));
        CHECK(hipMalloc((void**)&index_d[j], sizeof(Global_Table_Index) * param->table_cnt));
    }

    std::cout << "end gpudatabase.cu GPUdatabase::malloc_global_row()" << std::endl;
}

void GPUdatabase::copy_to_global_row(std::shared_ptr<Param> param, Global_Table_Info *table_for_gpu_info,
                                     Global_Table *table_for_gpu, Global_Table_Index *index_for_GPU) {
    std::cout << "start gpudatabase.cu GPUdatabase::copy_to_global_row()" << std::endl;

    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipSetDevice(param->device_IDs[i]));
        CHECK(hipMemcpy(tables_info_d[i], table_for_gpu_info, sizeof(Global_Table_Info) * param->table_cnt,
            hipMemcpyHostToDevice));
        CHECK(hipMemcpy(tables_info_h, table_for_gpu_info, sizeof(Global_Table_Info) * param->table_cnt,
            hipMemcpyHostToHost));
    }

    // 此处需要数据划分策略, 并构造数据地图以展示每张卡上存了什么数据

    data_partition_strategy(param);

    for (size_t j = 0; j < param->device_cnt; ++j) {
        CHECK(hipSetDevice(param->device_IDs[j]));

        for (size_t i = 0; i < param->table_cnt; ++i) {
            CHECK(hipMemcpy(strategy_d_h[j][i].int_target_GPU, strategy_h[i].int_target_GPU, sizeof(UINT32) *
                table_for_gpu_info[i].int_size, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(strategy_d_h[j][i].int_target_GPU_platform, strategy_h[i].int_target_GPU_platform, sizeof(
                UINT32) * table_for_gpu_info[i].int_size, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(strategy_d_h[j][i].string_target_GPU, strategy_h[i].string_target_GPU, sizeof(UINT32) *
                table_for_gpu_info[i].string_size, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(strategy_d_h[j][i].string_target_GPU_platform, strategy_h[i].string_target_GPU_platform,
                sizeof(UINT32) * table_for_gpu_info[i].string_size, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(strategy_d_h[j][i].double_target_GPU, strategy_h[i].double_target_GPU, sizeof(UINT32) *
                table_for_gpu_info[i].double_size, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(strategy_d_h[j][i].double_target_GPU_platform, strategy_h[i].double_target_GPU_platform,
                sizeof(UINT32) * table_for_gpu_info[i].double_size, hipMemcpyHostToDevice));
        }
        CHECK(hipMemcpy(strategy_d[j], strategy_d_h[j], sizeof(Global_Table_Strategy) * param->table_cnt,
            hipMemcpyHostToDevice));
    }

    for (size_t j = 0; j < param->device_cnt; ++j) {
        CHECK(hipSetDevice(param->device_IDs[j]));
        CHECK(hipMemcpy(metainfo_d[j], metainfo_h[j], sizeof(Global_Table_Meta) * param->table_cnt,
            hipMemcpyHostToDevice));
    }

    for (size_t i = 0; i < param->table_cnt; ++i) {
        CHECK(hipMemcpy(tables_h[i].int_data , table_for_gpu[i].int_data, sizeof(INT32) * table_for_gpu_info[
            i].int_size, hipMemcpyHostToHost));
        CHECK(hipMemcpy(tables_h[i].string_data , table_for_gpu[i].string_data, sizeof(UINT32) *
            table_for_gpu_info[i].string_size * table_for_gpu_info[i].string_length,
            hipMemcpyHostToHost));
        CHECK(hipMemcpy(tables_h[i].double_data , table_for_gpu[i].double_data, sizeof(DOUBLE) *
            table_for_gpu_info[i].double_size, hipMemcpyHostToHost));
    }

    for (size_t j = 0; j < param->device_cnt; ++j) {
        CHECK(hipSetDevice(param->device_IDs[j]));
        for (size_t i = 0; i < param->table_cnt; ++i) {
            CHECK(hipMemcpy(tables_d_h[j][i].int_data , table_for_gpu[i].int_data + metainfo_h[j][i].row_start *
                table_for_gpu_info[i].int_size, sizeof(INT32) * table_for_gpu_info [i].int_size *
                metainfo_h[j][i].table_slice_size, hipMemcpyHostToDevice));
            CHECK(hipMemcpy(tables_d_h[j][i].string_data , table_for_gpu[i].string_data + metainfo_h[j][i].row_start *
                table_for_gpu_info[i].string_size*table_for_gpu_info[i].string_length,
                sizeof(UINT32) * table_for_gpu_info[i].string_size * table_for_gpu_info[i].string_length*
                metainfo_h[j][i]. table_slice_size, hipMemcpyHostToDevice ));
            CHECK(hipMemcpy(tables_d_h[j][i].double_data , table_for_gpu[i].double_data + metainfo_h[j][i].row_start *
                table_for_gpu_info[i].double_size, sizeof(DOUBLE) * table_for_gpu_info[i].double_size*
                metainfo_h[j][i].table_slice_size, hipMemcpyHostToDevice));
        }
        CHECK(hipMemcpy(tables_d[j], tables_d_h[j], sizeof(Global_Table) * param->table_cnt, hipMemcpyHostToDevice));
    }

    //Index
    if (param->benchmark == "TEST") {
    } else if (param->benchmark == "TPCC_PART") {
        for (size_t j = 0; j < param->device_cnt; ++j) {
            CHECK(hipMemcpy(index_h[j][2].index, index_for_GPU[2].index+metainfo_h[j][2].row_start,
                sizeof(UINT32)*metainfo_h[j][2].table_slice_size, hipMemcpyHostToHost));
            CHECK(hipMemcpy(index_d_h[j][2].index,index_for_GPU[2].index+metainfo_h[j][2].row_start,
                sizeof(UINT32)*metainfo_h[j][2].table_slice_size, hipMemcpyHostToHost));
        }
        for (size_t j = 0; j < param->device_cnt; ++j) {
            CHECK(hipMemcpy(index_d[j],index_d_h[j],
                sizeof(Global_Table_Index)*param->table_cnt,hipMemcpyDeviceToHost));
        }
        // for (size_t j = 0; j < param->device_cnt; ++j) {
        //     std::cout << "index_h[" << j << "][2].index:";
        //     for (uint32_t i = 0; i < 10; ++i) {
        //         std::cout << std::hex << index_h[j][2].index[i] << ",";
        //     }
        //     std::cout << std::dec << std::endl;
        // }
    } else if (param->benchmark == "TPCC_ALL") {
        for (size_t j = 0; j < param->device_cnt; ++j) {
            CHECK(hipMemcpy(index_h[j][2].index, index_for_GPU[2].index+metainfo_h[j][2].row_start,
                sizeof(UINT32)*metainfo_h[j][2].table_slice_size, hipMemcpyHostToHost));
            CHECK(hipMemcpy(index_d_h[j][2].index,index_for_GPU[2].index+metainfo_h[j][2].row_start,
                sizeof(UINT32)*metainfo_h[j][2].table_slice_size, hipMemcpyHostToHost));
        }
        for (size_t j = 0; j < param->device_cnt; ++j) {
            CHECK(hipMemcpy(index_d[j],index_d_h[j],
                sizeof(Global_Table_Index)*param->table_cnt,hipMemcpyDeviceToHost));
        }
    } else if (param->benchmark == "YCSB_A") {
    } else if (param->benchmark == "YCSB_B") {
    } else if (param->benchmark == "YCSB_C") {
    } else if (param->benchmark == "YCSB_D") {
    } else if (param->benchmark == "YCSB_E") {
    }
    std::cout << "end gpudatabase.cu GPUdatabase::copy_to_global_row()" << std::endl;
}

void GPUdatabase::free_global_row(std::shared_ptr<Param> param, Global_Table_Info *table_for_gpu_info,
                                  Global_Table *table_for_gpu, Global_Table_Index *index_for_GPU) {
    std::cout << "start gpudatabase.cu GPUdatabase::free_global_row()" << std::endl;
    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipFree(tables_info_d[i]));
    }
    // CHECK(hipHostFree(tables_info_d));
    delete[] tables_info_d;
    // CHECK(hipHostFree(tables_info_h));
    delete[] tables_info_h;
    for (size_t i = 0; i < param->table_cnt; ++i) {
        // CHECK(hipHostFree(tables_h[i].int_data));
        // CHECK(hipHostFree(tables_h[i].string_data));
        // CHECK(hipHostFree(tables_h[i].double_data));
        delete[] tables_h[i].int_data;
        delete[] tables_h[i].string_data;
        delete[] tables_h[i].double_data;
    }
    for (size_t j = 0; j < param->device_cnt; ++j) {
        for (size_t i = 0; i < param->table_cnt; ++i) {
            CHECK(hipFree(tables_d_h[j][i].int_data));
            CHECK(hipFree(tables_d_h[j][i].string_data));
            CHECK(hipFree(tables_d_h[j][i].double_data));
        }
        CHECK(hipFree(tables_d[j]));
        CHECK(hipHostFree(tables_d_h[j]));
    }
    CHECK(hipHostFree(tables_d));
    CHECK(hipHostFree(tables_d_h));
    CHECK(hipHostFree(tables_h));

    for (size_t j = 0; j < param->device_cnt; ++j) {
        for (size_t i = 0; i < param->table_cnt; ++i) {
            CHECK(hipFree(strategy_d_h[j][i].int_target_GPU));
            CHECK(hipFree(strategy_d_h[j][i].int_target_GPU_platform));
            CHECK(hipFree(strategy_d_h[j][i].string_target_GPU));
            CHECK(hipFree(strategy_d_h[j][i].string_target_GPU_platform));
            CHECK(hipFree(strategy_d_h[j][i].double_target_GPU));
            CHECK(hipFree(strategy_d_h[j][i].double_target_GPU_platform));
        }
        CHECK(hipFree(strategy_d[j]));
        CHECK(hipHostFree(strategy_d_h[j]));
    }
    CHECK(hipHostFree(strategy_d));
    CHECK(hipHostFree(strategy_d_h));

    for (size_t i = 0; i < param->table_cnt; ++i) {
        CHECK(hipHostFree(strategy_h[i].int_target_GPU));
        CHECK(hipHostFree(strategy_h[i].int_target_GPU_platform));
        CHECK(hipHostFree(strategy_h[i].string_target_GPU));
        CHECK(hipHostFree(strategy_h[i].string_target_GPU_platform));
        CHECK(hipHostFree(strategy_h[i].double_target_GPU));
        CHECK(hipHostFree(strategy_h[i].double_target_GPU_platform));
    }
    CHECK(hipHostFree(strategy_h));

    for (size_t j = 0; j < param->device_cnt; ++j) {
        CHECK(hipFree(metainfo_d[j]));
        CHECK(hipHostFree(metainfo_h[j]));
    }
    CHECK(hipHostFree(metainfo_d));
    CHECK(hipHostFree(metainfo_h));

    if (param->benchmark == "TEST") {
    } else if (param->benchmark == "TPCC_PART") {
        for (size_t j = 0; j < param->device_cnt; ++j) {
            CHECK(hipHostFree(index_h[j][2].index));
        }
        for (size_t j = 0; j < param->device_cnt; ++j) {
            CHECK(hipFree(index_d_h[j][2].index));
        }
    } else if (param->benchmark == "TPCC_ALL") {
        for (size_t j = 0; j < param->device_cnt; ++j) {
            CHECK(hipHostFree(index_h[j][2].index));
        }
        for (size_t j = 0; j < param->device_cnt; ++j) {
            CHECK(hipFree(index_d_h[j][2].index));
        }
    } else if (param->benchmark == "YCSB_A") {
    } else if (param->benchmark == "YCSB_B") {
    } else if (param->benchmark == "YCSB_C") {
    } else if (param->benchmark == "YCSB_D") {
    } else if (param->benchmark == "YCSB_E") {
    }

    for (size_t j = 0; j < param->device_cnt; ++j) {
        CHECK(hipHostFree(index_h[j]));
    }
    CHECK(hipHostFree(index_h));
    for (size_t j = 0; j < param->device_cnt; ++j) {
        CHECK(hipHostFree(index_d_h[j]));
    }
    CHECK(hipHostFree(index_d_h));
    for (size_t j = 0; j < param->device_cnt; ++j) {
        CHECK(hipFree(index_d[j]));
    }
    CHECK(hipHostFree(index_d));

    std::cout << "end gpudatabase.cu GPUdatabase::free_global_row()" << std::endl;
}

void GPUdatabase::data_partition_strategy(std::shared_ptr<Param> param) {
    std::cout << "start gpudatabase.cu GPUdatabase::data_partition_strategy()" << std::endl;
    // TODO: 构造数据划分策略，启发式|代价模型计算并判断
    if (param->benchmark == "TEST") {
        // TODO: 策略划分，无需GPU执行事务的数据标记为GPU不可见
        for (size_t j = 0; j < param->table_cnt; j++) {
            // std::cout << "strategy_h[j].int_target_GPU:" << " ";
            for (size_t i = 0; i < tables_info_h[j].int_size; ++i) {
                strategy_h[j].int_target_GPU[i] = 0xff;
                strategy_h[j].int_target_GPU_platform[i] = 1;
                // std::cout << strategy_h[j].int_target_GPU[i] << " ";
            }
            // std::cout << std::endl;

            // std::cout << "strategy_h[j].string_target_GPU:" << " ";
            for (size_t i = 0; i < tables_info_h[j].string_size; ++i) {
                strategy_h[j].string_target_GPU[i] = 0xff;
                strategy_h[j].string_target_GPU_platform[i] = 1;
                // std::cout << strategy_h[j].string_target_GPU[i] << " ";
            }
            // std::cout << std::endl;

            // std::cout << "strategy_h[j].double_target_GPU:" << " ";
            for (size_t i = 0; i < tables_info_h[j].double_size; ++i) {
                strategy_h[j].double_target_GPU[i] = 0xff;
                strategy_h[j].double_target_GPU_platform[i] = 1;
                // std::cout << strategy_h[j].double_target_GPU[i] << " ";
            }
            // std::cout << std::endl;
        }

        for (size_t j = 0; j < param->device_cnt; ++j) {
            for (size_t i = 0; i < param->table_cnt; ++i) {
                uint32_t table_slice_size = tables_info_h[i].table_size / param->device_cnt;
                metainfo_h[j][i].row_start = table_slice_size * j;
                metainfo_h[j][i].row_end = table_slice_size * (j + 1);
                metainfo_h[j][i].table_slice_size = table_slice_size;

                std::cout << "table:" << i << ",start:" << metainfo_h[j][i].row_start << ",end:" << metainfo_h[j][i].
                        row_end
                        << ",slice_size:" << metainfo_h[j][i].table_slice_size << std::endl;
            }
            metainfo_h[j][0].table_size = param->test_1_size;
            metainfo_h[j][1].table_size = param->test_2_size;
        }
    } else if (param->benchmark == "TPCC_PART") {
        for (size_t j = 0; j < param->table_cnt; j++) {
            for (size_t i = 0; i < tables_info_h[j].int_size; ++i) {
                strategy_h[j].int_target_GPU[i] = 0xff;
                strategy_h[j].int_target_GPU_platform[i] = 1;
            }
            for (size_t i = 0; i < tables_info_h[j].string_size; ++i) {
                strategy_h[j].string_target_GPU[i] = 0xff;
                strategy_h[j].string_target_GPU_platform[i] = 1;
            }
            for (size_t i = 0; i < tables_info_h[j].double_size; ++i) {
                strategy_h[j].double_target_GPU[i] = 0xff;
                strategy_h[j].double_target_GPU_platform[i] = 1;
            }
        }

        for (size_t j = 0; j < param->device_cnt; ++j) {
            for (size_t i = 0; i < param->table_cnt; ++i) {
                uint32_t table_slice_size = tables_info_h[i].table_size / param->device_cnt;
                metainfo_h[j][i].row_start = table_slice_size * j;
                metainfo_h[j][i].row_end = table_slice_size * (j + 1);
                metainfo_h[j][i].table_slice_size = table_slice_size;
                if (i == 0 || i == 1) {
                    metainfo_h[j][i].bitmap_row_slice_size = table_slice_size;
                }

                std::cout << "table:" << i << ",start:" << metainfo_h[j][i].row_start << ",end:" << metainfo_h[j][i].
                        row_end << ",slice_size:" << metainfo_h[j][i].table_slice_size << std::endl;
            }
        }
        for (size_t j = 0; j < param->device_cnt; ++j) {
            CHECK(hipHostAlloc((void**)&index_h[j][2].index,sizeof(UINT32)*tables_info_h[2].table_size,
                hipHostMallocDefault));
        }
        for (size_t j = 0; j < param->device_cnt; ++j) {
            CHECK(hipSetDevice(param->device_IDs[j]));
            CHECK(hipMalloc((void**)&index_d_h[j][2].index,sizeof(UINT32)*tables_info_h[2].table_size));
        }
    } else if (param->benchmark == "TPCC_ALL") {
        for (size_t j = 0; j < param->table_cnt; j++) {
            for (size_t i = 0; i < tables_info_h[j].int_size; ++i) {
                strategy_h[j].int_target_GPU[i] = 0xff;
                strategy_h[j].int_target_GPU_platform[i] = 1;
            }
            for (size_t i = 0; i < tables_info_h[j].string_size; ++i) {
                strategy_h[j].string_target_GPU[i] = 0xff;
                strategy_h[j].string_target_GPU_platform[i] = 1;
            }
            for (size_t i = 0; i < tables_info_h[j].double_size; ++i) {
                strategy_h[j].double_target_GPU[i] = 0xff;
                strategy_h[j].double_target_GPU_platform[i] = 1;
            }
        }

        for (size_t j = 0; j < param->device_cnt; ++j) {
            for (size_t i = 0; i < param->table_cnt; ++i) {
                uint32_t table_slice_size = tables_info_h[i].table_size / param->device_cnt;
                metainfo_h[j][i].row_start = table_slice_size * j;
                metainfo_h[j][i].row_end = table_slice_size * (j + 1);
                metainfo_h[j][i].table_slice_size = table_slice_size;
                if (i == 0 || i == 1) {
                    metainfo_h[j][i].bitmap_row_slice_size = table_slice_size;
                }
                std::cout << "table:" << i << ",start:" << metainfo_h[j][i].row_start << ",end:" << metainfo_h[j][i].
                        row_end << ",slice_size:" << metainfo_h[j][i].table_slice_size << std::endl;
            }
        }
        for (size_t j = 0; j < param->device_cnt; ++j) {
            CHECK(hipHostAlloc((void**)&index_h[j][2].index,sizeof(UINT32)*tables_info_h[2].table_size,
                hipHostMallocDefault));
        }
        for (size_t j = 0; j < param->device_cnt; ++j) {
            CHECK(hipSetDevice(param->device_IDs[j]));
            CHECK(hipMalloc((void**)&index_d_h[j][2].index,sizeof(UINT32)*tables_info_h[2].table_size));
        }
    } else if (param->benchmark == "YCSB_A") {
        // TODO: 策略划分，无需GPU执行事务的数据标记为GPU不可见
        for (size_t j = 0; j < param->table_cnt; j++) {
            for (size_t i = 0; i < tables_info_h[j].int_size; ++i) {
                strategy_h[j].int_target_GPU[i] = 0xff;
                strategy_h[j].int_target_GPU_platform[i] = 1;
            }
            for (size_t i = 0; i < tables_info_h[j].string_size; ++i) {
                strategy_h[j].string_target_GPU[i] = 0xff;
                strategy_h[j].string_target_GPU_platform[i] = 1;
            }
            for (size_t i = 0; i < tables_info_h[j].double_size; ++i) {
                strategy_h[j].double_target_GPU[i] = 0xff;
                strategy_h[j].double_target_GPU_platform[i] = 1;
            }
        }

        for (size_t j = 0; j < param->device_cnt; ++j) {
            for (size_t i = 0; i < param->table_cnt; ++i) {
                uint32_t table_slice_size = tables_info_h[i].table_size / param->device_cnt;
                metainfo_h[j][i].row_start = table_slice_size * j;
                metainfo_h[j][i].row_end = table_slice_size * (j + 1);
                metainfo_h[j][i].table_slice_size = table_slice_size;
                metainfo_h[j][i].bitmap_row_slice_size = param->bitmap_row_cnt;
                std::cout << "table:" << i << ",start:" << metainfo_h[j][i].row_start << ",end:" << metainfo_h[j][i].
                        row_end << ",slice_size:" << metainfo_h[j][i].table_slice_size << std::endl;
            }
            metainfo_h[j][0].table_size = param->ycsb_size;
        }
    } else if (param->benchmark == "YCSB_B") {
        // TODO: 策略划分，无需GPU执行事务的数据标记为GPU不可见
        for (size_t j = 0; j < param->table_cnt; j++) {
            for (size_t i = 0; i < tables_info_h[j].int_size; ++i) {
                strategy_h[j].int_target_GPU[i] = 0xff;
                strategy_h[j].int_target_GPU_platform[i] = 1;
            }
            for (size_t i = 0; i < tables_info_h[j].string_size; ++i) {
                strategy_h[j].string_target_GPU[i] = 0xff;
                strategy_h[j].string_target_GPU_platform[i] = 1;
            }
            for (size_t i = 0; i < tables_info_h[j].double_size; ++i) {
                strategy_h[j].double_target_GPU[i] = 0xff;
                strategy_h[j].double_target_GPU_platform[i] = 1;
            }
        }

        for (size_t j = 0; j < param->device_cnt; ++j) {
            for (size_t i = 0; i < param->table_cnt; ++i) {
                uint32_t table_slice_size = tables_info_h[i].table_size / param->device_cnt;
                metainfo_h[j][i].row_start = table_slice_size * j;
                metainfo_h[j][i].row_end = table_slice_size * (j + 1);
                metainfo_h[j][i].table_slice_size = table_slice_size;
                metainfo_h[j][i].bitmap_row_slice_size = param->bitmap_row_cnt;

                std::cout << "table:" << i << ",start:" << metainfo_h[j][i].row_start << ",end:" << metainfo_h[j][i].
                        row_end << ",slice_size:" << metainfo_h[j][i].table_slice_size << std::endl;
            }
            metainfo_h[j][0].table_size = param->ycsb_size;
        }
    } else if (param->benchmark == "YCSB_C") {
        // TODO: 策略划分，无需GPU执行事务的数据标记为GPU不可见
        for (size_t j = 0; j < param->table_cnt; j++) {
            for (size_t i = 0; i < tables_info_h[j].int_size; ++i) {
                strategy_h[j].int_target_GPU[i] = 0xff;
                strategy_h[j].int_target_GPU_platform[i] = 1;
            }
            for (size_t i = 0; i < tables_info_h[j].string_size; ++i) {
                strategy_h[j].string_target_GPU[i] = 0xff;
                strategy_h[j].string_target_GPU_platform[i] = 1;
            }
            for (size_t i = 0; i < tables_info_h[j].double_size; ++i) {
                strategy_h[j].double_target_GPU[i] = 0xff;
                strategy_h[j].double_target_GPU_platform[i] = 1;
            }
        }

        for (size_t j = 0; j < param->device_cnt; ++j) {
            for (size_t i = 0; i < param->table_cnt; ++i) {
                uint32_t table_slice_size = tables_info_h[i].table_size / param->device_cnt;
                metainfo_h[j][i].row_start = table_slice_size * j;
                metainfo_h[j][i].row_end = table_slice_size * (j + 1);
                metainfo_h[j][i].table_slice_size = table_slice_size;
                metainfo_h[j][i].bitmap_row_slice_size = param->bitmap_row_cnt;

                std::cout << "table:" << i << ",start:" << metainfo_h[j][i].row_start << ",end:" << metainfo_h[j][i].
                        row_end << ",slice_size:" << metainfo_h[j][i].table_slice_size << std::endl;
            }
            metainfo_h[j][0].table_size = param->ycsb_size;
        }
    } else if (param->benchmark == "YCSB_D") {
        // TODO: 策略划分，无需GPU执行事务的数据标记为GPU不可见
        for (size_t j = 0; j < param->table_cnt; j++) {
            for (size_t i = 0; i < tables_info_h[j].int_size; ++i) {
                strategy_h[j].int_target_GPU[i] = 0xff;
                strategy_h[j].int_target_GPU_platform[i] = 1;
            }
            for (size_t i = 0; i < tables_info_h[j].string_size; ++i) {
                strategy_h[j].string_target_GPU[i] = 0xff;
                strategy_h[j].string_target_GPU_platform[i] = 1;
            }
            for (size_t i = 0; i < tables_info_h[j].double_size; ++i) {
                strategy_h[j].double_target_GPU[i] = 0xff;
                strategy_h[j].double_target_GPU_platform[i] = 1;
            }
        }

        for (size_t j = 0; j < param->device_cnt; ++j) {
            for (size_t i = 0; i < param->table_cnt; ++i) {
                uint32_t table_slice_size = tables_info_h[i].table_size / param->device_cnt;
                metainfo_h[j][i].row_start = table_slice_size * j;
                metainfo_h[j][i].row_end = table_slice_size * (j + 1);
                metainfo_h[j][i].table_slice_size = table_slice_size;
                metainfo_h[j][i].bitmap_row_slice_size = param->bitmap_row_cnt;

                std::cout << "table:" << i << ",start:" << metainfo_h[j][i].row_start << ",end:" << metainfo_h[j][i].
                        row_end << ",slice_size:" << metainfo_h[j][i].table_slice_size << std::endl;
            }
            metainfo_h[j][0].table_size = param->ycsb_size;
        }
    } else if (param->benchmark == "YCSB_E") {
        // TODO: 策略划分，无需GPU执行事务的数据标记为GPU不可见
        for (size_t j = 0; j < param->table_cnt; j++) {
            for (size_t i = 0; i < tables_info_h[j].int_size; ++i) {
                strategy_h[j].int_target_GPU[i] = 0xff;
                strategy_h[j].int_target_GPU_platform[i] = 1;
            }
            for (size_t i = 0; i < tables_info_h[j].string_size; ++i) {
                strategy_h[j].string_target_GPU[i] = 0xff;
                strategy_h[j].string_target_GPU_platform[i] = 1;
            }
            for (size_t i = 0; i < tables_info_h[j].double_size; ++i) {
                strategy_h[j].double_target_GPU[i] = 0xff;
                strategy_h[j].double_target_GPU_platform[i] = 1;
            }
        }

        for (size_t j = 0; j < param->device_cnt; ++j) {
            for (size_t i = 0; i < param->table_cnt; ++i) {
                uint32_t table_slice_size = tables_info_h[i].table_size / param->device_cnt;
                metainfo_h[j][i].row_start = table_slice_size * j;
                metainfo_h[j][i].row_end = table_slice_size * (j + 1);
                metainfo_h[j][i].table_slice_size = table_slice_size;
                metainfo_h[j][i].bitmap_row_slice_size = param->bitmap_row_cnt;

                std::cout << "table:" << i << ",start:" << metainfo_h[j][i].row_start << ",end:" << metainfo_h[j][i].
                        row_end << ",slice_size:" << metainfo_h[j][i].table_slice_size << std::endl;
            }
            metainfo_h[j][0].table_size = param->ycsb_size;
        }
    }
    std::cout << "end gpudatabase.cu GPUdatabase::data_partition_strategy()" << std::endl;
}

Global_Table_Info *GPUdatabase::get_table_info(const int deviceID) {
    return tables_info_d[deviceID];
}

Global_Table *GPUdatabase::get_table(const int deviceID) {
    return tables_d[deviceID];
}

Global_Table_Index *GPUdatabase::get_index(const int deviceID) {
    return index_d[deviceID];
}

Global_Table_Meta *GPUdatabase::get_meta(const int deviceID) {
    return metainfo_d[deviceID];
}

Global_Table_Strategy *GPUdatabase::get_strategy(const int deviceID) {
    return strategy_d[deviceID];
}

void GPUdatabase::launch_test(std::shared_ptr<Param> param) {
    hipStream_t *streams;
    streams = new hipStream_t[param->device_cnt];
    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipSetDevice(param->device_IDs[i]));
        CHECK(hipStreamCreate(&streams[i]));
    }

    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipSetDevice(param->device_IDs[i]));
        test<<<1, 1, 0, streams[i]>>>(i, get_table_info(i), get_table(i), get_strategy(i), get_meta(i), get_index(i));
    }

    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipStreamSynchronize(streams[i]));
    }

    for (size_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipStreamDestroy(streams[i]));
    }
    delete[] streams;
}

__global__ void test(int ID, Global_Table_Info *table_info, Global_Table *table, Global_Table_Strategy *strategy,
                     Global_Table_Meta *metainfo, Global_Table_Index *index) {
    // UINT32 tableID = 0;
    // UINT32 dataID = 0;

    // printf("ID:%d,tableID:%d,DataID:%d,int_size:%d,int_data:%d,int_target_GPU:%d\n",
    //        ID, tableID, dataID, table_info[tableID].int_size,
    //        table[tableID].int_data[dataID], strategy[tableID].int_target_GPU[dataID]);
    // printf("ID:%d,tableID:%d,DataID:%d,table[0].int_data[0]:%d,table[1].int_data[0]:%d\n",
    //        ID, tableID, dataID, table[0].int_data[0], table[1].int_data[0]);

    // for (size_t i = 0; i < metainfo[0].table_slice_size; ++i) {
    //     printf("ID:%d,tableID:%d,DataID:%d,table[0].int_data[]:%d\n",
    //            ID, tableID, dataID, table[0].int_data[i]);
    // }
    // printf("\n");
    // for (size_t i = 0; i < metainfo[1].table_slice_size; ++i) {
    //     printf("ID:%d,tableID:%d,DataID:%d,table[1].int_data[]:%d\n",
    //            ID, tableID, dataID, table[1].int_data[i]);
    // }
    // printf("\n");
    // for (size_t i = 0; i < 5; ++i) {
    //     printf("ID:%d,tableID:%d,DataID:%d,table[0].int_data[]:%d\n",
    //            ID, tableID, dataID, table[0].int_data[i]);
    // }
    // printf("\n");
    // for (size_t i = 0; i < 5; ++i) {
    //     printf("ID:%d,tableID:%d,DataID:%d,table[1].int_data[]:%d\n",
    //            ID, tableID, dataID, table[1].int_data[i]);
    // }
    // printf("\n");
    for (uint32_t i = 0; i < 10; ++i) {
        printf("ID:%d,index[2].index[%d]:%x\n", ID, i, index[2].index[i]);
    }
}
