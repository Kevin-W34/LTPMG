#include "hip/hip_runtime.h"
#include "../include/gpulauncher.cuh"

// __device__ void print(uint32_t device_ID) {
//     // uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
//     // printf("__device__ void GPUexecutor::print() thID:%d\n", thID);
// }

template<typename D>
__device__ void select_operator(uint32_t tableID,
                                uint32_t rowID,
                                uint32_t data_ID,
                                u_char data_type,
                                Global_Table *tables,
                                Global_Table_Info *table_info,
                                Global_Table_Meta *metainfo) {
    D data;
    rowID = rowID % metainfo[tableID].table_slice_size;
    if (data_type == 1) {
        data = tables[tableID].int_data[rowID * table_info[tableID].int_size + data_ID];
    } else if (data_type == 0) {
        data = tables[tableID].string_data[
            rowID * table_info[tableID].string_size * table_info[tableID].string_length + data_ID];
    } else if (data_type == 3) {
        data = tables[tableID].double_data[
            rowID * table_info[tableID].double_size + data_ID];
    }
}

template<typename D>
__device__ void scan_operator(uint32_t tableID,
                              uint32_t rowID,
                              uint32_t data_ID,
                              u_char data_type,
                              Global_Table *tables,
                              Global_Table_Info *table_info,
                              Global_Table_Meta *metainfo) {
    D data;
    rowID = rowID % metainfo[tableID].table_slice_size;

    if (data_type == 1) {
        data = tables[tableID].int_data[rowID * table_info[tableID].int_size + data_ID];
    } else if (data_type == 0) {
        data = tables[tableID].string_data[
            rowID * table_info[tableID].string_size * table_info[tableID].string_length + data_ID];
    } else if (data_type == 3) {
        data = tables[tableID].double_data[
            rowID * table_info[tableID].double_size + data_ID];
    }
}

template<typename D>
__device__ void insert_operator(uint32_t tableID,
                                uint32_t rowID,
                                uint32_t data_ID,
                                u_char data_type,
                                D data,
                                Global_Table *tables,
                                Global_Table_Info *table_info,
                                Global_Table_Meta *metainfo) {
    rowID = rowID % metainfo[tableID].table_slice_size;

    if (data_type == 1) {
        INT32 int_data = tables[tableID].int_data[rowID * table_info[tableID].int_size + data_ID];
        // tables[tableID].int_data[rowID * table_info[tableID].int_size + data_ID] = int_data;
    } else if (data_type == 0) {
        UINT32 string_data = tables[tableID].string_data[
            rowID * table_info[tableID].string_size * table_info[tableID].string_length];
        // tables[tableID].string_data[
        //     rowID * table_info[tableID].string_size * table_info[tableID].string_length + data_ID] = string_data;
    } else if (data_type == 3) {
        DOUBLE double_data = tables[tableID].double_data[
            rowID * table_info[tableID].double_size + data_ID];
        // tables[tableID].double_data[
        //     rowID * table_info[tableID].double_size + data_ID] = double_data;
    }
}

template<typename D>
__device__ void update_operator(uint32_t tableID,
                                uint32_t rowID,
                                uint32_t data_ID,
                                u_char data_type,
                                D data,
                                Global_Table *tables,
                                Global_Table_Info *table_info,
                                Global_Table_Meta *metainfo) {
    rowID = rowID % metainfo[tableID].table_slice_size;

    if (data_type == 1) {
        INT32 int_data = tables[tableID].int_data[rowID * table_info[tableID].int_size + data_ID];
        // tables[tableID].int_data[rowID * table_info[tableID].int_size + data_ID] = int_data;
    } else if (data_type == 0) {
        UINT32 string_data = tables[tableID].string_data[
            rowID * table_info[tableID].string_size * table_info[tableID].string_length + data_ID];
        // tables[tableID].string_data[
        //     rowID * table_info[tableID].string_size * table_info[tableID].string_length + data_ID] = string_data;
    } else if (data_type == 3) {
        DOUBLE double_data = tables[tableID].double_data[
            rowID * table_info[tableID].double_size + data_ID];
        // tables[tableID].double_data[
        //     rowID * table_info[tableID].double_size + data_ID] = double_data;
    }
}

template<typename D>
__device__ void delete_operator(uint32_t tableID,
                                uint32_t rowID,
                                uint32_t data_ID,
                                u_char data_type,
                                Global_Table *tables,
                                Global_Table_Info *table_info,
                                Global_Table_Meta *metainfo) {
    rowID = rowID % metainfo[tableID].table_slice_size;

    D data;
    if (data_type == 1) {
        data = tables[tableID].int_data[rowID * table_info[tableID].int_size + data_ID];
        // tables[tableID].int_data[rowID * table_info[tableID].int_size + data_ID] = data;
    } else if (data_type == 0) {
        data = tables[tableID].string_data[
            rowID * table_info[tableID].string_size * table_info[tableID].string_length + data_ID];
        // tables[tableID].string_data[
        //     rowID * table_info[tableID].string_size * table_info[tableID].string_length + data_ID] = data;
    } else if (data_type == 3) {
        data = tables[tableID].double_data[
            rowID * table_info[tableID].double_size + data_ID];
        // tables[tableID].double_data[
        //     rowID * table_info[tableID].double_size + data_ID] = data;
    }
}

template<typename D>
__device__ void select_operator_shared(D &d) {
    D data;
    data = d;
    d = data;
}

template<typename D>
__device__ void scan_operator_shared(D &d) {
    D data;
    data = d;
    d = data;
}

template<typename D>
__device__ void insert_operator_shared(D data,
                                       D &d) {
    data = d;
    d = data;
}

template<typename D>
__device__ void update_operator_shared(D data,
                                       D &d) {
    data = d;
    d = data;
}

template<typename D>
__device__ void delete_operator_shared(D &d) {
    D data;
    data = d;
    d = data;
}

__device__ void select_executor(uint32_t device_ID,
                                uint32_t cur_txn,
                                uint32_t device_cnt,
                                Global_Table_Info *table_info,
                                Global_Table *tables,
                                Global_Table_Index *indexes,
                                Global_Table_Meta *metainfo,
                                Global_Table_Strategy *strategy,
                                Global_Txn_Info *txn_info,
                                Global_Txn *txn,
                                Global_Txn_Exec *txn_exec,
                                Global_Txn_Result *txn_result,
                                Global_Txn_Exec_Param *exec_param,
                                Global_Txn_Aux_Struct *aux_struct,
                                Global_Data_Packet *data_packet) {
    uint32_t tableID = txn[cur_txn].subtxn.table_ID;
    uint32_t row_1 = txn[cur_txn].subtxn.dest_Row_1;
    uint32_t tid = txn[cur_txn].subtxn.TID;
    uint32_t type = txn[cur_txn].subtxn.type;
    uint32_t ispopular = txn[cur_txn].subtxn.ispopular;
    uint32_t dest_device = txn[cur_txn].subtxn.dest_device;

    if (dest_device == device_ID) {
        if (ispopular == 0 || ispopular == 1) {
            register_txn_exec(device_ID, type, cur_txn, txn_exec);

            register_cc(cur_txn, ispopular, tableID,
                        row_1, tid, table_info, tables, indexes, metainfo, strategy, txn_info,
                        txn, txn_exec, txn_result, exec_param, aux_struct, data_packet);
        } else {
        }
    } else if (dest_device < 0xffffffff) {
        make_data_packet(device_ID, cur_txn, dest_device, device_cnt, table_info, tables, indexes,
                         metainfo, strategy, txn_info, txn, txn_exec, txn_result, exec_param, aux_struct,
                         data_packet);
    }
}

__device__ void insert_executor(uint32_t device_ID,
                                uint32_t cur_txn,
                                uint32_t device_cnt,
                                Global_Table_Info *table_info,
                                Global_Table *tables,
                                Global_Table_Index *indexes,
                                Global_Table_Meta *metainfo,
                                Global_Table_Strategy *strategy,
                                Global_Txn_Info *txn_info,
                                Global_Txn *txn,
                                Global_Txn_Exec *txn_exec,
                                Global_Txn_Result *txn_result,
                                Global_Txn_Exec_Param *exec_param,
                                Global_Txn_Aux_Struct *aux_struct,
                                Global_Data_Packet *data_packet) {
    uint32_t tableID = txn[cur_txn].subtxn.table_ID;
    uint32_t row_1 = txn[cur_txn].subtxn.dest_Row_1;
    uint32_t tid = txn[cur_txn].subtxn.TID;
    uint32_t type = txn[cur_txn].subtxn.type;
    uint32_t ispopular = txn[cur_txn].subtxn.ispopular;
    uint32_t dest_device = txn[cur_txn].subtxn.dest_device;

    if (dest_device == device_ID) {
        register_txn_exec(device_ID, type, cur_txn, txn_exec);

        register_cc(cur_txn, ispopular, tableID,
                    row_1, tid, table_info, tables, indexes, metainfo, strategy, txn_info,
                    txn, txn_exec, txn_result, exec_param, aux_struct, data_packet);
    } else if (dest_device < 0xffffffff) {
        make_data_packet(device_ID, cur_txn, dest_device, device_cnt, table_info, tables, indexes,
                         metainfo, strategy, txn_info, txn, txn_exec, txn_result, exec_param, aux_struct,
                         data_packet);
    }
}

__device__ void update_executor(uint32_t device_ID,
                                uint32_t cur_txn,
                                uint32_t device_cnt,
                                Global_Table_Info *table_info,
                                Global_Table *tables,
                                Global_Table_Index *indexes,
                                Global_Table_Meta *metainfo,
                                Global_Table_Strategy *strategy,
                                Global_Txn_Info *txn_info,
                                Global_Txn *txn,
                                Global_Txn_Exec *txn_exec,
                                Global_Txn_Result *txn_result,
                                Global_Txn_Exec_Param *exec_param,
                                Global_Txn_Aux_Struct *aux_struct,
                                Global_Data_Packet *data_packet) {
    uint32_t tableID = txn[cur_txn].subtxn.table_ID;
    uint32_t row_1 = txn[cur_txn].subtxn.dest_Row_1;
    uint32_t tid = txn[cur_txn].subtxn.TID;
    uint32_t type = txn[cur_txn].subtxn.type;
    uint32_t ispopular = txn[cur_txn].subtxn.ispopular;
    uint32_t dest_device = txn[cur_txn].subtxn.dest_device;
    if (dest_device == device_ID) {
        register_txn_exec(device_ID, type, cur_txn, txn_exec);

        register_cc(cur_txn, ispopular, tableID,
                    row_1, tid, table_info, tables, indexes, metainfo, strategy, txn_info,
                    txn, txn_exec, txn_result, exec_param, aux_struct, data_packet);
    } else if (dest_device < 0xffffffff) {
        make_data_packet(device_ID, cur_txn, dest_device, device_cnt, table_info, tables, indexes,
                         metainfo, strategy, txn_info, txn, txn_exec, txn_result, exec_param, aux_struct,
                         data_packet);
    }
}

__device__ void scan_executor(uint32_t device_ID,
                              uint32_t cur_txn,
                              uint32_t device_cnt,
                              Global_Table_Info *table_info,
                              Global_Table *tables,
                              Global_Table_Index *indexes,
                              Global_Table_Meta *metainfo,
                              Global_Table_Strategy *strategy,
                              Global_Txn_Info *txn_info,
                              Global_Txn *txn,
                              Global_Txn_Exec *txn_exec,
                              Global_Txn_Result *txn_result,
                              Global_Txn_Exec_Param *exec_param,
                              Global_Txn_Aux_Struct *aux_struct,
                              Global_Data_Packet *data_packet) {
    uint32_t tableID = txn[cur_txn].subtxn.table_ID;
    uint32_t row_1 = txn[cur_txn].subtxn.dest_Row_1;
    uint32_t row_2 = txn[cur_txn].subtxn.dest_Row_2;
    uint32_t tid = txn[cur_txn].subtxn.TID;
    uint32_t row_start = metainfo[tableID].row_start;
    uint32_t row_end = metainfo[tableID].row_end;
    uint32_t type = txn[cur_txn].subtxn.type;
    uint32_t ispopular = txn[cur_txn].subtxn.ispopular;

    bool contain_local = false;
    bool contain_remote = false;
    for (uint32_t row = row_1; row < row_2; ++row) {
        if (row >= row_start && row < row_end) {
            contain_local = true;
            register_cc(cur_txn, ispopular, tableID, row,
                        tid, table_info, tables, indexes, metainfo, strategy, txn_info, txn, txn_exec,
                        txn_result, exec_param, aux_struct, data_packet);
        } else {
            contain_remote = true;
        }
    }
    if (contain_local) {
        register_txn_exec(device_ID, type, cur_txn, txn_exec);
    }
    if (contain_remote) {
        uint32_t dest_device_1 = row_1 / metainfo[tableID].table_slice_size;
        uint32_t dest_device_2 = row_2 / metainfo[tableID].table_slice_size;
        for (uint32_t dest_device = dest_device_1; dest_device < dest_device_2; ++dest_device) {
            make_data_packet(device_ID, cur_txn, dest_device, device_cnt, table_info, tables, indexes,
                             metainfo, strategy, txn_info, txn, txn_exec, txn_result, exec_param, aux_struct,
                             data_packet);
        }
    }
}

__device__ void delete_executor(uint32_t device_ID,
                                uint32_t cur_txn,
                                uint32_t device_cnt,
                                Global_Table_Info *table_info,
                                Global_Table *tables,
                                Global_Table_Index *indexes,
                                Global_Table_Meta *metainfo,
                                Global_Table_Strategy *strategy,
                                Global_Txn_Info *txn_info,
                                Global_Txn *txn,
                                Global_Txn_Exec *txn_exec,
                                Global_Txn_Result *txn_result,
                                Global_Txn_Exec_Param *exec_param,
                                Global_Txn_Aux_Struct *aux_struct,
                                Global_Data_Packet *data_packet) {
    uint32_t tableID = txn[cur_txn].subtxn.table_ID;
    uint32_t row_1 = txn[cur_txn].subtxn.dest_Row_1;
    uint32_t tid = txn[cur_txn].subtxn.TID;
    uint32_t type = txn[cur_txn].subtxn.type;
    uint32_t ispopular = txn[cur_txn].subtxn.ispopular;
    uint32_t dest_device = txn[cur_txn].subtxn.dest_device;

    if (dest_device == device_ID) {
        register_txn_exec(device_ID, type, cur_txn, txn_exec);

        register_cc(cur_txn, ispopular, tableID, row_1,
                    tid, table_info, tables, indexes, metainfo, strategy, txn_info, txn,
                    txn_exec, txn_result, exec_param, aux_struct, data_packet);
    } else if (dest_device < 0xffffffff) {
        make_data_packet(device_ID, cur_txn, dest_device, device_cnt, table_info, tables, indexes,
                         metainfo, strategy, txn_info, txn, txn_exec, txn_result, exec_param, aux_struct, data_packet);
    }
}

__device__ void register_txn_exec(uint32_t device_ID,
                                  uint32_t type,
                                  uint32_t cur_txn,
                                  Global_Txn_Exec *txn_exec) {
    uint32_t txn_exec_loc = 0;
    uint32_t txn_mark = cur_txn;
    if (type == 0) {
        txn_exec_loc = atomicAdd(&txn_exec[0].select_cur, 1);
        txn_exec[0].select_txn_mark[txn_exec_loc] = txn_mark;
    } else if (type == 1) {
        txn_exec_loc = atomicAdd(&txn_exec[0].insert_cur, 1);
        txn_exec[0].insert_txn_mark[txn_exec_loc] = txn_mark;
    } else if (type == 2) {
        txn_exec_loc = atomicAdd(&txn_exec[0].update_cur, 1);
        txn_exec[0].update_txn_mark[txn_exec_loc] = txn_mark;
    } else if (type == 4) {
        txn_exec_loc = atomicAdd(&txn_exec[0].scan_cur, 1);
        txn_exec[0].scan_txn_mark[txn_exec_loc] = txn_mark;
    } else if (type == 3) {
        txn_exec_loc = atomicAdd(&txn_exec[0].delete_cur, 1);
        txn_exec[0].delete_txn_mark[txn_exec_loc] = txn_mark;
    }
}

__device__ void register_cc(uint32_t cur_txn,
                            uint32_t ispopular,
                            uint32_t tableID,
                            uint32_t row,
                            uint32_t tid,
                            Global_Table_Info *table_info,
                            Global_Table *tables,
                            Global_Table_Index *indexes,
                            Global_Table_Meta *metainfo,
                            Global_Table_Strategy *strategy,
                            Global_Txn_Info *txn_info,
                            Global_Txn *txn,
                            Global_Txn_Exec *txn_exec,
                            Global_Txn_Result *txn_result,
                            Global_Txn_Exec_Param *exec_param,
                            Global_Txn_Aux_Struct *aux_struct,
                            Global_Data_Packet *data_packet) {
    uint32_t txn_mark = 0;
    uint32_t cur = row % metainfo[tableID].table_slice_size;
    if (ispopular == 1) {
        // aux_struct
        uint32_t bitmap_size = exec_param[0].bitmap_size;
        uint32_t loc_in = 1 << (tid & 31);
        uint32_t loc_out = tid >> 5;
        loc_out += bitmap_size * cur;
        atomicOr(&aux_struct[tableID].bitmap[loc_out], loc_in);
        txn_mark = cur_txn;
        uint32_t bitmark_offset = bitmap_size * 32 * cur;
        atomicExch(&aux_struct[tableID].bitmap_mark[bitmark_offset + tid], txn_mark);
        atomicAdd(&aux_struct[tableID].bitmap_used_size[cur], 1);
        // aux_struct[tableID].bitmap_all_row[cur_txn] = cur;
    } else if (ispopular == 0) {
        // atomicMin(&aux_struct[tableID].min_TID[cur], tid);
        uint32_t tmp = 0;
        tmp = atomicAdd(&aux_struct[tableID].cnt_TID[cur], 1);
    }
}

__device__ void make_data_packet(uint32_t device_ID,
                                 uint32_t cur_txn,
                                 uint32_t dest_device,
                                 uint32_t device_cnt,
                                 Global_Table_Info *table_info,
                                 Global_Table *tables,
                                 Global_Table_Index *indexes,
                                 Global_Table_Meta *metainfo,
                                 Global_Table_Strategy *strategy,
                                 Global_Txn_Info *txn_info,
                                 Global_Txn *txn,
                                 Global_Txn_Exec *txn_exec,
                                 Global_Txn_Result *txn_result,
                                 Global_Txn_Exec_Param *exec_param,
                                 Global_Txn_Aux_Struct *aux_struct,
                                 Global_Data_Packet *data_packet) {
#ifdef LTPMG_GPUQUERY_TRANSFER_GROUP
    uint32_t cur_group = 0;
    uint32_t dest_group = 0;
    if (device_cnt > 2) {
        cur_group = device_ID / 2;
        dest_group = dest_device / 2;
    } else if (device_cnt > 1) {
        cur_group = device_ID & 1;
        dest_group = dest_device & 1;
    }
    // if (dest_device > device_cnt / 2)
    if (cur_group != dest_group) {
        uint32_t result = atomicAdd(&aux_struct[0].data_packet_cur, 1);
        data_packet[result].mark = cur_txn;
    }
#endif

#ifndef LTPMG_GPUQUERY_TRANSFER_GROUP
    uint32_t result = atomicAdd(&aux_struct[0].data_packet_cur, 1);
    data_packet[result].mark = cur_txn;
#endif
}

__global__ void txn_executor(uint32_t device_ID,
                             uint32_t device_cnt,
                             uint32_t sub_txn_size,
                             Global_Table_Info *table_info,
                             Global_Table *tables,
                             Global_Table_Index *indexes,
                             Global_Table_Meta *metainfo,
                             Global_Table_Strategy *strategy,
                             Global_Txn_Info *txn_info,
                             Global_Txn *txn,
                             Global_Txn_Exec *txn_exec,
                             Global_Txn_Result *txn_result,
                             Global_Txn_Exec_Param *exec_param,
                             Global_Txn_Aux_Struct *aux_struct,
                             Global_Data_Packet *data_packet) {
    const uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    const uint32_t thSize = blockDim.x * gridDim.x;
#ifdef LTPMG_GPUQUERY_TRANSFER_GROUP
    uint32_t cur_group = 0;
    if (device_cnt > 2) {
        cur_group = device_ID / 2;
    } else if (device_cnt > 1) {
        cur_group = device_ID & 1;
    }
    const uint32_t start = sub_txn_size * cur_group / device_cnt;
    const uint32_t end = sub_txn_size * (cur_group + 1) / device_cnt;
#endif
#ifndef LTPMG_GPUQUERY_TRANSFER_GROUP
    const uint32_t start = sub_txn_size * device_ID / device_cnt;
    const uint32_t end = sub_txn_size * (device_ID + 1) / device_cnt;
#endif
    uint32_t cur = thID + start;
    while (cur >= start && cur < end) {
        uint32_t type = txn[cur].subtxn.type;
        if (type == 0) {
            select_executor(device_ID, cur, device_cnt, table_info, tables, indexes, metainfo, strategy, txn_info, txn,
                            txn_exec, txn_result, exec_param, aux_struct, data_packet);
        } else if (type == 1) {
            insert_executor(device_ID, cur, device_cnt, table_info, tables, indexes, metainfo, strategy, txn_info, txn,
                            txn_exec, txn_result, exec_param, aux_struct, data_packet);
        } else if (type == 2) {
            update_executor(device_ID, cur, device_cnt, table_info, tables, indexes, metainfo, strategy, txn_info, txn,
                            txn_exec, txn_result, exec_param, aux_struct, data_packet);
        } else if (type == 3) {
            delete_executor(device_ID, cur, device_cnt, table_info, tables, indexes, metainfo, strategy, txn_info, txn,
                            txn_exec, txn_result, exec_param, aux_struct, data_packet);
        } else if (type == 4) {
            scan_executor(device_ID, cur, device_cnt, table_info, tables, indexes, metainfo, strategy, txn_info, txn,
                          txn_exec, txn_result, exec_param, aux_struct, data_packet);
        }

        cur += thSize;
    }
}

void execute_on_thread(std::shared_ptr<Param> param,
                       uint32_t cur,
                       Global_Txn *global_txn,
                       Global_Txn_Info *global_txn_info,
                       uint32_t txn_offset,
                       Global_Table *table,
                       Global_Table_Info *table_info) {
    printf("thID:%d execute,txn_offset:%d\n", cur, txn_offset);
    uint32_t global_txn_info_ID = global_txn[txn_offset].global_txn_info_ID;

    uint32_t cur_sub_txn_size = global_txn_info[global_txn_info_ID].cur_subtxn_cnt;
    for (uint32_t j = 0; j < cur_sub_txn_size; j++) {
        uint32_t type = global_txn[txn_offset].subtxn.type;
        uint32_t table_ID = global_txn[txn_offset].subtxn.table_ID;
        // uint32_t dest_device = global_txn[txn_offset].subtxn.dest_device;
        uint32_t row_1 = global_txn[txn_offset].subtxn.dest_Row_1;
        uint32_t row_2 = global_txn[txn_offset].subtxn.dest_Row_2;
#ifdef LTPMG_GPULAUNCHER_PRINT
        // std::cout << "global_txn[" << txn_offset << "].subtxn[" << j << "].type:" << global_txn[txn_offset].subtxn.
        //         type << ",dest_device:" << global_txn[txn_offset].subtxn.dest_device << std::endl;
        for (uint32_t i = 0; i < param->table_cnt; ++i) {
            std::cout << table_info[i].int_size << "," << table_info[i].string_size << "," << table_info[i].
                    double_size << std::endl;
        }
#endif
        switch (type) {
            case 0: {
                // std::cout << "select dest_device:" << dest_device << ",tableID:" << table_ID <<
                //         "," << std::endl;
                INT32 *int_data = new INT32[table_info[table_ID].int_size];
                UINT32 *string_data = new UINT32[table_info[table_ID].string_size * table_info[table_ID].string_length];
                DOUBLE *double_data = new DOUBLE[table_info[table_ID].double_size];
                for (uint32_t i = 0; i < table_info[table_ID].int_size; ++i) {
                    int_data[i] = table[table_ID].int_data[row_1 * table_info[table_ID].int_size + i];
                }
                for (uint32_t i = 0; i < table_info[table_ID].string_size; ++i) {
                    for (uint32_t k = 0; k < table_info[table_ID].string_length; ++k) {
                        string_data[i * table_info[table_ID].string_length + k] = table[table_ID].string_data[
                            row_1 * table_info[table_ID].string_size + i * table_info[table_ID].string_length + k];
                    }
                }
                for (uint32_t i = 0; i < table_info[table_ID].double_size; ++i) {
                    double_data[i] = table[table_ID].double_data[row_1 * table_info[table_ID].double_size + i];
                }
#ifdef LTPMG_GPULAUNCHER_PRINT
                std::cout << row_1 << ":";
                for (uint32_t i = 0; i < table_info[table_ID].string_size * table_info[table_ID].string_length; ++i) {
                    std::cout << std::hex << string_data[i] << " ";
                }
                std::cout << std::endl;
                std::cout << std::dec;
#endif
                delete[] int_data;
                delete[] string_data;
                delete[] double_data;
                break;
            }
            case 4: {
                // std::cout << "scan dest_device:" << global_txn[txn_offset].subtxn.dest_device << std::endl;
                uint32_t row_cnt = row_2 - row_1;
                INT32 *int_data = new INT32[table_info[table_ID].int_size * row_cnt];
                UINT32 *string_data = new UINT32[
                    table_info[table_ID].string_size * table_info[table_ID].string_length * row_cnt];
                DOUBLE *double_data = new DOUBLE[table_info[table_ID].double_size * row_cnt];
                for (uint32_t l = row_1; l < row_2; ++l) {
                    for (uint32_t i = 0; i < table_info[table_ID].int_size; ++i) {
                        int_data[i] = table[table_ID].int_data[l * table_info[table_ID].int_size + i];
                    }
                    for (uint32_t i = 0; i < table_info[table_ID].string_size; ++i) {
                        for (uint32_t k = 0; k < table_info[table_ID].string_length; ++k) {
                            string_data[i * table_info[table_ID].string_length + k] = table[table_ID].string_data[
                                l * table_info[table_ID].string_size + i * table_info[table_ID].string_length + k];
                        }
                    }
                    for (uint32_t i = 0; i < table_info[table_ID].double_size; ++i) {
                        double_data[i] = table[table_ID].double_data[l * table_info[table_ID].double_size + i];
                    }
                }
#ifdef LTPMG_GPULAUNCHER_PRINT
                std::cout << row_1 << ":";
                for (uint32_t i = 0; i < table_info[table_ID].string_size * table_info[table_ID].string_length; ++i) {
                    std::cout << std::hex << string_data[i] << " ";
                }
                std::cout << std::endl;
                std::cout << std::dec;
#endif
                delete[] int_data;
                delete[] string_data;
                delete[] double_data;
                break;
            }
            case 1: {
                // std::cout << "insert dest_device:" << global_txn[txn_offset].subtxn.dest_device << std::endl;
                INT32 *int_data = new INT32[table_info[table_ID].int_size];
                UINT32 *string_data = new UINT32[
                    table_info[table_ID].string_size * table_info[table_ID].string_length];
                DOUBLE *double_data = new DOUBLE[table_info[table_ID].double_size];
                for (uint32_t i = 0; i < table_info[table_ID].int_size; ++i) {
                    int_data[i] = table[table_ID].int_data[row_1 * table_info[table_ID].int_size + i];
                }
                for (uint32_t i = 0; i < table_info[table_ID].string_size; ++i) {
                    for (uint32_t k = 0; k < table_info[table_ID].string_length; ++k) {
                        string_data[i * table_info[table_ID].string_length + k] = table[table_ID].string_data[
                            row_1 * table_info[table_ID].string_size + i * table_info[table_ID].string_length + k];
                    }
                }
                for (uint32_t i = 0; i < table_info[table_ID].double_size; ++i) {
                    double_data[i] = table[table_ID].double_data[row_1 * table_info[table_ID].double_size + i];
                }

#ifdef LTPMG_GPULAUNCHER_PRINT
                std::cout << row_1 << ":";
                for (uint32_t i = 0; i < table_info[table_ID].string_size * table_info[table_ID].string_length; ++i) {
                    std::cout << std::hex << string_data[i] << " ";
                }
                std::cout << std::endl;
                std::cout << std::dec;
#endif
                delete[] int_data;
                delete[] string_data;
                delete[] double_data;
                break;
            }
            case 2: {
                // std::cout << "update dest_device:" << global_txn[txn_offset].subtxn.dest_device << std::endl;
                INT32 *int_data = new INT32[table_info[table_ID].int_size];
                UINT32 *string_data = new UINT32[
                    table_info[table_ID].string_size * table_info[table_ID].string_length];
                DOUBLE *double_data = new DOUBLE[table_info[table_ID].double_size];
                for (uint32_t i = 0; i < table_info[table_ID].int_size; ++i) {
                    int_data[i] = table[table_ID].int_data[row_1 * table_info[table_ID].int_size + i];
                }
                for (uint32_t i = 0; i < table_info[table_ID].string_size; ++i) {
                    for (uint32_t k = 0; k < table_info[table_ID].string_length; ++k) {
                        string_data[i * table_info[table_ID].string_length + k] = table[table_ID].string_data[
                            row_1 * table_info[table_ID].string_size + i * table_info[table_ID].string_length + k];
                    }
                }
                for (uint32_t i = 0; i < table_info[table_ID].double_size; ++i) {
                    double_data[i] = table[table_ID].double_data[row_1 * table_info[table_ID].double_size + i];
                }

#ifdef LTPMG_GPULAUNCHER_PRINT
                std::cout << row_1 << ":";
                for (uint32_t i = 0; i < table_info[table_ID].string_size * table_info[table_ID].string_length; ++i) {
                    std::cout << std::hex << string_data[i] << " ";
                }
                std::cout << std::endl;
                std::cout << std::dec;
#endif
                delete[] int_data;
                delete[] string_data;
                delete[] double_data;
                break;
            }
            case 3: {
                // std::cout << "delete dest_device:" << global_txn[txn_offset].subtxn.dest_device << std::endl;
                INT32 *int_data = new INT32[table_info[table_ID].int_size];
                UINT32 *string_data = new UINT32[
                    table_info[table_ID].string_size * table_info[table_ID].string_length];
                DOUBLE *double_data = new DOUBLE[table_info[table_ID].double_size];
                for (uint32_t i = 0; i < table_info[table_ID].int_size; ++i) {
                    int_data[i] = table[table_ID].int_data[row_1 * table_info[table_ID].int_size + i];
                }
                for (uint32_t i = 0; i < table_info[table_ID].string_size; ++i) {
                    for (uint32_t k = 0; k < table_info[table_ID].string_length; ++k) {
                        string_data[i * table_info[table_ID].string_length + k] = table[table_ID].string_data[
                            row_1 * table_info[table_ID].string_size + i * table_info[table_ID].string_length + k];
                    }
                }
                for (uint32_t i = 0; i < table_info[table_ID].double_size; ++i) {
                    double_data[i] = table[table_ID].double_data[row_1 * table_info[table_ID].double_size + i];
                }

#ifdef LTPMG_GPULAUNCHER_PRINT
                std::cout << row_1 << ":";
                for (uint32_t i = 0; i < table_info[table_ID].string_size * table_info[table_ID].string_length; ++i) {
                    std::cout << std::hex << string_data[i] << " ";
                }
                std::cout << std::endl;
                std::cout << std::dec;
#endif
                delete[] int_data;
                delete[] string_data;
                delete[] double_data;
                break;
            }
            default: {
                break;
            }
        }
    }
}

void execute_on_CPU(std::shared_ptr<Param> param,
                    GPUdatabase *gpudatabase,
                    GPUquery *gpuquery) {
    std::cout << "start execute_on_CPU" << std::endl;

    Global_Txn *global_txn = gpuquery->get_txn_for_cpu();
    Global_Txn_Info *global_txn_info_h = gpuquery->get_txn_info_for_cpu();
    // Global_Table_Strategy *strategy = gpudatabase->get_strategy_for_cpu();
    Global_Table *table = gpudatabase->get_table_for_cpu();
    Global_Table_Info *table_info = gpudatabase->get_table_info_for_cpu(); {
        TaskQueue taskQueue;

        uint32_t cur_sub_txn = 0;
        for (uint32_t i = 0; i < param->get_sub_txn_size(); i++) {
            uint32_t global_txn_info_ID = global_txn[i].global_txn_info_ID;
            uint32_t cur_sub_txn_size = global_txn_info_h[global_txn_info_ID].cur_subtxn_cnt;
            bool is_op_cpu = false;
            for (uint32_t j = 0; j < cur_sub_txn_size; j++) {
                uint32_t dest_device = global_txn[i].subtxn.dest_device;
                if (dest_device == 0xffffffff) {
                    is_op_cpu = true;
                    break;
                }
            }
            if (is_op_cpu) {
                ++cur_sub_txn;
                taskQueue.addTask([param,cur_sub_txn,global_txn,global_txn_info_h,i,table,table_info]() {
                        execute_on_thread(param, cur_sub_txn, global_txn, global_txn_info_h, i, table, table_info);
                    }
                );
            }
        }
        // uint32_t numThreads = std::thread::hardware_concurrency();
        // std::cout << "cur_sub_txn_cnt: " << cur_sub_txn << std::endl;
        if (cur_sub_txn > 0) {
            uint32_t numThreads = 16;
            ThreadPool threadPool(numThreads);
            Semaphore semephore(cur_sub_txn);

            std::thread taskProcessor(processTasks, std::ref(taskQueue), std::ref(threadPool), std::ref(semephore));

            taskProcessor.join();
        }
    }
    std::cout << "end execute_on_CPU" << std::endl;
}

void GPUlauncher::txn_kernel_launcher(std::shared_ptr<Param> param, GPUdatabase *gpudatabase,
                                      GPUquery *gpuquery) {
    std::cout << "start gpulauncher.cu GPUlauncher::txn_kernel_launcher()" << std::endl;

    hipStream_t *streams;
    streams = new hipStream_t[param->device_cnt];

    for (uint32_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipSetDevice(param->device_IDs[i]));
        CHECK(hipStreamCreate(&streams[i]));
    }

    std::cout << "start execute" << std::endl;

    long long start_all = gpu_current_time();

    // cpu_exec = std::thread(&execute_on_CPU, param, gpudatabase, gpuquery);

    // std::cout << "param->get_sub_txn_size():" << param->get_sub_txn_size() << std::endl;

    long long start_executor = gpu_current_time();


    for (uint32_t i = 0; i < param->device_cnt; ++i) {
        // execute
        CHECK(hipSetDevice(param->device_IDs[i]));
        txn_executor<<<512, 512, 0, streams[i]>>>(i,
                                                  param->device_cnt,
                                                  param->get_sub_txn_size(),
                                                  gpudatabase->get_table_info(i),
                                                  gpudatabase->get_table(i),
                                                  gpudatabase->get_index(i),
                                                  gpudatabase->get_meta(i),
                                                  gpudatabase->get_strategy(i),
                                                  gpuquery->get_txn_info(i),
                                                  gpuquery->get_txn(i),
                                                  gpuquery->get_txn_exec(i),
                                                  gpuquery->get_txn_result(i),
                                                  gpuquery->get_exec_param(i),
                                                  gpuquery->get_aux_struct(i),
                                                  gpuquery->get_data_packet(i));
    }

    for (uint32_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipStreamSynchronize(streams[i]));
    }

    long long end_executor = gpu_current_time();

    // std::cout << "end execute" << std::endl;

    long long start_transfer = gpu_current_time();

    if (param->device_cnt > 1) {
        gpuquery->transfer_data_packet(param, streams);
    }

    for (uint32_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipStreamSynchronize(streams[i]));
    }

    long long end_transfer = gpu_current_time();

    // std::cout << "start merge" << std::endl;

    long long start_merge = gpu_current_time();


    if (param->device_cnt > 1) {
        for (uint32_t i = 0; i < param->device_cnt; ++i) {
            // merge
            CHECK(hipSetDevice(param->device_IDs[i]));
            txn_merge<<<512, 512, 0, streams[i]>>>(i,
                                                   param->device_cnt,
                                                   param->get_datapacket_size(),
                                                   gpudatabase->get_table_info(i),
                                                   gpudatabase->get_table(i),
                                                   gpudatabase->get_index(i),
                                                   gpudatabase->get_meta(i),
                                                   gpudatabase->get_strategy(i),
                                                   gpuquery->get_txn_info(i),
                                                   gpuquery->get_txn(i),
                                                   gpuquery->get_txn_exec(i),
                                                   gpuquery->get_txn_result(i),
                                                   gpuquery->get_exec_param(i),
                                                   gpuquery->get_aux_struct(i),
                                                   gpuquery->get_data_packet(i));
        }

        for (uint32_t i = 0; i < param->device_cnt; ++i) {
            CHECK(hipStreamSynchronize(streams[i]));
        }
    }
    long long end_merge = gpu_current_time();
    // std::cout << "end merge" << std::endl;

    long long start_analyse = gpu_current_time();

    for (uint32_t j = 0; j < param->table_cnt; ++j) {
        for (uint32_t i = 0; i < param->device_cnt; ++i) {
            // txn_prefix_offset
            CHECK(hipSetDevice(param->device_IDs[i]));
            txn_prefix_offset<<<512, 512, 0, streams[i]>>>(i,
                                                           param->device_cnt,
                                                           j,
                                                           gpudatabase->get_table_info(i),
                                                           gpudatabase->get_table(i),
                                                           gpudatabase->get_index(i),
                                                           gpudatabase->get_meta(i),
                                                           gpudatabase->get_strategy(i),
                                                           gpuquery->get_txn_info(i),
                                                           gpuquery->get_txn(i),
                                                           gpuquery->get_txn_exec(i),
                                                           gpuquery->get_txn_result(i),
                                                           gpuquery->get_exec_param(i),
                                                           gpuquery->get_aux_struct(i),
                                                           gpuquery->get_data_packet(i));
        }
    }
    for (uint32_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipStreamSynchronize(streams[i]));
    }

    // std::cout << "start analyse" << std::endl;
#ifdef LTPMG_GPULAUNCHER_SCAN_OPT_BITMAP_POPULAR
    for (uint32_t j = 0; j < param->table_cnt; ++j) {
        for (uint32_t i = 0; i < param->device_cnt; ++i) {
            // analyse
            CHECK(hipSetDevice(param->device_IDs[i]));
            prefix_bitmap<<<512, 512, 0, streams[i]>>>(i,
                                                       j,
                                                       param->get_benchmark_ID(),
                                                       gpudatabase->get_table_info(i),
                                                       gpudatabase->get_table(i),
                                                       gpudatabase->get_meta(i),
                                                       gpuquery->get_txn(i),
                                                       gpuquery->get_exec_param(i),
                                                       gpuquery->get_aux_struct(i),
                                                       gpuquery->get_data_packet(i));
        }
    }

    for (uint32_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipStreamSynchronize(streams[i]));
    }

    long long analyse_node0 = gpu_current_time();

#ifdef LTPMG_GPULAUNCHER_BLOCK_SCAN_OPT_SHM_BITMAP_POPULAR
    for (uint32_t j = 0; j < param->table_cnt; ++j) {
        for (uint32_t i = 0; i < param->device_cnt; ++i) {
            // analyse
            CHECK(hipSetDevice(param->device_IDs[i]));
            partSum_bitmap<<<512, 512, 0, streams[i]>>>(i,
                                                        j,
                                                        param->get_benchmark_ID(),
                                                        gpudatabase->get_table_info(i),
                                                        gpudatabase->get_table(i),
                                                        gpudatabase->get_meta(i),
                                                        gpuquery->get_txn(i),
                                                        gpuquery->get_exec_param(i),
                                                        gpuquery->get_aux_struct(i),
                                                        gpuquery->get_data_packet(i));
        }
    }
    // for (uint32_t i = 0; i < param->device_cnt; ++i) {
    //     CHECK(hipStreamSynchronize(streams[i]));
    // }
#endif

    for (uint32_t j = 0; j < param->table_cnt; ++j) {
        for (uint32_t i = 0; i < param->device_cnt; ++i) {
            // analyse
            CHECK(hipSetDevice(param->device_IDs[i]));
            compact_bitmark<<<512, 512, 0, streams[i]>>>(i,
                                                         j,
                                                         param->get_benchmark_ID(),
                                                         gpudatabase->get_table_info(i),
                                                         gpudatabase->get_table(i),
                                                         gpudatabase->get_meta(i),
                                                         gpuquery->get_txn(i),
                                                         gpuquery->get_exec_param(i),
                                                         gpuquery->get_aux_struct(i),
                                                         gpuquery->get_data_packet(i));
        }
    }
#endif


    for (uint32_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipStreamSynchronize(streams[i]));
    }
    long long analyse_node1 = gpu_current_time();

    for (uint32_t j = 0; j < param->bitmap_table_cnt; ++j) {
        for (uint32_t i = 0; i < param->device_cnt; ++i) {
            // analyse
            CHECK(hipSetDevice(param->device_IDs[i]));
            txn_analyze_popular<<<512, 512, 0, streams[i]>>>(i,
                                                             j,
                                                             param->get_benchmark_ID(),
                                                             gpudatabase->get_table_info(i),
                                                             gpudatabase->get_table(i),
                                                             gpudatabase->get_index(i),
                                                             gpudatabase->get_meta(i),
                                                             gpudatabase->get_strategy(i),
                                                             gpuquery->get_txn_info(i),
                                                             gpuquery->get_txn(i),
                                                             gpuquery->get_txn_exec(i),
                                                             gpuquery->get_txn_result(i),
                                                             gpuquery->get_exec_param(i),
                                                             gpuquery->get_aux_struct(i),
                                                             gpuquery->get_data_packet(i));
        }
    }

    for (uint32_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipStreamSynchronize(streams[i]));
    }
    long long analyse_node2 = gpu_current_time();

    for (uint32_t i = 0; i < param->device_cnt; ++i) {
        // analyse
        CHECK(hipSetDevice(param->device_IDs[i]));
        txn_analyze_regular<<<512, 512, 0, streams[i]>>>(i,
                                                         param->device_cnt,
                                                         param->get_benchmark_ID(),
                                                         gpudatabase->get_table_info(i),
                                                         gpudatabase->get_table(i),
                                                         gpudatabase->get_index(i),
                                                         gpudatabase->get_meta(i),
                                                         gpudatabase->get_strategy(i),
                                                         gpuquery->get_txn_info(i),
                                                         gpuquery->get_txn(i),
                                                         gpuquery->get_txn_exec(i),
                                                         gpuquery->get_txn_result(i),
                                                         gpuquery->get_exec_param(i),
                                                         gpuquery->get_aux_struct(i),
                                                         gpuquery->get_data_packet(i));
    }

    for (uint32_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipStreamSynchronize(streams[i]));
    }
    long long end_analyse = gpu_current_time();
    // std::cout << "end analyse" << std::endl;

    // std::cout << "start commit" << std::endl;
    long long start_commit = gpu_current_time();
    for (uint32_t j = 0; j < param->table_cnt; ++j) {
        for (uint32_t i = 0; i < param->device_cnt; ++i) {
            // commit
            CHECK(hipSetDevice(param->device_IDs[i]));
            filter_commit<<<512, 512, 0, streams[i]>>>(i,
                                                       param->device_cnt,
                                                       j,
                                                       gpudatabase->get_table_info(i),
                                                       gpudatabase->get_table(i),
                                                       gpudatabase->get_index(i),
                                                       gpudatabase->get_meta(i),
                                                       gpudatabase->get_strategy(i),
                                                       gpuquery->get_txn_info(i),
                                                       gpuquery->get_txn(i),
                                                       gpuquery->get_txn_exec(i),
                                                       gpuquery->get_txn_result(i),
                                                       gpuquery->get_exec_param(i),
                                                       gpuquery->get_aux_struct(i),
                                                       gpuquery->get_data_packet(i));
        }
    }

    for (uint32_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipStreamSynchronize(streams[i]));
    }
    long long commit_node0 = gpu_current_time();

    for (uint32_t j = 0; j < param->table_cnt; ++j) {
        for (uint32_t i = 0; i < param->device_cnt; ++i) {
            // commit
            CHECK(hipSetDevice(param->device_IDs[i]));
            txn_commit<<<512, 512, 0, streams[i]>>>(i,
                                                    param->device_cnt,
                                                    j,
                                                    gpudatabase->get_table_info(i),
                                                    gpudatabase->get_table(i),
                                                    gpudatabase->get_index(i),
                                                    gpudatabase->get_meta(i),
                                                    gpudatabase->get_strategy(i),
                                                    gpuquery->get_txn_info(i),
                                                    gpuquery->get_txn(i),
                                                    gpuquery->get_txn_exec(i),
                                                    gpuquery->get_txn_result(i),
                                                    gpuquery->get_exec_param(i),
                                                    gpuquery->get_aux_struct(i),
                                                    gpuquery->get_data_packet(i));
        }
    }

    for (uint32_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipStreamSynchronize(streams[i]));
    }
    long long end_commit = gpu_current_time();

    for (uint32_t i = 0; i < param->device_cnt; ++i) {
        CHECK(hipStreamDestroy(streams[i]));
    }

    // cpu_exec.join();

    long long end_all = gpu_current_time();

    std::cout << "end commit" << std::endl;

    delete[] streams;

    float cost_all = gpu_duration(start_all, end_all);
    float cost_gpu = gpu_duration(start_executor, end_commit);
    float cost_executor = gpu_duration(start_executor, end_executor);
    float cost_transfer = gpu_duration(start_transfer, end_transfer);
    float cost_merge = gpu_duration(start_merge, end_merge);
    float cost_analyse = gpu_duration(start_analyse, end_analyse);
    float cost_commit = gpu_duration(start_commit, end_commit);
    param->result.cost = cost_gpu;
    // std::cout<<"param->result.cost: "<<param->result.cost<<std::endl;
    std::cout << "cost_all: " << cost_all << " s." << std::endl;
    std::cout << "cost_gpu: " << cost_gpu << " s." << std::endl;
    std::cout << "cost_executor: " << cost_executor << " s." << std::endl;
    std::cout << "cost_transfer: " << cost_transfer << " s." << std::endl;
    std::cout << "cost_merge: " << cost_merge << " s." << std::endl;
    std::cout << "cost_analyse: " << cost_analyse << " s." << std::endl;
    std::cout << "cost_commit: " << cost_commit << " s." << std::endl;
    // float node0 = gpu_duration(start_analyse, analyse_node0);
    // float node1 = gpu_duration(analyse_node0, analyse_node1);
    float node2 = gpu_duration(analyse_node1, analyse_node2);
    float node3 = gpu_duration(analyse_node2, end_analyse);
    // std::cout << "analyse_node0: " << node0 << std::endl;
    // std::cout << "analyse_node1: " << node1 << std::endl;
    std::cout << "analyse_node2: " << node2 << std::endl;
    std::cout << "analyse_node3: " << node3 << std::endl;
    float node4 = gpu_duration(start_commit, commit_node0);
    float node5 = gpu_duration(commit_node0, end_commit);
    std::cout << "commit_node0: " << node4 << std::endl;
    std::cout << "commit_node1: " << node5 << std::endl;
    std::cout << "TPS: " << param->batch_size / cost_all << " ." << std::endl;
    std::cout << "TPS: " << param->batch_size / cost_gpu << " ." << std::endl;
#ifdef LTPMG_GPUQUERY_TRANSFER_GROUP
    std::cout << "Transfer data size: " << sizeof(Global_Data_Packet) * param->get_datapacket_size()/2 *
            (param->device_cnt) / (1 << 20) << " MB." << std::endl;
    std::cout << "Bandwidth: " <<
            sizeof(Global_Data_Packet) * param->get_datapacket_size()/2 * (param->device_cnt) /
            (cost_transfer * (1 << 30)) << " GB/s." << std::endl;
#endif

#ifndef LTPMG_GPUQUERY_TRANSFER_GROUP
    std::cout << "Transfer data size: " << sizeof(Global_Data_Packet) * param->get_datapacket_size() *
            (param->device_cnt-1) / (1 << 20) << " MB." << std::endl;
    std::cout << "Bandwidth: " <<
            sizeof(Global_Data_Packet) * param->get_datapacket_size() * (param->device_cnt-1) /
            (cost_transfer * (1 << 30)) << " GB/s." << std::endl;
#endif
    std::cout << "end gpulauncher.cu GPUlauncher::txn_kernel_launcher()" << std::endl;
}

__global__ void mergeSort(uint32_t *unsorted, uint32_t *sorted, uint32_t arrSize) {
    const uint32_t laneID = threadIdx.x % 32;
    __syncwarp();
    for (uint32_t i = 2; i <= (1 << 32 - __clz(arrSize)); i *= 2) {
        for (uint32_t start = i * laneID; start < arrSize; start += i * 32) {
            uint32_t size = i;
            if (start + size >= arrSize) {
                size = arrSize - start;
            }
            if (start < arrSize) {
                uint32_t sub_size = i >> 1;
                uint32_t offset = start;
                uint32_t left = start;
                uint32_t right = start + sub_size;
                while (left < start + sub_size && right < start + size) {
                    if (unsorted[left] < unsorted[right]) {
                        sorted[offset] = unsorted[left];
                        ++left;
                    } else {
                        sorted[offset] = unsorted[right];
                        ++right;
                    }
                    ++offset;
                }
                while (left < start + sub_size) {
                    sorted[offset] = unsorted[left];
                    ++left;
                    ++offset;
                }
                while (right < start + size) {
                    sorted[offset] = unsorted[right];
                    ++right;
                    ++offset;
                }
            }
        }
        for (uint32_t j = laneID; j < arrSize; j += 32) {
            unsorted[j] = sorted[j];
        }
        __syncwarp();
    }
    __syncwarp();
}

__global__ void partSum(uint32_t *array_d,
                        uint32_t *sorted_array_d,
                        uint32_t seg_Size,
                        uint32_t slice_size) {
    const uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    const uint32_t thSize = blockDim.x * gridDim.x;
    const uint32_t wpID = threadIdx.x >> 5;
    const uint32_t laneID = threadIdx.x & 0x1f;
    const uint32_t SHM_start = threadIdx.x & 0xffffffe0;

    __shared__ uint32_t bitmap_tmp[512];
    __shared__ uint32_t partSum[32];
    __shared__ uint32_t block_sum[1];
    for (uint32_t i = blockIdx.x; i < slice_size; i += gridDim.x) {
        if (threadIdx.x == 0) {
            block_sum[0] = 0;
        }
        __syncthreads();
        const uint32_t bitmap_start = i * seg_Size;
        for (uint32_t j = threadIdx.x; j < seg_Size; j += blockDim.x) {
            const uint32_t cur_block_sum = block_sum[0];
            if (wpID == 0) {
                partSum[laneID] = 0;
            }
            if (j < seg_Size) {
                bitmap_tmp[threadIdx.x] = array_d[bitmap_start + j];
            } else {
                bitmap_tmp[threadIdx.x] = 0;
            }
            __syncwarp();
            uint32_t tmp_res = bitmap_tmp[threadIdx.x];
            for (uint32_t k = 16; k > 0; k >>= 1) {
                uint32_t remote = __shfl_up_sync(0xffffffff, tmp_res, k);
                if (laneID >= k) {
                    tmp_res += remote;
                }
            }
            if (laneID == 31) {
                partSum[wpID] = tmp_res;
                printf("partSum[%d]:%d\n", wpID, partSum[wpID]);
            }
            __syncthreads();
            if (wpID == 0) {
                uint32_t tmp_part_res = partSum[laneID];
                // printf("laneID:%d,tmp_part_res:%d\n", laneID, tmp_part_res);
                for (uint32_t k = 16; k > 0; k >>= 1) {
                    uint32_t remote = __shfl_up_sync(0xffffffff, tmp_part_res, k);
                    if (laneID >= k) {
                        tmp_part_res += remote;
                    }
                }
                // printf("laneID:%d,tmp_part_res:%d\n", laneID, tmp_part_res);
                if (laneID == 31) {
                    block_sum[0] = tmp_part_res;
                    // printf("tmp_part_res:%d\n", tmp_part_res);
                }
                partSum[laneID] = tmp_part_res - partSum[laneID];
            }
            __syncthreads();
            tmp_res = tmp_res + partSum[wpID] + cur_block_sum - bitmap_tmp[threadIdx.x];
            if (j < seg_Size) {
                sorted_array_d[bitmap_start + j] = tmp_res;
            } else {
                sorted_array_d[bitmap_start + j] = 0;
            }
        }
        __syncthreads();
    }
}

void GPUlauncher::txn_executor_launcher(std::shared_ptr<Param> param, GPUdatabase *gpudatabase,
                                        GPUquery *gpuquery) {
    std::cout << "start gpulauncher.cu GPUlauncher::txn_executor_launcher()" << std::endl;
    txn_kernel_launcher(param, gpudatabase, gpuquery);
    std::cout << "end gpulauncher.cu GPUlauncher::txn_executor_launcher()" << std::endl;
#ifdef LTPMG_GPULAUNCHER_TEST_MERGESORT
    CHECK(hipSetDevice(0));
    uint32_t array_size = 1000000;
    uint32_t h_unsorted[array_size];
    for (uint32_t i = 0; i < array_size; ++i) {
        h_unsorted[i] = array_size - i;
    }
    uint32_t h_sorted[array_size];
    uint32_t *d_unsorted;
    uint32_t *d_sorted;
    CHECK(hipMalloc((void**)&d_unsorted, sizeof(uint32_t) * array_size));
    CHECK(hipMalloc((void**)&d_sorted, sizeof(uint32_t) * array_size));
    CHECK(hipMemset(d_unsorted, 0, sizeof(uint32_t) * array_size));
    CHECK(hipMemset(d_sorted, 0, sizeof(uint32_t) * array_size));
    CHECK(hipMemcpy(d_unsorted,h_unsorted,sizeof(uint32_t)*array_size,hipMemcpyHostToDevice));
    mergeSort<<<1,32>>>(d_unsorted, d_sorted, array_size);
    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(h_sorted,d_sorted ,sizeof(uint32_t) * array_size,hipMemcpyDeviceToHost));
    for (uint32_t i = 0; i < array_size; ++i) {
        if (h_sorted[i] != i + 1) {
            std::cout << i << "," << h_sorted[i] << std::endl;
        }
        // std::cout << i << "," << h_sorted[i] << std::endl;
    }
    std::cout << std::endl;
    CHECK(hipFree(d_unsorted));
    CHECK(hipFree(d_sorted));
#endif


#ifdef LTPMG_GPULAUNCHER_TEST_PREFIXSUM
    CHECK(hipSetDevice(0));
    hipStream_t stream;
    CHECK(hipStreamCreate(&stream));
    uint32_t array_size = 128;
    uint32_t segment_size = 64;
    uint32_t *h_unsorted;
    uint32_t *cpu_result;
    CHECK(hipHostAlloc((void**)&h_unsorted,sizeof(uint32_t)*array_size,hipHostMallocDefault));
    CHECK(hipHostAlloc((void**)&cpu_result,sizeof(uint32_t)*array_size,hipHostMallocDefault));
    uint32_t tmp = 0;
    for (uint32_t i = 0; i < array_size; ++i) {
        h_unsorted[i] = i % segment_size;
    }
    for (uint32_t i = 0; i < array_size; ++i) {
        if (i % segment_size == 0) {
            tmp = 0;
        }
        tmp += h_unsorted[i];
        cpu_result[i] = tmp;
    }
    for (uint32_t i = 0; i < array_size; ++i) {
        std::cout << h_unsorted[i] << " ";
    }
    std::cout << std::endl;
    for (uint32_t i = 0; i < array_size; ++i) {
        cpu_result[i] -= h_unsorted[i];
        std::cout << cpu_result[i] << " ";
    }
    std::cout << std::endl;
    uint32_t *h_sorted;
    uint32_t *d_unsorted;
    uint32_t *d_sorted;
    CHECK(hipHostAlloc((void**)&h_sorted,sizeof(uint32_t)*array_size,hipHostMallocDefault));
    CHECK(hipMalloc((void**)&d_unsorted, sizeof(uint32_t) * array_size));
    CHECK(hipMalloc((void**)&d_sorted, sizeof(uint32_t) * array_size));
    CHECK(hipMemsetAsync(d_sorted, 0, sizeof(uint32_t) * array_size,stream));
    CHECK(hipMemcpyAsync(d_unsorted,h_unsorted,sizeof(uint32_t)*array_size,hipMemcpyHostToDevice,stream));
    partSum<<<1,32,0,stream>>>(d_unsorted, d_sorted, segment_size, array_size / segment_size);
    // CHECK(hipStreamSynchronize(stream));
    CHECK(hipMemcpyAsync(h_sorted,d_sorted ,sizeof(uint32_t) * array_size,hipMemcpyDeviceToHost,stream));
    for (uint32_t i = 0; i < array_size; ++i) {
        // if (h_sorted[i] != i + 1) {
        //     std::cout << i << "," << h_sorted[i] << std::endl;
        // }
        std::cout << h_sorted[i] << " ";
    }
    std::cout << std::endl;
    CHECK(hipFree(d_unsorted));
    CHECK(hipFree(d_sorted));
    CHECK(hipHostFree(h_sorted));
    CHECK(hipHostFree(cpu_result));
    CHECK(hipHostFree(h_unsorted));
    CHECK(hipStreamDestroy(stream));
#endif


#ifdef LTPMG_GPULAUNCHER_TEST_MEMCPYP2P

    for (uint32_t bitsize = 20; bitsize < 28; ++bitsize) {
        hipStream_t *streams = new hipStream_t[param->device_cnt];
        for (uint32_t i = 0; i < param->device_cnt; ++i) {
            CHECK(hipSetDevice(i));
            CHECK(hipStreamCreate(&streams[i]));
        }
        uint32_t array_size = 1 << bitsize;
        uint32_t flag = 2;
        uint32_t *array_0 = new uint32_t[array_size];
        uint32_t *array_1 = new uint32_t[array_size];
        uint32_t *array_2 = new uint32_t[array_size];
        uint32_t *array_3 = new uint32_t[array_size];

        memset(array_0, 0x00, sizeof(uint32_t) * array_size);
        memset(array_1, 0xff, sizeof(uint32_t) * array_size);
        memset(array_2, 0x0f, sizeof(uint32_t) * array_size);
        memset(array_3, 0xf0, sizeof(uint32_t) * array_size);
        uint32_t *array_0_d;
        uint32_t *array_1_d;
        uint32_t *array_2_d;
        uint32_t *array_3_d;

        CHECK(hipSetDevice(param->device_IDs[0]));
        CHECK(hipMalloc((void**)&array_0_d, sizeof(uint32_t) * array_size));
        CHECK(hipSetDevice(param->device_IDs[1]));
        CHECK(hipMalloc((void**)&array_1_d, sizeof(uint32_t) * array_size));
        CHECK(hipSetDevice(param->device_IDs[2]));
        CHECK(hipMalloc((void**)&array_2_d, sizeof(uint32_t) * array_size));
        CHECK(hipSetDevice(param->device_IDs[3]));
        CHECK(hipMalloc((void**)&array_3_d, sizeof(uint32_t) * array_size));

        CHECK(hipSetDevice(param->device_IDs[0]));
        CHECK(hipMemcpyAsync(array_0_d,array_0,sizeof(uint32_t) * array_size,hipMemcpyHostToDevice,streams[0]));
        CHECK(hipSetDevice(param->device_IDs[1]));
        CHECK(hipMemcpyAsync(array_1_d,array_1,sizeof(uint32_t) * array_size,hipMemcpyHostToDevice,streams[1]));
        CHECK(hipSetDevice(param->device_IDs[2]));
        CHECK(hipMemcpyAsync(array_2_d,array_2,sizeof(uint32_t) * array_size,hipMemcpyHostToDevice,streams[2]));
        CHECK(hipSetDevice(param->device_IDs[3]));
        CHECK(hipMemcpyAsync(array_3_d,array_3,sizeof(uint32_t) * array_size,hipMemcpyHostToDevice,streams[3]));
        for (uint32_t i = 0; i < param->device_cnt; ++i) {
            CHECK(hipStreamSynchronize(streams[i]));
        }

        long long start = gpu_current_time();
        if (flag == 2) {
            CHECK(hipMemcpyPeerAsync(array_0_d+array_size/2,param->device_IDs[0],
                array_1_d,param->device_IDs[1],
                array_size/2,streams[1]));

            CHECK(hipMemcpyPeerAsync(array_1_d+array_size/2,param->device_IDs[1],
                array_0_d,param->device_IDs[0],
                array_size/2,streams[0]));
        } else if (flag == 4) {
            // CHECK(hipMemcpyPeerAsync(array_0_d+array_size/4,param->device_IDs[0],
            //     array_1_d,param->device_IDs[1],
            //     array_size/4,streams[1]));
            //
            // CHECK(hipMemcpyPeerAsync(array_1_d+array_size/4,param->device_IDs[1],
            //     array_0_d,param->device_IDs[0],
            //     array_size/4,streams[0]));
            //
            // CHECK(hipMemcpyPeerAsync(array_2_d+array_size/4,param->device_IDs[2],
            //     array_3_d,param->device_IDs[3],
            //     array_size/4,streams[3]));
            //
            // CHECK(hipMemcpyPeerAsync(array_3_d+array_size/4,param->device_IDs[3],
            //     array_2_d,param->device_IDs[2],
            //     array_size/4,streams[2]));
            //
            // for (uint32_t i = 0; i < param->device_cnt; ++i) {
            //     CHECK(hipStreamSynchronize(streams[i]));
            // }

            CHECK(hipMemcpyPeerAsync(array_2_d+array_size/2,param->device_IDs[2],
                array_0_d,param->device_IDs[0],
                array_size/2,streams[0]));

            CHECK(hipMemcpyPeerAsync(array_3_d+array_size/2,param->device_IDs[3],
                array_1_d,param->device_IDs[1],
                array_size/2,streams[1]));

            CHECK(hipMemcpyPeerAsync(array_0_d+array_size/2,param->device_IDs[0],
                array_2_d,param->device_IDs[2],
                array_size/2,streams[2]));

            CHECK(hipMemcpyPeerAsync(array_1_d+array_size/2,param->device_IDs[1],
                array_3_d,param->device_IDs[3],
                array_size/2,streams[3]));
        }

        for (uint32_t i = 0; i < param->device_cnt; ++i) {
            CHECK(hipStreamSynchronize(streams[i]));
        }

        long long end = gpu_current_time();
        float cost = gpu_duration(start, end);
        float size = 0.0;
        if (flag == 2) {
            size = (float) 1 * array_size * sizeof(uint32_t) / (1 << 30);
        } else if (flag == 4) {
            size = (float) 2 * array_size * sizeof(uint32_t) / (1 << 30);
        }
        std::cout << "array_size:" << (float) array_size / (1 << 20) << " MB." << std::endl;
        std::cout << "cost:" << cost << " s." << std::endl;
        std::cout << "size:" << size << " GB." << std::endl;
        std::cout << "bandwidth:" << size / cost << " GB/s." << std::endl;;
        for (uint32_t i = 0; i < param->device_cnt; ++i) {
            CHECK(hipStreamDestroy(streams[i]));
        }
        delete[] streams;

        CHECK(hipFree(array_0_d));
        CHECK(hipFree(array_1_d));
        CHECK(hipFree(array_2_d));
        CHECK(hipFree(array_3_d));
        delete[] array_0;
        delete[] array_1;
        delete[] array_2;
        delete[] array_3;
    }
#endif
}

__device__ void merge(uint32_t device_ID,
                      uint32_t cur_datapacket,
                      Global_Table_Info *table_info,
                      Global_Table *tables,
                      Global_Table_Index *indexes,
                      Global_Table_Meta *metainfo,
                      Global_Table_Strategy *strategy,
                      Global_Txn_Info *txn_info,
                      Global_Txn *txn,
                      Global_Txn_Exec *txn_exec,
                      Global_Txn_Result *txn_result,
                      Global_Txn_Exec_Param *exec_param,
                      Global_Txn_Aux_Struct *aux_struct,
                      Global_Data_Packet *data_packet) {
    uint32_t cur_mark = data_packet[cur_datapacket].mark;
    if (cur_mark != 0xffffffff) {
        uint32_t cur_txn = cur_mark;
        uint32_t type = txn[cur_txn].subtxn.type;
        uint32_t tid = txn[cur_txn].subtxn.TID;
        uint32_t tableID = txn[cur_txn].subtxn.table_ID;
        uint32_t row_1 = txn[cur_txn].subtxn.dest_Row_1;
        uint32_t row_2 = txn[cur_txn].subtxn.dest_Row_2;
        uint32_t ispopular = txn[cur_txn].subtxn.ispopular;
        uint32_t row_start = metainfo[tableID].row_start;
        uint32_t row_end = metainfo[tableID].row_end;
        uint32_t dest_device = txn[cur_txn].subtxn.dest_device;
        if (type == 0) {
            if (dest_device == device_ID) {
                if (ispopular == 0 || ispopular == 1) {
                    register_txn_exec(device_ID, type, cur_txn, txn_exec);

                    register_cc(cur_txn, ispopular, tableID, row_1, tid,
                                table_info, tables, indexes, metainfo, strategy, txn_info, txn, txn_exec,
                                txn_result, exec_param, aux_struct, data_packet);
                }
            }
        } else if (type == 1) {
            if (dest_device == device_ID) {
                register_txn_exec(device_ID, type, cur_txn, txn_exec);

                register_cc(cur_txn, ispopular, tableID, row_1, tid,
                            table_info, tables, indexes, metainfo, strategy, txn_info, txn, txn_exec,
                            txn_result, exec_param, aux_struct, data_packet);
            }
        } else if (type == 2) {
            if (dest_device == device_ID) {
                register_txn_exec(device_ID, type, cur_txn, txn_exec);

                register_cc(cur_txn, ispopular, tableID, row_1, tid,
                            table_info, tables, indexes, metainfo, strategy, txn_info, txn, txn_exec,
                            txn_result, exec_param, aux_struct, data_packet);
            }
        } else if (type == 4) {
            bool contain_local = false;
            for (uint32_t row = row_1; row < row_2; ++row) {
                if (row >= row_start && row < row_end) {
                    contain_local = true;
                    register_cc(cur_txn, ispopular, tableID, row, tid,
                                table_info, tables, indexes, metainfo, strategy, txn_info, txn, txn_exec,
                                txn_result, exec_param, aux_struct, data_packet);
                }
            }
            if (contain_local) {
                register_txn_exec(device_ID, type, cur_txn, txn_exec);
            }
        } else if (type == 3) {
            if (dest_device == device_ID) {
                register_txn_exec(device_ID, type, cur_txn, txn_exec);

                register_cc(cur_txn, ispopular, tableID, row_1, tid,
                            table_info, tables, indexes, metainfo, strategy, txn_info, txn, txn_exec,
                            txn_result, exec_param, aux_struct, data_packet);
            }
        }
    }
}

__global__ void txn_merge(uint32_t device_ID,
                          uint32_t device_cnt,
                          uint32_t sub_txn_size,
                          Global_Table_Info *table_info,
                          Global_Table *tables,
                          Global_Table_Index *indexes,
                          Global_Table_Meta *metainfo,
                          Global_Table_Strategy *strategy,
                          Global_Txn_Info *txn_info,
                          Global_Txn *txn,
                          Global_Txn_Exec *txn_exec,
                          Global_Txn_Result *txn_result,
                          Global_Txn_Exec_Param *exec_param,
                          Global_Txn_Aux_Struct *aux_struct,
                          Global_Data_Packet *data_packet) {
    const uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    const uint32_t thSize = blockDim.x * gridDim.x;
    const uint32_t size = sub_txn_size; // * device_cnt;
#ifndef LTPMG_GPUQUERY_TRANSFER_GROUP
    uint32_t cur = thID + sub_txn_size / device_cnt;
#endif

#ifdef LTPMG_GPUQUERY_TRANSFER_GROUP
    uint32_t cur = thID + sub_txn_size / 2;
#endif
    while (cur < size) {
        merge(device_ID, cur, table_info, tables, indexes, metainfo, strategy, txn_info, txn,
              txn_exec, txn_result, exec_param, aux_struct, data_packet);
        cur += thSize;
    }
}

__global__ void txn_prefix_offset(uint32_t device_ID,
                                  uint32_t device_cnt,
                                  uint32_t tableID,
                                  Global_Table_Info *table_info,
                                  Global_Table *tables,
                                  Global_Table_Index *indexes,
                                  Global_Table_Meta *metainfo,
                                  Global_Table_Strategy *strategy,
                                  Global_Txn_Info *txn_info,
                                  Global_Txn *txn,
                                  Global_Txn_Exec *txn_exec,
                                  Global_Txn_Result *txn_result,
                                  Global_Txn_Exec_Param *exec_param,
                                  Global_Txn_Aux_Struct *aux_struct,
                                  Global_Data_Packet *data_packet) {
    uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t thSize = blockDim.x * gridDim.x;
    uint32_t cur = thID;
    uint32_t size = __ldg(&metainfo[tableID].table_slice_size);

    while (cur < size) {
        uint32_t cur_cnt = __ldg(&aux_struct[tableID].cnt_TID[cur]);
        if (cur_cnt > 0) {
            uint32_t start_offset = atomicAdd(&aux_struct[tableID].used_rows_cnt, 1);
            // printf("start_offset:%d\n", start_offset);
            aux_struct[tableID].used_rows[start_offset] = cur;
            start_offset = atomicAdd(&aux_struct[tableID].mark_TID_start_offset, cur_cnt);
            aux_struct[tableID].mark_TID_offset[cur] = start_offset;
        }
        cur += thSize;
    }
}

// TODO: 加入popular handle的压缩bitmap优化

__global__ void prefix_bitmap(uint32_t device_ID,
                              uint32_t table_ID,
                              uint32_t benchmark,
                              Global_Table_Info *table_info,
                              Global_Table *tables,
                              Global_Table_Meta *metainfo,
                              Global_Txn *txn,
                              Global_Txn_Exec_Param *exec_param,
                              Global_Txn_Aux_Struct *aux_struct,
                              Global_Data_Packet *data_packet) {
    if (benchmark == 1 || benchmark == 4) {
    } else if (benchmark == 2 || benchmark == 3) {
        if (table_ID == 0 || table_ID == 1) {
        } else {
            return;
        }
    }
    // __shared__ uint32_t SHM_bitmap[512];
    // __shared__ uint32_t SHM_warp_max[16];
    // __shared__ uint32_t SHM_warp_row[16];
    // __shared__ uint32_t SHM_tmp_res[512];
    const uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    const uint32_t thSize = blockDim.x * gridDim.x;
    const uint32_t wpID = thID >> 5;
    const uint32_t wpSize = thSize >> 5;
    const uint32_t laneID = threadIdx.x & 0x1f;
    const uint32_t bitmap_mark_size = exec_param[0].bitmap_size * 32;
    const uint32_t table_slice_size = metainfo[table_ID].bitmap_row_slice_size;

    uint32_t tmp_res = 0;
    uint32_t cur_bit = 0;
    for (uint32_t i = wpID;
         i < (table_slice_size * bitmap_mark_size) >> 5;
         i += wpSize) {
        //scan warp
        // cur_bit = aux_struct[table_ID].bitmap[i];
        // tmp_res = cur_bit && (1 << laneID);
        cur_bit = aux_struct[table_ID].bitmap_mark[i * 32 + laneID];
        tmp_res = cur_bit != 0 ? 1 : 0;
        if (tmp_res > 0) {
            cooperative_groups::coalesced_group active = cooperative_groups::coalesced_threads();
            tmp_res = active.thread_rank();
            if (active.thread_rank() == active.num_threads() - 1) {
                aux_struct[table_ID].bitmap_tmp[i] = tmp_res;
            }
        }
        __syncwarp();
        aux_struct[table_ID].bitmap_mark_offset[i * 32 + laneID] = tmp_res; // - (cur_bit && (1 << laneID));
    }
}

__global__ void partSum_bitmap(uint32_t device_ID,
                               uint32_t table_ID,
                               uint32_t benchmark,
                               Global_Table_Info *table_info,
                               Global_Table *tables,
                               Global_Table_Meta *metainfo,
                               Global_Txn *txn,
                               Global_Txn_Exec_Param *exec_param,
                               Global_Txn_Aux_Struct *aux_struct,
                               Global_Data_Packet *data_packet) {
    if (benchmark == 1 || benchmark == 4) {
    } else if (benchmark == 2 || benchmark == 3) {
        if (table_ID == 0 || table_ID == 1) {
        } else {
            return;
        }
    }
    const uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    const uint32_t thSize = blockDim.x * gridDim.x;
    const uint32_t wpID = threadIdx.x >> 5;
    const uint32_t laneID = threadIdx.x & 0x1f;
    const uint32_t SHM_start = threadIdx.x & 0xffffffe0;
    const uint32_t bitmap_size = exec_param[0].bitmap_size;
    const uint32_t table_slice_size = metainfo[table_ID].bitmap_row_slice_size;
    __shared__ uint32_t bitmap_tmp[512];
    __shared__ uint32_t partSum[32];
    __shared__ uint32_t block_sum[1];

    for (uint32_t i = blockIdx.x; i < table_slice_size; i += gridDim.x) {
        if (threadIdx.x == 0) {
            block_sum[0] = 0;
        }
        __syncthreads();
        const uint32_t bitmap_start = i * bitmap_size;
        if (aux_struct[table_ID].bitmap_used_size[i] > 0) {
            for (uint32_t j = threadIdx.x; j < bitmap_size; j += blockDim.x) {
                const uint32_t cur_block_sum = block_sum[0];
                if (wpID == 0) {
                    partSum[laneID] = 0;
                }
                if (j < bitmap_size) {
                    bitmap_tmp[threadIdx.x] = aux_struct[table_ID].bitmap_tmp[bitmap_start + j];
                } else {
                    bitmap_tmp[threadIdx.x] = 0;
                }
                __syncwarp();
                uint32_t tmp_res = bitmap_tmp[threadIdx.x];
                for (uint32_t k = 16; k > 0; k >>= 1) {
                    uint32_t remote = __shfl_up_sync(0xffffffff, tmp_res, k);
                    if (laneID >= k) {
                        tmp_res += remote;
                    }
                }
                if (laneID == 31) {
                    partSum[wpID] = tmp_res;
                }
                __syncthreads();
                if (wpID == 0) {
                    uint32_t tmp_part_res = partSum[laneID];
                    for (uint32_t k = 16; k > 0; k >>= 1) {
                        uint32_t remote = __shfl_up_sync(0xffffffff, tmp_part_res, k);
                        if (laneID >= k) {
                            tmp_part_res += remote;
                        }
                    }
                    if (laneID == 31) {
                        block_sum[0] = tmp_part_res;
                    }
                    __syncwarp();
                    partSum[laneID] = tmp_part_res - partSum[laneID];
                }
                __syncthreads();
                tmp_res = tmp_res + partSum[wpID] + cur_block_sum - bitmap_tmp[threadIdx.x];
                if (j < bitmap_size) {
                    aux_struct[table_ID].bitmap_tmp[bitmap_start + j] = tmp_res;
                }
            }
        }
        __syncthreads();
    }
}

__global__ void compact_bitmark(uint32_t device_ID,
                                uint32_t table_ID,
                                uint32_t benchmark,
                                Global_Table_Info *table_info,
                                Global_Table *tables,
                                Global_Table_Meta *metainfo,
                                Global_Txn *txn,
                                Global_Txn_Exec_Param *exec_param,
                                Global_Txn_Aux_Struct *aux_struct,
                                Global_Data_Packet *data_packet) {
    if (benchmark == 1 || benchmark == 4) {
    } else if (benchmark == 2 || benchmark == 3) {
        if (table_ID == 0 || table_ID == 1) {
        } else {
            return;
        }
    }

    const uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    const uint32_t laneID = threadIdx.x & 0x1f;
    const uint32_t thSize = blockDim.x * gridDim.x;
    const uint32_t bitmap_size = exec_param[0].bitmap_size;
    const uint32_t bitmap_mark_size = exec_param[0].bitmap_size * 32;
    const uint32_t table_slice_size = metainfo[table_ID].bitmap_row_slice_size;
    __shared__ uint32_t SHM_part_res[512];
    uint32_t cur_row = 0;
    uint32_t cur_start = 0;
    uint32_t cur_ID = 0;
    uint32_t bitmap_mark_offset = 0;
    for (uint32_t i = thID; i < bitmap_mark_size * table_slice_size; i += thSize) {
        cur_row = i / bitmap_mark_size;
        cur_start = cur_row * bitmap_size;
        cur_ID = i >> 5;
        bitmap_mark_offset = __ldg(&aux_struct[table_ID].bitmap_mark_offset[i]);
        bitmap_mark_offset += bitmap_mark_size * cur_row;
#ifndef LTPMG_GPULAUNCHER_BLOCK_SCAN_OPT_SHM_BITMAP_POPULAR
        for (uint32_t k = cur_start; k < cur_ID; k += 32) {
            if (k + laneID < cur_ID) {
                SHM_part_res[threadIdx.x] = aux_struct[table_ID].bitmap_tmp[k + laneID];
            } else {
                SHM_part_res[threadIdx.x] = 0;
            }
            __syncwarp();
            uint32_t part_sum = SHM_part_res[threadIdx.x];
            for (uint32_t j = 16; j > 0; j >>= 1) {
                part_sum += __shfl_up_sync(0xffffffff, part_sum, j);
            }
            part_sum = __shfl_sync(0xffffffff, part_sum, 31);
            bitmap_mark_offset += part_sum;
        }
#endif

#ifdef LTPMG_GPULAUNCHER_BLOCK_SCAN_OPT_SHM_BITMAP_POPULAR
        bitmap_mark_offset += __ldg(&aux_struct[table_ID].bitmap_tmp[cur_ID]);
#endif

#ifndef LTPMG_GPULAUNCHER_SCAN_OPT_SHM_BITMAP_POPULAR
        for (uint32_t j=cur_start;j < cur_ID; ++j) {
            bitmap_mark_offset +=aux_struct[table_ID].bitmap_tmp[j];
        }
#endif

        if (aux_struct[table_ID].bitmap_mark[i] != 0) {
            aux_struct[table_ID].bitmap_mark_compressed[bitmap_mark_offset] =
                    __ldg(&aux_struct[table_ID].bitmap_mark[i]);
        }
    }
}

__device__ void popular_handler(uint32_t device_ID,
                                uint32_t table_ID,
                                uint32_t row,
                                uint32_t cur_bitmap,
                                Global_Table_Info *table_info,
                                Global_Table *tables,
                                Global_Table_Meta *metainfo,
                                Global_Txn *txn,
                                Global_Txn_Exec_Param *exec_param,
                                Global_Txn_Aux_Struct *aux_struct,
                                Global_Data_Packet *data_packet) {
    const uint32_t warpID = threadIdx.x >> 5;
    const uint32_t laneID = threadIdx.x & 0x1f;
    const uint32_t bitmap_size = exec_param[0].bitmap_size;
    const uint32_t cur_row = row % metainfo[table_ID].table_slice_size;
    const uint32_t bitmark_start = bitmap_size * 32 * cur_row;
    __shared__ INT32 SHM_INT32_data[16 * 16];
    __shared__ UINT32 SHM_STRING_data[16 * 8 * 16];
    __shared__ DOUBLE SHM_DOUBLE_data[16 * 16];
    for (uint32_t i = laneID;
         i < __ldg(&table_info[table_ID].int_size);
         i += 32) {
        SHM_INT32_data[(threadIdx.x >> 5) + i] =
                tables[table_ID].int_data[cur_row * __ldg(&table_info[table_ID].int_size) + i];
    }
    for (uint32_t i = laneID;
         i < __ldg(&table_info[table_ID].string_size) * __ldg(&table_info[table_ID].string_length);
         i += 32) {
        SHM_STRING_data[(threadIdx.x >> 5) + i] =
                tables[table_ID].string_data[cur_row *
                                             __ldg(&table_info[table_ID].string_size) *
                                             __ldg(&table_info[table_ID].string_length) +
                                             i];
    }
    for (uint32_t i = laneID;
         i < __ldg(&table_info[table_ID].double_size);
         i += 32) {
        SHM_DOUBLE_data[(threadIdx.x >> 5) + i] =
                tables[table_ID].double_data[cur_row *
                                             __ldg(&table_info[table_ID].double_size) +
                                             i];
    }
    __syncwarp();

#ifdef LTPMG_GPULAUNCHER_SCAN_BITMAP_POPULAR
    __shared__ uint32_t SHM_bitmap[32];

    for (uint32_t b = 0; b < bitmap_size; ++b) {
        if (laneID == 0) {
            SHM_bitmap[warpID] = aux_struct[table_ID].bitmap[b + bitmap_size * cur_row];
        }
        __syncwarp();
        if (SHM_bitmap[warpID] == 0) {
            continue;
        }
        uint32_t result = SHM_bitmap[warpID] & (1 << laneID);

        // SHM_bitmark[threadIdx.x] = aux_struct[table_ID].bitmap_mark[bitmark_start + b * 32 + laneID];

#pragma unroll
        for (uint32_t curlane = 0; curlane < 32; ++curlane) {
            uint32_t cur_res = __shfl_sync(0xffffffff, result, curlane);
            __syncwarp();
            // uint32_t mark = SHM_bitmark[threadIdx.x & 0xffffffe0 + curlane];
            if (cur_res != 0) {
                uint32_t mark = aux_struct[table_ID].bitmap_mark[bitmark_start + b * 32 + curlane];
                uint32_t type = txn[mark].subtxn.type;

                if (type == 0) {
                    // select
                    for (uint32_t i = laneID; i < table_info[table_ID].int_size; i += 32) {
                        // select_operator<INT32>(table_ID, cur_row, i, 1, tables, table_info, metainfo);
                        select_operator_shared<INT32>(SHM_INT32_data[warpID * 16 + i]);
                    }
                    for (uint32_t i = laneID;
                         i < table_info[table_ID].string_size * table_info[table_ID].string_length; i += 32) {
                        // select_operator<UINT32>(table_ID, cur_row, i, 0, tables, table_info, metainfo);
                        select_operator_shared<UINT32>(SHM_STRING_data[warpID * 16 * 8 + i]);
                    }
                    for (uint32_t i = laneID; i < table_info[table_ID].double_size; i += 32) {
                        // select_operator<DOUBLE>(table_ID, cur_row, i, 3, tables, table_info, metainfo);
                        select_operator_shared<DOUBLE>(SHM_DOUBLE_data[warpID * 16 + i]);
                    }
                } else if (type == 4) {
                    for (uint32_t i = laneID; i < table_info[table_ID].int_size; i += 32) {
                        // scan_operator<INT32>(table_ID, cur_row, i, 1, tables, table_info, metainfo);
                        scan_operator_shared<INT32>(SHM_INT32_data[warpID * 16 + i]);
                    }
                    for (uint32_t i = laneID;
                         i < table_info[table_ID].string_size * table_info[table_ID].string_length; i += 32) {
                        // scan_operator<UINT32>(table_ID, cur_row, i, 0, tables, table_info, metainfo);
                        scan_operator_shared<UINT32>(SHM_STRING_data[warpID * 16 * 8 + i]);
                    }
                    for (uint32_t i = laneID; i < table_info[table_ID].double_size; i += 32) {
                        // scan_operator<DOUBLE>(table_ID, cur_row, i, 3, tables, table_info, metainfo);
                        scan_operator_shared<DOUBLE>(SHM_DOUBLE_data[warpID * 16 + i]);
                    }
                } else if (type == 2) {
                    // update
                    for (uint32_t i = laneID; i < table_info[table_ID].int_size; i += 32) {
                        // update_operator<INT32>(table_ID, cur_row, i, 1, 0, tables, table_info, metainfo);
                        update_operator_shared<INT32>(0, SHM_INT32_data[warpID * 16 + i]);
                    }
                    for (uint32_t i = laneID;
                         i < table_info[table_ID].string_size * table_info[table_ID].string_length; i += 32) {
                        // update_operator<UINT32>(table_ID, cur_row, i, 0, 0, tables, table_info, metainfo);
                        update_operator_shared<UINT32>(0, SHM_STRING_data[warpID * 16 * 8 + i]);
                    }
                    for (uint32_t i = laneID; i < table_info[table_ID].double_size; i += 32) {
                        // update_operator<DOUBLE>(table_ID, cur_row, i, 3, 0.0, tables, table_info, metainfo);
                        update_operator_shared<DOUBLE>(0.0, SHM_DOUBLE_data[warpID * 16 + i]);
                    }
                } else if (type == 3) {
                    // delete
                    for (uint32_t i = laneID; i < table_info[table_ID].int_size; i += 32) {
                        // delete_operator<INT32>(table_ID, cur_row, i, 1, tables, table_info, metainfo);
                        delete_operator_shared<INT32>(SHM_INT32_data[warpID * 16 + i]);
                    }
                    for (uint32_t i = laneID;
                         i < table_info[table_ID].string_size * table_info[table_ID].string_length; i += 32) {
                        // delete_operator<UINT32>(table_ID, cur_row, i, 0, tables, table_info, metainfo);
                        delete_operator_shared<UINT32>(SHM_STRING_data[warpID * 16 * 8 + i]);
                    }
                    for (uint32_t i = laneID; i < table_info[table_ID].double_size; i += 32) {
                        // delete_operator<DOUBLE>(table_ID, cur_row, i, 3, tables, table_info, metainfo);
                        delete_operator_shared<DOUBLE>(SHM_DOUBLE_data[warpID * 16 + i]);
                    }
                } else if (type == 1) {
                    // insert
                    for (uint32_t i = laneID; i < table_info[table_ID].int_size; i += 32) {
                        // insert_operator<INT32>(table_ID, cur_row, i, 1, 0, tables, table_info, metainfo);
                        insert_operator_shared<INT32>(0, SHM_INT32_data[warpID * 16 + i]);
                    }
                    for (uint32_t i = laneID;
                         i < table_info[table_ID].string_size * table_info[table_ID].string_length; i += 32) {
                        // insert_operator<UINT32>(table_ID, cur_row, i, 0, 0, tables, table_info, metainfo);
                        insert_operator_shared<UINT32>(0, SHM_STRING_data[warpID * 16 * 8 + i]);
                    }
                    for (uint32_t i = laneID; i < table_info[table_ID].double_size; i += 32) {
                        // insert_operator<DOUBLE>(table_ID, cur_row, i, 3, 0.0, tables, table_info, metainfo);
                        insert_operator_shared<DOUBLE>(0.0, SHM_DOUBLE_data[warpID * 16 + i]);
                    }
                }
            }
            __syncwarp();
        }
    }
#endif

#ifdef LTPMG_GPULAUNCHER_SCAN_OPT_BITMAP_POPULAR
    __shared__ uint32_t SHM_mark[512];
    const uint32_t SHM_mark_offset_start = threadIdx.x & 0xffffffe0;
    const uint32_t bitmap_used_size = aux_struct[table_ID].bitmap_used_size[cur_row];
    for (uint32_t b = 0; b < bitmap_used_size; b += 32) {
        // uint32_t mark = aux_struct[table_ID].bitmap_mark_compressed[bitmark_start + b];
        if (b + laneID < bitmap_used_size) {
            SHM_mark[threadIdx.x] = aux_struct[table_ID].bitmap_mark_compressed[bitmark_start + b + laneID];
        } else {
            SHM_mark[threadIdx.x] = 0;
        }
        __syncwarp();

        for (uint32_t j = 0; j < 32 && SHM_mark[SHM_mark_offset_start + j] != 0; ++j) {
            uint32_t mark = SHM_mark[SHM_mark_offset_start + j];
            // if (laneID == 0 && mark > 0) {
            //     printf("mark:%d\n", mark);
            // }
            // continue;
            uint32_t type = txn[mark].subtxn.type;

            if (type == 0) {
                // select
                for (uint32_t i = laneID; i < table_info[table_ID].int_size; i += 32) {
                    // select_operator<INT32>(table_ID, cur_row, i, 1, tables, table_info, metainfo);
                    select_operator_shared<INT32>(SHM_INT32_data[warpID * 16 + i]);
                }
                for (uint32_t i = laneID;
                     i < table_info[table_ID].string_size * table_info[table_ID].string_length; i += 32) {
                    // select_operator<UINT32>(table_ID, cur_row, i, 0, tables, table_info, metainfo);
                    select_operator_shared<UINT32>(SHM_STRING_data[warpID * 16 * 8 + i]);
                }
                for (uint32_t i = laneID; i < table_info[table_ID].double_size; i += 32) {
                    // select_operator<DOUBLE>(table_ID, cur_row, i, 3, tables, table_info, metainfo);
                    select_operator_shared<DOUBLE>(SHM_DOUBLE_data[warpID * 16 + i]);
                }
            } else if (type == 4) {
                for (uint32_t i = laneID; i < table_info[table_ID].int_size; i += 32) {
                    // scan_operator<INT32>(table_ID, cur_row, i, 1, tables, table_info, metainfo);
                    scan_operator_shared<INT32>(SHM_INT32_data[warpID * 16 + i]);
                }
                for (uint32_t i = laneID;
                     i < table_info[table_ID].string_size * table_info[table_ID].string_length; i += 32) {
                    // scan_operator<UINT32>(table_ID, cur_row, i, 0, tables, table_info, metainfo);
                    scan_operator_shared<UINT32>(SHM_STRING_data[warpID * 16 * 8 + i]);
                }
                for (uint32_t i = laneID; i < table_info[table_ID].double_size; i += 32) {
                    // scan_operator<DOUBLE>(table_ID, cur_row, i, 3, tables, table_info, metainfo);
                    scan_operator_shared<DOUBLE>(SHM_DOUBLE_data[warpID * 16 + i]);
                }
            } else if (type == 2) {
                // update
                for (uint32_t i = laneID; i < table_info[table_ID].int_size; i += 32) {
                    // update_operator<INT32>(table_ID, cur_row, i, 1, 0, tables, table_info, metainfo);
                    update_operator_shared<INT32>(0, SHM_INT32_data[warpID * 16 + i]);
                }
                for (uint32_t i = laneID;
                     i < table_info[table_ID].string_size * table_info[table_ID].string_length; i += 32) {
                    // update_operator<UINT32>(table_ID, cur_row, i, 0, 0, tables, table_info, metainfo);
                    update_operator_shared<UINT32>(0, SHM_STRING_data[warpID * 16 * 8 + i]);
                }
                for (uint32_t i = laneID; i < table_info[table_ID].double_size; i += 32) {
                    // update_operator<DOUBLE>(table_ID, cur_row, i, 3, 0.0, tables, table_info, metainfo);
                    update_operator_shared<DOUBLE>(0.0, SHM_DOUBLE_data[warpID * 16 + i]);
                }
            } else if (type == 3) {
                // delete
                for (uint32_t i = laneID; i < table_info[table_ID].int_size; i += 32) {
                    // delete_operator<INT32>(table_ID, cur_row, i, 1, tables, table_info, metainfo);
                    delete_operator_shared<INT32>(SHM_INT32_data[warpID * 16 + i]);
                }
                for (uint32_t i = laneID;
                     i < table_info[table_ID].string_size * table_info[table_ID].string_length; i += 32) {
                    // delete_operator<UINT32>(table_ID, cur_row, i, 0, tables, table_info, metainfo);
                    delete_operator_shared<UINT32>(SHM_STRING_data[warpID * 16 * 8 + i]);
                }
                for (uint32_t i = laneID; i < table_info[table_ID].double_size; i += 32) {
                    // delete_operator<DOUBLE>(table_ID, cur_row, i, 3, tables, table_info, metainfo);
                    delete_operator_shared<DOUBLE>(SHM_DOUBLE_data[warpID * 16 + i]);
                }
            } else if (type == 1) {
                // insert
                for (uint32_t i = laneID; i < table_info[table_ID].int_size; i += 32) {
                    // insert_operator<INT32>(table_ID, cur_row, i, 1, 0, tables, table_info, metainfo);
                    insert_operator_shared<INT32>(0, SHM_INT32_data[warpID * 16 + i]);
                }
                for (uint32_t i = laneID;
                     i < table_info[table_ID].string_size * table_info[table_ID].string_length; i += 32) {
                    // insert_operator<UINT32>(table_ID, cur_row, i, 0, 0, tables, table_info, metainfo);
                    insert_operator_shared<UINT32>(0, SHM_STRING_data[warpID * 16 * 8 + i]);
                }
                for (uint32_t i = laneID; i < table_info[table_ID].double_size; i += 32) {
                    // insert_operator<DOUBLE>(table_ID, cur_row, i, 3, 0.0, tables, table_info, metainfo);
                    insert_operator_shared<DOUBLE>(0.0, SHM_DOUBLE_data[warpID * 16 + i]);
                }
            }
        }
        __syncwarp();
    }
#endif

    for (uint32_t i = laneID; i < table_info[table_ID].int_size; i += 32) {
        tables[table_ID].int_data[cur_row * table_info[table_ID].int_size + i] = SHM_INT32_data[(threadIdx.x >> 5) + i];
    }
    for (uint32_t i = laneID;
         i < table_info[table_ID].string_size * table_info[table_ID].string_length;
         i += 32) {
        tables[table_ID].string_data[cur_row *
                                     table_info[table_ID].string_size *
                                     table_info[table_ID].string_length + i] = SHM_STRING_data[(threadIdx.x >> 5) + i];
    }
    for (uint32_t i = laneID; i < table_info[table_ID].double_size; i += 32) {
        tables[table_ID].double_data[cur_row * table_info[table_ID].double_size + i] = SHM_DOUBLE_data[
            (threadIdx.x >> 5) + i];
    }
    __syncwarp();
}


__global__ void txn_analyze_popular(uint32_t device_ID,
                                    uint32_t table_ID,
                                    uint32_t benchmark,
                                    Global_Table_Info *table_info,
                                    Global_Table *tables,
                                    Global_Table_Index *indexes,
                                    Global_Table_Meta *metainfo,
                                    Global_Table_Strategy *strategy,
                                    Global_Txn_Info *txn_info,
                                    Global_Txn *txn,
                                    Global_Txn_Exec *txn_exec,
                                    Global_Txn_Result *txn_result,
                                    Global_Txn_Exec_Param *exec_param,
                                    Global_Txn_Aux_Struct *aux_struct,
                                    Global_Data_Packet *data_packet) {
    uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    // const uint32_t bitmap_size = exec_param[0].bitmap_size;
    uint32_t cur = 0;
    cur = thID >> 5; // cur_row
    if (benchmark == 1) {
        while (cur >= metainfo[table_ID].row_start && cur < metainfo[table_ID].row_end) {
            popular_handler(device_ID, table_ID, cur, 0, table_info, tables, metainfo,
                            txn, exec_param, aux_struct, data_packet);
            cur += (blockDim.x * gridDim.x) >> 5;
        }
    } else if (benchmark == 4) {
        while (cur < 100) {
            popular_handler(device_ID, table_ID, cur, 0, table_info, tables, metainfo,
                            txn, exec_param, aux_struct, data_packet);
            cur += (blockDim.x * gridDim.x) >> 5;
        }
    } else if (benchmark == 2 || benchmark == 3) {
        // benchmark TPCC_PART, warehouse district has bitmap
        if (table_ID == 0 || table_ID == 1) {
            while (cur >= metainfo[table_ID].row_start && cur < metainfo[table_ID].row_end) {
                popular_handler(device_ID, table_ID, cur, 0, table_info, tables, metainfo,
                                txn, exec_param, aux_struct, data_packet);
                cur += (blockDim.x * gridDim.x) >> 5;
            }
        }
    }
}

__device__ void select_analyze(uint32_t device_ID,
                               uint32_t cur_txn,
                               Global_Table_Info *table_info,
                               Global_Table *tables,
                               Global_Table_Index *indexes,
                               Global_Table_Meta *metainfo,
                               Global_Table_Strategy *strategy,
                               Global_Txn_Info *txn_info,
                               Global_Txn *txn,
                               Global_Txn_Exec *txn_exec,
                               Global_Txn_Result *txn_result,
                               Global_Txn_Exec_Param *exec_param,
                               Global_Txn_Aux_Struct *aux_struct,
                               Global_Data_Packet *data_packet) {
    uint32_t mark = txn_exec[0].select_txn_mark[cur_txn];
    uint32_t __cur_txn = mark;
    uint32_t ispopular = txn[__cur_txn].subtxn.ispopular;
    // uint32_t cur_tid = txn[__cur_txn].subtxn.TID;
    uint32_t row_1 = txn[__cur_txn].subtxn.dest_Row_1;
    uint32_t tableID = txn[__cur_txn].subtxn.table_ID;

    uint32_t cur = row_1 % metainfo[tableID].table_slice_size;
    if (ispopular == 1) {
    } else if (ispopular == 0) {
        // uint32_t mintITD = aux_struct[tableID].min_TID[cur];
        // if (mintITD == cur_tid) {
        //     // commit
        //     for (uint32_t i = 0; i < table_info[tableID].int_size; ++i) {
        //         select_operator<INT32>(tableID, cur, i, 1, tables, table_info, metainfo);
        //     }
        //     for (uint32_t i = 0; i < table_info[tableID].string_size * table_info[tableID].string_length; ++i) {
        //         select_operator<UINT32>(tableID, cur, i, 0, tables, table_info, metainfo);
        //     }
        //     for (uint32_t i = 0; i < table_info[tableID].double_size; ++i) {
        //         select_operator<DOUBLE>(tableID, cur, i, 3, tables, table_info, metainfo);
        //     }
        // } else
        {
            // plan redo
            uint32_t start_offset = aux_struct[tableID].mark_TID_offset[cur];
            uint32_t inside_offset = atomicAdd(&aux_struct[tableID].tmp_TID[cur], 1);
            aux_struct[0].mark_TID[start_offset + inside_offset] = mark;
        }
    }
}

__device__ void update_analyze(uint32_t device_ID,
                               uint32_t cur_txn,
                               Global_Table_Info *table_info,
                               Global_Table *tables,
                               Global_Table_Index *indexes,
                               Global_Table_Meta *metainfo,
                               Global_Table_Strategy *strategy,
                               Global_Txn_Info *txn_info,
                               Global_Txn *txn,
                               Global_Txn_Exec *txn_exec,
                               Global_Txn_Result *txn_result,
                               Global_Txn_Exec_Param *exec_param,
                               Global_Txn_Aux_Struct *aux_struct,
                               Global_Data_Packet *data_packet) {
    uint32_t mark = txn_exec[0].update_txn_mark[cur_txn];
    uint32_t __cur_txn = mark;
    uint32_t ispopular = txn[__cur_txn].subtxn.ispopular;
    // uint32_t tid = txn[__cur_txn].subtxn.TID;
    uint32_t row_1 = txn[__cur_txn].subtxn.dest_Row_1;
    uint32_t tableID = txn[__cur_txn].subtxn.table_ID;

    uint32_t cur = row_1 % metainfo[tableID].table_slice_size;

    if (ispopular == 1) {
    } else if (ispopular == 0) {
        // uint32_t min_tid = aux_struct[tableID].min_TID[cur];
        // if (min_tid == tid) {
        //     // commit
        //     INT32 int_data = 0;
        //     for (uint32_t i = 0; i < table_info[tableID].int_size; ++i) {
        //         int_data = 0;
        //         update_operator<INT32>(tableID, cur, i, 1, int_data, tables, table_info, metainfo);
        //     }
        //     UINT32 string_data = 0;
        //     for (uint32_t i = 0; i < table_info[tableID].string_size * table_info[tableID].string_length;
        //          i += 32) {
        //         string_data = 0;
        //         update_operator<UINT32>(tableID, cur, i, 0, string_data, tables, table_info, metainfo);
        //     }
        //     DOUBLE double_data = 0.0;
        //     for (uint32_t i = 0; i < table_info[tableID].double_size; ++i) {
        //         double_data = 0.0;
        //         update_operator<DOUBLE>(tableID, cur, i, 3, double_data, tables, table_info, metainfo);
        //     }
        // } else
        {
            // plan redo
            uint32_t start_offset = aux_struct[tableID].mark_TID_offset[cur];
            uint32_t inside_offset = atomicAdd(&aux_struct[tableID].tmp_TID[cur], 1);
            aux_struct[0].mark_TID[start_offset + inside_offset] = mark;
        }
    }
}

__device__ void insert_analyze(uint32_t device_ID,
                               uint32_t cur_txn,
                               Global_Table_Info *table_info,
                               Global_Table *tables,
                               Global_Table_Index *indexes,
                               Global_Table_Meta *metainfo,
                               Global_Table_Strategy *strategy,
                               Global_Txn_Info *txn_info,
                               Global_Txn *txn,
                               Global_Txn_Exec *txn_exec,
                               Global_Txn_Result *txn_result,
                               Global_Txn_Exec_Param *exec_param,
                               Global_Txn_Aux_Struct *aux_struct,
                               Global_Data_Packet *data_packet) {
    uint32_t mark = txn_exec[0].insert_txn_mark[cur_txn];
    uint32_t __cur_txn = mark;
    uint32_t ispopular = txn[__cur_txn].subtxn.ispopular;
    // uint32_t tid = txn[__cur_txn].subtxn.TID;
    uint32_t row_1 = txn[__cur_txn].subtxn.dest_Row_1;
    uint32_t tableID = txn[__cur_txn].subtxn.table_ID;

    uint32_t cur = row_1 % metainfo[tableID].table_slice_size;

    if (ispopular == 1) {
    } else if (ispopular == 0) {
        // uint32_t min_tid = aux_struct[tableID].min_TID[cur];
        // if (min_tid == tid) {
        //     // commit
        //     INT32 int_data = 0;
        //     for (uint32_t i = 0; i < table_info[tableID].int_size; ++i) {
        //         int_data = 0;
        //         insert_operator<INT32>(tableID, cur, i, 1, int_data, tables, table_info, metainfo);
        //     }
        //     UINT32 string_data = 0;
        //     for (uint32_t i = 0; i < table_info[tableID].string_size * table_info[tableID].string_length; ++i) {
        //         string_data = 0;
        //         insert_operator<UINT32>(tableID, cur, i, 0, string_data, tables, table_info, metainfo);
        //     }
        //     DOUBLE double_data = 0.0;
        //     for (uint32_t i = 0; i < table_info[tableID].double_size; ++i) {
        //         double_data = 0.0;
        //         insert_operator<DOUBLE>(tableID, cur, i, 3, double_data, tables, table_info, metainfo);
        //     }
        // } else
        {
            // plan redo
            uint32_t start_offset = aux_struct[tableID].mark_TID_offset[cur];
            uint32_t inside_offset = atomicAdd(&aux_struct[tableID].tmp_TID[cur], 1);
            aux_struct[0].mark_TID[start_offset + inside_offset] = mark;
        }
    }
}

__device__ void scan_analyze(uint32_t device_ID,
                             uint32_t cur_txn,
                             Global_Table_Info *table_info,
                             Global_Table *tables,
                             Global_Table_Index *indexes,
                             Global_Table_Meta *metainfo,
                             Global_Table_Strategy *strategy,
                             Global_Txn_Info *txn_info,
                             Global_Txn *txn,
                             Global_Txn_Exec *txn_exec,
                             Global_Txn_Result *txn_result,
                             Global_Txn_Exec_Param *exec_param,
                             Global_Txn_Aux_Struct *aux_struct,
                             Global_Data_Packet *data_packet) {
    uint32_t mark = txn_exec[0].scan_txn_mark[cur_txn];
    uint32_t __cur_txn = mark;
    uint32_t ispopular = txn[__cur_txn].subtxn.ispopular;
    // uint32_t tid = txn[__cur_txn].subtxn.TID;
    uint32_t row_1 = txn[__cur_txn].subtxn.dest_Row_1;
    uint32_t row_2 = txn[__cur_txn].subtxn.dest_Row_2;
    uint32_t tableID = txn[__cur_txn].subtxn.table_ID;


    if (ispopular == 1) {
    } else if (ispopular == 0) {
        uint32_t row_start = metainfo[tableID].row_start;
        uint32_t row_end = metainfo[tableID].row_end;

        // bool canCommit = true;
        // for (uint32_t row = row_1; row < row_2; ++row) {
        //     if (row >= row_start && row < row_end) {
        //         uint32_t cur = row % metainfo[tableID].table_slice_size;
        //         uint32_t min_tid = aux_struct[tableID].min_TID[row];
        //         if (min_tid != tid) {
        //             canCommit = false;
        //             break;
        //         }
        //     }
        // }
        // if (canCommit) {
        //     // commit
        //     for (uint32_t row = row_1; row < row_2; ++row) {
        //         if (row >= row_start && row < row_end) {
        //             for (uint32_t i = 0; i < table_info[tableID].int_size; ++i) {
        //                 scan_operator<INT32>(tableID, row, i, 1, tables, table_info, metainfo);
        //             }
        //             for (uint32_t i = 0;
        //                  i < table_info[tableID].string_size * table_info[tableID].string_length; ++i) {
        //                 scan_operator<UINT32>(tableID, row, i, 0, tables, table_info, metainfo);
        //             }
        //             for (uint32_t i = 0; i < table_info[tableID].double_size; ++i) {
        //                 scan_operator<DOUBLE>(tableID, row, i, 3, tables, table_info, metainfo);
        //             }
        //         }
        //     }
        // } else
        {
            for (uint32_t row = row_1; row < row_2; ++row) {
                if (row >= row_start && row < row_end) {
                    uint32_t cur = row % metainfo[tableID].table_slice_size;
                    uint32_t start_offset = aux_struct[tableID].mark_TID_offset[cur];
                    uint32_t inside_offset = atomicAdd(&aux_struct[tableID].tmp_TID[cur], 1);
                    aux_struct[0].mark_TID[start_offset + inside_offset] = mark;
                }
            }
        }
    }
}

__device__ void delete_analyze(uint32_t device_ID,
                               uint32_t cur_txn,
                               Global_Table_Info *table_info,
                               Global_Table *tables,
                               Global_Table_Index *indexes,
                               Global_Table_Meta *metainfo,
                               Global_Table_Strategy *strategy,
                               Global_Txn_Info *txn_info,
                               Global_Txn *txn,
                               Global_Txn_Exec *txn_exec,
                               Global_Txn_Result *txn_result,
                               Global_Txn_Exec_Param *exec_param,
                               Global_Txn_Aux_Struct *aux_struct,
                               Global_Data_Packet *data_packet) {
    uint32_t mark = txn_exec[0].delete_txn_mark[cur_txn];
    uint32_t __cur_txn = mark;
    uint32_t ispopular = txn[__cur_txn].subtxn.ispopular;
    // uint32_t tid = txn[__cur_txn].subtxn.TID;
    uint32_t row_1 = txn[__cur_txn].subtxn.dest_Row_1;
    uint32_t tableID = txn[__cur_txn].subtxn.table_ID;

    uint32_t cur = row_1 % metainfo[tableID].table_slice_size;

    if (ispopular == 1) {
    } else if (ispopular == 0) {
        // uint32_t min_tid = aux_struct[tableID].min_TID[cur];
        // if (min_tid == tid) {
        //     // commit
        //     for (uint32_t i = 0; i < table_info[tableID].int_size; ++i) {
        //         delete_operator<INT32>(tableID, cur, i, 1, tables, table_info, metainfo);
        //     }
        //     for (uint32_t i = 0; i < table_info[tableID].string_size * table_info[tableID].string_length; ++i) {
        //         delete_operator<UINT32>(tableID, cur, i, 0, tables, table_info, metainfo);
        //     }
        //     for (uint32_t i = 0; i < table_info[tableID].double_size; ++i) {
        //         delete_operator<DOUBLE>(tableID, cur, i, 3, tables, table_info, metainfo);
        //     }
        // } else
        {
            // plan redo
            uint32_t start_offset = aux_struct[tableID].mark_TID_offset[cur];
            uint32_t inside_offset = atomicAdd(&aux_struct[tableID].tmp_TID[cur], 1);
            aux_struct[0].mark_TID[start_offset + inside_offset] = mark;
        }
    }
}

__global__ void txn_analyze_regular(uint32_t device_ID,
                                    uint32_t device_cnt,
                                    uint32_t benchmark,
                                    Global_Table_Info *table_info,
                                    Global_Table *tables,
                                    Global_Table_Index *indexes,
                                    Global_Table_Meta *metainfo,
                                    Global_Table_Strategy *strategy,
                                    Global_Txn_Info *txn_info,
                                    Global_Txn *txn,
                                    Global_Txn_Exec *txn_exec,
                                    Global_Txn_Result *txn_result,
                                    Global_Txn_Exec_Param *exec_param,
                                    Global_Txn_Aux_Struct *aux_struct,
                                    Global_Data_Packet *data_packet) {
    uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t cur = 0;

    cur = thID;
    // cur = atomicAdd(&txn_exec[0].select_tmp, 1);
    while (cur < txn_exec[0].select_cur) {
        select_analyze(device_ID, cur, table_info, tables, indexes, metainfo, strategy,
                       txn_info, txn, txn_exec, txn_result, exec_param, aux_struct, data_packet);
        cur += blockDim.x * gridDim.x;
        // cur = atomicAdd(&txn_exec[0].select_tmp, 1);
    }

    cur = thID;
    // cur = atomicAdd(&txn_exec[0].update_tmp, 1);
    while (cur < txn_exec[0].update_cur) {
        update_analyze(device_ID, cur, table_info, tables, indexes, metainfo, strategy,
                       txn_info, txn, txn_exec, txn_result, exec_param, aux_struct, data_packet);
        // cur += blockDim.x * gridDim.x;
        cur = atomicAdd(&txn_exec[0].update_tmp, 1);
    }

    cur = thID;
    // cur = atomicAdd(&txn_exec[0].insert_tmp, 1);
    while (cur < txn_exec[0].insert_cur) {
        insert_analyze(device_ID, cur, table_info, tables, indexes, metainfo, strategy,
                       txn_info, txn, txn_exec, txn_result, exec_param, aux_struct, data_packet);
        // cur += blockDim.x * gridDim.x;
        cur = atomicAdd(&txn_exec[0].insert_tmp, 1);
    }

    cur = thID;
    // cur = atomicAdd(&txn_exec[0].scan_tmp, 1);
    while (cur < txn_exec[0].scan_cur) {
        scan_analyze(device_ID, cur, table_info, tables, indexes, metainfo, strategy,
                     txn_info, txn, txn_exec, txn_result, exec_param, aux_struct, data_packet);
        cur += blockDim.x * gridDim.x;
        // cur = atomicAdd(&txn_exec[0].scan_tmp, 1);
    }

    cur = thID;
    // cur = atomicAdd(&txn_exec[0].delete_tmp, 1);
    while (cur < txn_exec[0].delete_cur) {
        delete_analyze(device_ID, cur, table_info, tables, indexes, metainfo, strategy,
                       txn_info, txn, txn_exec, txn_result, exec_param, aux_struct, data_packet);
        cur += blockDim.x * gridDim.x;
        // cur = atomicAdd(&txn_exec[0].delete_tmp, 1);
    }
}

__global__ void filter_commit(uint32_t device_ID,
                              uint32_t device_cnt,
                              uint32_t table_ID,
                              Global_Table_Info *table_info,
                              Global_Table *tables,
                              Global_Table_Index *indexes,
                              Global_Table_Meta *metainfo,
                              Global_Table_Strategy *strategy,
                              Global_Txn_Info *txn_info,
                              Global_Txn *txn,
                              Global_Txn_Exec *txn_exec,
                              Global_Txn_Result *txn_result,
                              Global_Txn_Exec_Param *exec_param,
                              Global_Txn_Aux_Struct *aux_struct,
                              Global_Data_Packet *data_packet) {
    uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t step = (blockDim.x * gridDim.x) >> 5;
    uint32_t laneID = threadIdx.x & 0x1f;
    uint32_t used_rows_offset = thID >> 5;

    while (used_rows_offset < aux_struct[table_ID].used_rows_cnt) {
        const uint32_t row = aux_struct[table_ID].used_rows[used_rows_offset];
        const uint32_t cur_row = row % metainfo[table_ID].table_slice_size;
        const uint32_t txn_cnt = aux_struct[table_ID].tmp_TID[cur_row];
        const uint32_t start_offset = aux_struct[table_ID].mark_TID_offset[cur_row];
        if (txn_cnt > 1) {
            uint32_t size = 0;
            uint32_t left = 0;
            uint32_t right = 0;
            uint32_t offset = 0;
            uint32_t sub_size = 0;
            uint32_t tid = 0xffffffff;
            uint32_t mark = 0;
            uint32_t tid_r = 0xffffffff;
            uint32_t mark_r = 0;
            for (uint32_t i = 2; i <= (1 << 32 - __clz(txn_cnt)); i *= 2) {
                for (uint32_t start = i * laneID; start < txn_cnt; start += i * 32) {
                    size = i;
                    if (start + size >= txn_cnt) {
                        size = txn_cnt - start;
                    }
                    if (start < txn_cnt) {
                        sub_size = i >> 2;
                        offset = start;
                        left = start;
                        right = start + sub_size;
                        while (left < start + sub_size && right < start + size) {
                            mark = aux_struct[0].mark_TID[left + start_offset];
                            tid = txn[mark].subtxn.TID;
                            mark_r = aux_struct[0].mark_TID[right + start_offset];
                            tid_r = txn[mark_r].subtxn.TID;
                            if (tid < tid_r) {
                                aux_struct[0].merge_tmp[offset + start_offset] =
                                        aux_struct[0].mark_TID[left + start_offset];
                                ++left;
                            } else {
                                aux_struct[0].merge_tmp[offset + start_offset] =
                                        aux_struct[0].mark_TID[right + start_offset];
                                ++right;
                            }
                            ++offset;
                        }
                        while (left < start + sub_size) {
                            aux_struct[0].merge_tmp[offset + start_offset] =
                                    aux_struct[0].mark_TID[left + start_offset];
                            ++left;
                            ++offset;
                        }
                        while (right < start + size) {
                            aux_struct[0].merge_tmp[offset + start_offset] =
                                    aux_struct[0].mark_TID[right + start_offset];
                            ++right;
                            ++offset;
                        }
                    }
                }
                for (uint32_t j = laneID; j < txn_cnt; j += 32) {
                    aux_struct[0].mark_TID[start_offset + j] = aux_struct[0].merge_tmp[start_offset + j];
                }
            }
        }
        __syncwarp();
        used_rows_offset += step;
    }
}

__device__ void commit(uint32_t device_ID,
                       uint32_t table_ID,
                       uint32_t row,
                       Global_Table_Info *table_info,
                       Global_Table *tables,
                       Global_Table_Meta *metainfo,
                       Global_Txn *txn,
                       Global_Txn_Exec_Param *exec_param,
                       Global_Txn_Aux_Struct *aux_struct) {
    const uint32_t warpID = threadIdx.x >> 5;
    const uint32_t laneID = threadIdx.x & 0x1f;
    const uint32_t cur_row = row % metainfo[table_ID].table_slice_size;
    const uint32_t txn_cnt = aux_struct[table_ID].tmp_TID[cur_row];
    const uint32_t start_offset = aux_struct[table_ID].mark_TID_offset[cur_row];

    __shared__ INT32 SHM_INT32_data[16 * 16];
    __shared__ UINT32 SHM_STRING_data[16 * 8 * 16];
    __shared__ DOUBLE SHM_DOUBLE_data[16 * 16];
    for (uint32_t i = laneID; i < table_info[table_ID].int_size; i += 32) {
        SHM_INT32_data[(threadIdx.x >> 5) + i] = tables[table_ID].int_data[cur_row * table_info[table_ID].int_size + i];
    }
    for (uint32_t i = laneID;
         i < table_info[table_ID].string_size * table_info[table_ID].string_length;
         i += 32) {
        SHM_STRING_data[(threadIdx.x >> 5) + i] = tables[table_ID].string_data[cur_row *
                                                                               table_info[table_ID].string_size *
                                                                               table_info[table_ID].string_length + i];
    }
    for (uint32_t i = laneID; i < table_info[table_ID].double_size; i += 32) {
        SHM_DOUBLE_data[(threadIdx.x >> 5) + i] = tables[table_ID].double_data[
            cur_row * table_info[table_ID].double_size + i];
    }
    __syncwarp();
    uint32_t cur = 0;
    __shared__ uint32_t SHM_mark[512];
    const uint32_t SHM_mark_offset_start = threadIdx.x & 0xffffffe0;
    while (cur < txn_cnt) {
        //execute
        // uint32_t mark = aux_struct[0].mark_TID[start_offset + cur];
        // ++cur;
        if (cur + laneID < txn_cnt) {
            SHM_mark[threadIdx.x] = aux_struct[0].mark_TID[start_offset + cur + laneID];
        } else {
            SHM_mark[threadIdx.x] = 0;
        }
        __syncwarp();
        cur += 32;
        for (uint32_t j = 0; j < 32 && SHM_mark[SHM_mark_offset_start + j] > 0; ++j) {
            uint32_t mark = SHM_mark[SHM_mark_offset_start + j];
            uint32_t type = txn[mark].subtxn.type;

            if (type == 0) {
                // select
                for (uint32_t i = laneID; i < table_info[table_ID].int_size; i += 32) {
                    // select_operator<INT32>(table_ID, cur_row, i, 1, tables, table_info, metainfo);
                    select_operator_shared<INT32>(SHM_INT32_data[warpID * 16 + i]);
                }
                for (uint32_t i = laneID;
                     i < table_info[table_ID].string_size * table_info[table_ID].string_length; i += 32) {
                    // select_operator<UINT32>(table_ID, cur_row, i, 0, tables, table_info, metainfo);
                    select_operator_shared<UINT32>(SHM_STRING_data[warpID * 16 * 8 + i]);
                }
                for (uint32_t i = laneID; i < table_info[table_ID].double_size; i += 32) {
                    // select_operator<DOUBLE>(table_ID, cur_row, i, 3, tables, table_info, metainfo);
                    select_operator_shared<DOUBLE>(SHM_DOUBLE_data[warpID * 16 + i]);
                }
            } else if (type == 4) {
                for (uint32_t i = laneID; i < table_info[table_ID].int_size; i += 32) {
                    // scan_operator<INT32>(table_ID, cur_row, i, 1, tables, table_info, metainfo);
                    scan_operator_shared<INT32>(SHM_INT32_data[warpID * 16 + i]);
                }
                for (uint32_t i = laneID;
                     i < table_info[table_ID].string_size * table_info[table_ID].string_length; i += 32) {
                    // scan_operator<UINT32>(table_ID, cur_row, i, 0, tables, table_info, metainfo);
                    scan_operator_shared<UINT32>(SHM_STRING_data[warpID * 16 * 8 + i]);
                }
                for (uint32_t i = laneID; i < table_info[table_ID].double_size; i += 32) {
                    // scan_operator<DOUBLE>(table_ID, cur_row, i, 3, tables, table_info, metainfo);
                    scan_operator_shared<DOUBLE>(SHM_DOUBLE_data[warpID * 16 + i]);
                }
            } else if (type == 2) {
                // update
                for (uint32_t i = laneID; i < table_info[table_ID].int_size; i += 32) {
                    // update_operator<INT32>(table_ID, cur_row, i, 1, 0, tables, table_info, metainfo);
                    update_operator_shared<INT32>(0, SHM_INT32_data[warpID * 16 + i]);
                }
                for (uint32_t i = laneID;
                     i < table_info[table_ID].string_size * table_info[table_ID].string_length; i += 32) {
                    // update_operator<UINT32>(table_ID, cur_row, i, 0, 0, tables, table_info, metainfo);
                    update_operator_shared<UINT32>(0, SHM_STRING_data[warpID * 16 * 8 + i]);
                }
                for (uint32_t i = laneID; i < table_info[table_ID].double_size; i += 32) {
                    // update_operator<DOUBLE>(table_ID, cur_row, i, 3, 0.0, tables, table_info, metainfo);
                    update_operator_shared<DOUBLE>(0.0, SHM_DOUBLE_data[warpID * 16 + i]);
                }
            } else if (type == 3) {
                // delete
                for (uint32_t i = laneID; i < table_info[table_ID].int_size; i += 32) {
                    // delete_operator<INT32>(table_ID, cur_row, i, 1, tables, table_info, metainfo);
                    delete_operator_shared<INT32>(SHM_INT32_data[warpID * 16 + i]);
                }
                for (uint32_t i = laneID;
                     i < table_info[table_ID].string_size * table_info[table_ID].string_length; i += 32) {
                    // delete_operator<UINT32>(table_ID, cur_row, i, 0, tables, table_info, metainfo);
                    delete_operator_shared<UINT32>(SHM_STRING_data[warpID * 16 * 8 + i]);
                }
                for (uint32_t i = laneID; i < table_info[table_ID].double_size; i += 32) {
                    // delete_operator<DOUBLE>(table_ID, cur_row, i, 3, tables, table_info, metainfo);
                    delete_operator_shared<DOUBLE>(SHM_DOUBLE_data[warpID * 16 + i]);
                }
            } else if (type == 1) {
                // insert
                for (uint32_t i = laneID; i < table_info[table_ID].int_size; i += 32) {
                    // insert_operator<INT32>(table_ID, cur_row, i, 1, 0, tables, table_info, metainfo);
                    insert_operator_shared<INT32>(0, SHM_INT32_data[warpID * 16 + i]);
                }
                for (uint32_t i = laneID;
                     i < table_info[table_ID].string_size * table_info[table_ID].string_length; i += 32) {
                    // insert_operator<UINT32>(table_ID, cur_row, i, 0, 0, tables, table_info, metainfo);
                    insert_operator_shared<UINT32>(0, SHM_STRING_data[warpID * 16 * 8 + i]);
                }
                for (uint32_t i = laneID; i < table_info[table_ID].double_size; i += 32) {
                    // insert_operator<DOUBLE>(table_ID, cur_row, i, 3, 0.0, tables, table_info, metainfo);
                    insert_operator_shared<DOUBLE>(0.0, SHM_DOUBLE_data[warpID * 16 + i]);
                }
            }
            __syncwarp();
        }
    }

    for (uint32_t i = laneID; i < table_info[table_ID].int_size; i += 32) {
        tables[table_ID].int_data[cur_row * table_info[table_ID].int_size + i] =
                SHM_INT32_data[(threadIdx.x >> 5) + i];
    }
    for (uint32_t i = laneID;
         i < table_info[table_ID].string_size * table_info[table_ID].string_length;
         i += 32) {
        tables[table_ID].string_data[cur_row *
                                     table_info[table_ID].string_size *
                                     table_info[table_ID].string_length + i] =
                SHM_STRING_data[(threadIdx.x >> 5) + i];
    }
    for (uint32_t i = laneID; i < table_info[table_ID].double_size; i += 32) {
        tables[table_ID].double_data[cur_row * table_info[table_ID].double_size + i] =
                SHM_DOUBLE_data[(threadIdx.x >> 5) + i];
    }
}


__global__ void txn_commit(uint32_t device_ID,
                           uint32_t device_cnt,
                           uint32_t table_ID,
                           Global_Table_Info *table_info,
                           Global_Table *tables,
                           Global_Table_Index *indexes,
                           Global_Table_Meta *metainfo,
                           Global_Table_Strategy *strategy,
                           Global_Txn_Info *txn_info,
                           Global_Txn *txn,
                           Global_Txn_Exec *txn_exec,
                           Global_Txn_Result *txn_result,
                           Global_Txn_Exec_Param *exec_param,
                           Global_Txn_Aux_Struct *aux_struct,
                           Global_Data_Packet *data_packet) {
    uint32_t thID = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t step = (blockDim.x * gridDim.x) >> 5;

    uint32_t row = 0; // cur_row
    uint32_t offset = thID >> 5;
    while (offset < aux_struct[table_ID].used_rows_cnt) {
        row = aux_struct[table_ID].used_rows[offset];
        commit(device_ID, table_ID, row, table_info, tables, metainfo,
               txn, exec_param, aux_struct);
        offset += step;
    }
}
